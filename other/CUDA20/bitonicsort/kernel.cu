#include "hip/hip_runtime.h"
//pass
//--blockDim=[32,1] --gridDim=[1,1]

#include <hip/hip_runtime.h>

#define NUM 32

__global__ void BitonicKernel(int * values)
{
  extern __shared__ int shared[NUM];

  unsigned int tid = threadIdx.x;

  // Copy input to shared mem.
  shared[tid] = values[tid];

#ifdef MUTATION
  if (threadIdx.x == 0) {
#endif
  __syncthreads();
#ifdef MUTATION
   /* BUGINJECT: NON_UNIFORM_CONTROL_FLOW, UP */
  }
#endif

  // Parallel bitonic sort.
  for (unsigned int k = 2;
       k <= NUM; k *= 2)
  {
    // Bitonic merge:
    for (unsigned int j = k / 2;
       j>0; j /= 2)
    {
      unsigned int ixj = tid ^ j;

      if (ixj > tid)
      {
        if ((tid & k) == 0)
        {
          if (shared[tid] > shared[ixj])
          {
            unsigned int tmp = shared[tid];
            shared[tid] = shared[ixj];
            shared[ixj] = shared[tid];
          }
        }
        else
        {
          if (shared[tid] < shared[ixj])
          {
            unsigned int tmp = shared[tid];
            shared[tid] = shared[ixj];
            shared[ixj] = shared[tid];
          }
        }
      }

      __syncthreads();
    }
  }

  // Write result.
  values[tid] = shared[tid];
}
