//pass
//--gridDim=128 --blockDim=128

#include <hip/hip_runtime.h>

__global__ void uniformAdd(float *g_data, 
                           float *uniforms, 
                           int n, 
                           int blockOffset, 
                           int baseIndex)
{
    __shared__ float uni[1];
    if (threadIdx.x == 0)
        uni[0] = uniforms[blockIdx.x + blockOffset];
         /* BUGINJECT: MUTATE_OFFSET, UP, ZERO */
    
    unsigned int address = blockIdx.x * (blockDim.x << 1) + baseIndex + threadIdx.x; 

    __syncthreads();
    
    // note two adds per thread
#ifdef MUTATION // couldn't apply mutation above; apply here instead
    g_data[0]                    += uni[0];
#else
    g_data[address]              += uni[0];
#endif
    g_data[address + blockDim.x] += (threadIdx.x + blockDim.x < n) * uni[0];
}

