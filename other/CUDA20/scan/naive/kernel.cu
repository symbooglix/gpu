#include "hip/hip_runtime.h"
//pass
//--blockDim=[32,1] --gridDim=[1,1]

#include <hip/hip_runtime.h>

#define N 32

///////////////////////////////////////////////////////////////////////////////
//! Naive compute implementation of scan, one thread per element
//! Not work efficient: log(n) steps, but n * (log(n) - 1) adds.
//! Not shared storage efficient either -- this requires ping-ponging
//! arrays in shared memory due to hazards so 2 * n storage space.
//!
//! Pro: Simple
//! Con: Not work efficient
//!
//! @param g_odata  output data in global memory
//! @param g_idata  input data in global memory
//! @param n        input number of elements to scan from input data
///////////////////////////////////////////////////////////////////////////////
__global__ void kernel(float *g_odata, float *g_idata, int n)
{
    __requires(n == blockDim.x); //< n is a pow2 and equal to blockDim.x

    // REVISIT: removed extern
    // REVISIT: give temp static size
    // Dynamically allocated shared memory for scan kernels
    /*extern*/__shared__  float temp[N*2];

    int thid = threadIdx.x;

    int pout = 0;
    int pin = 1;

    // Cache the computational window in shared memory
    temp[pout*n + thid] = (thid > 0) ? g_idata[thid-1] : 0;

    for (int offset = 1;
         offset < n; offset *= 2)
    {

        pout = 1 - pout;
        pin  = 1 - pout;

#ifndef MUTATION
        /* BUGINJECT: REMOVE_BARRIER, DOWN */
        __syncthreads();
#endif
        temp[pout*n+thid] = temp[pin*n+thid];

        if (thid >= offset) {
             temp[pout*n+thid] += temp[pin*n+thid - offset];
        }
    }

    __syncthreads();

    g_odata[thid] = temp[pout*n+thid];
}
