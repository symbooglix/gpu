//pass
//--blockDim=[17,17] --gridDim=[1,1]

#include <hip/hip_runtime.h>

// code example	for blog: Use extent instead of grid class - Sample 2
//                 created by: Tamer Afify              Date:1/1/2012

//This sample shows how to replace grid with extent in the 
//previously illustrated image blur solution.
//For code porting process follow those three simple steps;
//1. Wherever grid type or array/aray_view value is used replace with extent
//2. If array is constructed with a grid origin index value, then whenever 
//   this array is used add the origin index to its index value.
//3. If the compute domain grid - for parallel_for_each � is constructed with origin, 
//   add this origin to every index use in the kernel.

// Note: to compile this code you need to use Visual Studio 2011 Beta Release

#define width 17
#define height 17


__global__ void boxblur(float* blurimage, float* img, int originX, int originY)
{
    int idxX = blockIdx.x*blockDim.x + threadIdx.x;
    int idxY = blockIdx.y*blockDim.y + threadIdx.y;
    float r = 0.0f;
    int samples = 0;
    idxX += originX;
    idxY += originY;

    for (int dy = -1; dy <= 1; dy++)
    {
      for (int dx = -1; dx <= 1; dx++)
      {
        r += img[(idxY+dy)*width + idxX + dx];
        samples++;
      }
    }

    blurimage[idxY*width + idxX] = r/samples;
#if MUTATION
    blurimage[idxY*width + idxX + 1] = blurimage[idxY*width + idxX + 1];
#endif

}
