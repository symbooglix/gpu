//pass
//--blockDim=[17,17] --gridDim=[1,1]

#include <hip/hip_runtime.h>

// code example	for blog: Use extent instead of grid class - Sample 1
//                 created by: Tamer Afify              Date:1/1/2012

//This is a sample function for using the grid class to do an image blur
//The pixel blur can be performed by changing every pixel color RBG band to 
//the arithmetic average of this pixel value with all its 8 neighbors� pixels.

//The grid offset feature can be of great benefit when the compute domain origin 
//is different from the data origin. In other words, (0,0) for the data is not 
//matching the (0,0) starting point for computation.

//In this sample we will use this feature to blur the inner image box without the 
//boarder pixels as they don�t have 8 neighbors pixel. So the compute domain origin 
//is (1, 1) in the data index. And also compute domain extent is smaller than data 
//extent by 2 rows and 2 columns.

// Note: to compile this code you need to use C++ AMP Developer Preview destributed
// During the TAP progrm.

#define width 17
#define height 17

__global__ void boxblur(float* blurimage, float* img)
{
    int idxX = blockIdx.x*blockDim.x + threadIdx.x;
    int idxY = blockIdx.y*blockDim.y + threadIdx.y;

    float r = 0.0f;
    int samples = 0;

    for (int dy = -1; dy <= 1; dy++)
    {
      for (int dx = -1; dx <= 1; dx++)
      {
        r += img[(idxY + dy)*width + idxX + dx];
        samples++;
      }
    }

    blurimage[idxY*width + idxX] = r/samples;
#if MUTATION
    blurimage[idxY*width + idxX + 1] = blurimage[idxY*width + idxX + 1];
#endif

}
