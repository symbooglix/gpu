#include "hip/hip_runtime.h"
//pass
//--blockDim=[64,64] --gridDim=[8,8]

#include <hip/hip_runtime.h>

#define dimX 512
#define dimY 512

//--------------------------------------------------------------------------------------
// File: ocean_simulator.cpp
//
// Main class of ocean simulation
//
// Copyright (c) Microsoft Corporation. All rights reserved.
//--------------------------------------------------------------------------------------

#define dmap_dim 512

#define actual_dim dmap_dim
#define input_width (actual_dim + 4)
    // We use full sized data here. The value "output_width" should be actual_dim/2+1 though.
#define output_width actual_dim
#define output_height actual_dim
#define dtx_offset (actual_dim * actual_dim)
#define dty_offset (actual_dim * actual_dim * 2)


// Pre-FFT data preparation
__global__ void update_spectrum(
                     const float2* input_h0,
                     const float* input_omega,
                     float2* output_ht, 
                     unsigned int immutable_actualdim,
                     unsigned int immutable_inwidth,
                     unsigned int immutable_outwidth,
                     unsigned int immutable_outheight,
                     unsigned int immutable_dddressoffset,
                     unsigned int immutable_addressoffset,
                     float perframe_time)
{
    __requires(immutable_actualdim == 512 /*actual_dim*/);
    __requires(immutable_inwidth == 516 /*input_width*/);
    __requires(immutable_outwidth == 512 /*output_width*/);
    __requires(immutable_outheight == 512 /*output_height*/);
    __requires(immutable_dddressoffset == 512*512 /*dtx_offset*/);
    __requires(immutable_addressoffset == 512*512*2 /*dty_offset*/);
    {
        int in_index = (blockIdx.y * blockDim.y + threadIdx.y) * immutable_inwidth + (blockIdx.x * blockDim.x + threadIdx.x);
        int in_mindex = (immutable_actualdim - (blockIdx.y * blockDim.y + threadIdx.y)) * immutable_inwidth + (immutable_actualdim - (blockIdx.x * blockDim.x + threadIdx.x));
        int out_index = (blockIdx.y * blockDim.y + threadIdx.y) * immutable_outwidth + (blockIdx.x * blockDim.x + threadIdx.x);

        // H(0) -> H(t)
        float2 h0_k  = input_h0[in_index];
        float2 h0_mk = input_h0[in_mindex];
        float sin_v, cos_v;

      //sin_v = sincos(input_omega[in_index] * perframe_time, &cos_v);
        sin_v = sin(input_omega[in_index] * perframe_time);
        cos_v = cos(input_omega[in_index] * perframe_time);

        float2 ht;
        ht.x = (h0_k.x + h0_mk.x) * cos_v - (h0_k.y + h0_mk.y) * sin_v;
        ht.y = (h0_k.x - h0_mk.x) * sin_v + (h0_k.y - h0_mk.y) * cos_v;

        // H(t) -> Dx(t), Dy(t)
        float kx = (blockIdx.x * blockDim.x + threadIdx.x) - immutable_actualdim * 0.5f;
        float ky = (blockIdx.y * blockDim.y + threadIdx.y) - immutable_actualdim * 0.5f;
        float sqr_k = kx * kx + ky * ky;
        float rsqr_k = 0;
        if (sqr_k > 1e-12f) {
            rsqr_k = 1 / sqrt(sqr_k);
        }
        kx *= rsqr_k;
        ky *= rsqr_k;

        float2 dt_x;

        dt_x.x = ht.y * kx;
        dt_x.y = -ht.x * kx;

        float2 dt_y;

        dt_y.x = ht.y * ky;
        dt_y.y = -ht.x * ky;

        if (((blockIdx.x * blockDim.x + threadIdx.x) < immutable_outwidth) && 
            ((blockIdx.y * blockDim.y + threadIdx.y) < immutable_outwidth))
        {
            output_ht[out_index] = ht;
            output_ht[out_index + immutable_dddressoffset] = dt_x;
            output_ht[out_index + immutable_addressoffset] = dt_y;		
#ifdef MUTATION
            output_ht[out_index+1] = output_ht[out_index+1];
               /* BUGINJECT: ADD_ACCESS, UP */
#endif
        }    
    }
}

