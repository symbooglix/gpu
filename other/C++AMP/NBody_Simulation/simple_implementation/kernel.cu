#include "hip/hip_runtime.h"
//pass
//--blockDim=256 --gridDim=1024

#include <hip/hip_runtime.h>

#define TILE_SIZE							256
#define SOFTENING_SQUARED 					0.0000015625f
#define _FG									(6.67300e-11f*10000.0f)
#define F_PARTICLE_MASS						(_FG*10000.0f*10000.0f)
#define DELTA_TIME 							0.1f
#define DAMPENING 							1.0f

#define UINT unsigned int


// GPU based functions
static __attribute__((always_inline))
__device__ void bodybody_interaction(float4 *acc, const float4 my_curr_pos, float4 other_element_old_pos)
{
    float4 r = other_element_old_pos - my_curr_pos;
    
    float dist_sqr = r.x*r.x + r.y*r.y + r.z*r.z;
    dist_sqr += SOFTENING_SQUARED;
    
    float inv_dist = rsqrt(dist_sqr);
    float inv_dist_cube =  inv_dist*inv_dist*inv_dist;
    
    float s = F_PARTICLE_MASS*inv_dist_cube;
	
    (*acc) += r*s;
}

__global__ void simple_implementation(float4* data_in_pos, float4* data_in_vel, float4* data_out_pos, float4* data_out_vel, unsigned int num_bodies)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
	{
        float4 p_pos;
        float4 p_vel;

        p_pos = data_in_pos[idx];
        p_vel = data_in_vel[idx];
      //float4 acc = (float4)(0, 0, 0, 0);
        float4 acc;
        acc.x = 0; acc.y = 0; acc.z = 0; acc.w = 0;

        // Update current particle using all other particles
        for (UINT j = 0; j < num_bodies; j++) 
        {
	        bodybody_interaction(&acc, p_pos, data_in_pos[j]);
        }

        p_vel += acc*DELTA_TIME;
        p_vel *= DAMPENING;

        p_pos += p_vel*DELTA_TIME;

        data_out_pos[idx] = p_pos;
        data_out_vel[idx] = p_vel;
#ifdef MUTATION
        data_out_vel[idx+1] = data_out_vel[idx+1];
         /* BUGINJECT: ADD_ACCESS, UP */
#endif
	}
}
