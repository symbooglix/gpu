#include "hip/hip_runtime.h"
//pass
//--blockDim=256 --gridDim=1024

#include <hip/hip_runtime.h>

#define TILE_SIZE							256
#define SOFTENING_SQUARED 					0.0000015625f
#define _FG									(6.67300e-11f*10000.0f)
#define F_PARTICLE_MASS						(_FG*10000.0f*10000.0f)
#define DELTA_TIME 							0.1f
#define DAMPENING 							1.0f


#define to_d3dxv4(X) X
#define to_float4(X) X

#define D3DXVECTOR4 float4

#define UINT unsigned int

// GPU based functions
static __attribute__((always_inline))
__device__ void bodybody_interaction(float4* acc, const float4 my_curr_pos, float4 other_element_old_pos)
{
    float4 r = other_element_old_pos - my_curr_pos;
    
    float dist_sqr = r.x*r.x + r.y*r.y + r.z*r.z;
    dist_sqr += SOFTENING_SQUARED;
    
    float inv_dist = rsqrt(dist_sqr);
    float inv_dist_cube =  inv_dist*inv_dist*inv_dist;
    
    float s = F_PARTICLE_MASS*inv_dist_cube;
	
    (*acc) += r*s;
}

__global__ void tiling_implementation(float4* data_in_pos, float4* data_in_vel, float4* data_out_pos, float4* data_out_vel, int offset, int size, int num_bodies)
{
    UINT num_of_tiles = num_bodies/TILE_SIZE;

	{
		__shared__ D3DXVECTOR4 tile_mem[TILE_SIZE];
    
		int idx_local = threadIdx.x;
		int idx_global = blockIdx.x * blockDim.x + threadIdx.x;
    
		idx_global += offset;

		float4 p_pos;
    float4 p_vel;
		p_pos = data_in_pos[idx_global];
		p_vel = data_in_vel[idx_global];
		//float4 acc = (float4)(0, 0, 0, 0);
		float4 acc;
    acc.x = 0; acc.y = 0; acc.z = 0; acc.w = 0;
    
		// Update current particle using all other particles
		int particle_idx = idx_local;
		for (UINT tile = 0;
                                tile <num_of_tiles; tile++)
		{
			// Cache a tile of particles into shared memory to increase IO efficiency
			tile_mem[idx_local] = to_d3dxv4(data_in_pos[particle_idx]);

#ifndef MUTATION
       /* BUGINJECT: REMOVE_BARRIER, DOWN */
      __syncthreads();
#endif
        
      // Unroll size should be multile of TILE_SIZE
			// Unrolling 4 helps improve perf on both ATI and nVidia cards
			// 4 is the sweet spot - increasing further adds no perf improvement while decreasing reduces perf
			for (UINT j = 0; j < TILE_SIZE; j+=4 )
			{
				bodybody_interaction(&acc, p_pos, to_float4(tile_mem[j+0]));
				bodybody_interaction(&acc, p_pos, to_float4(tile_mem[j+1]));
				bodybody_interaction(&acc, p_pos, to_float4(tile_mem[j+2]));
				bodybody_interaction(&acc, p_pos, to_float4(tile_mem[j+3]));
			}
      __syncthreads();
      particle_idx += TILE_SIZE;
		}

    p_vel += acc*DELTA_TIME;
		p_vel *= DAMPENING;

		p_pos += p_vel*DELTA_TIME;

		data_out_pos[idx_global] = p_pos;
		data_out_vel[idx_global] = p_vel;
	}

}
