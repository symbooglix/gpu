//pass
//--blockDim=256 --gridDim=256

#include <hip/hip_runtime.h>

//////////////////////////////////////////////////////////////////////////////
//// THIS CODE AND INFORMATION IS PROVIDED "AS IS" WITHOUT WARRANTY OF
//// ANY KIND, EITHER EXPRESSED OR IMPLIED, INCLUDING BUT NOT LIMITED TO
//// THE IMPLIED WARRANTIES OF MERCHANTABILITY AND/OR FITNESS FOR A
//// PARTICULAR PURPOSE.
////
//// Copyright (c) Microsoft Corporation. All rights reserved
//////////////////////////////////////////////////////////////////////////////

//----------------------------------------------------------------------------
// File: Histogram.cpp
// 
// Implements histogram in C++ AMP
// Refer README.txt
//----------------------------------------------------------------------------

#define histogram_bin_count 256 /* Bin count */

#define log2_thread_size 5U
#define thread_count 8 /* number of partial histogram per tile */

#define histogram256_tile_size (thread_count * (1U << log2_thread_size))
#define histogram256_tile_static_memory (thread_count * histogram_bin_count)

#define merge_tile_size histogram_bin_count /* Partial result Merge size */
#define partial_histogram256_count (thread_count * (1U << log2_thread_size))

// This function aggregates partial results
__global__ void histo_merge_kernel(unsigned int* partial_result, unsigned int* histogram_amp)
{

        {
            unsigned sum = 0;
            for (unsigned i = threadIdx.x;
                   i < partial_histogram256_count * histogram_bin_count; i += merge_tile_size)
            {
                sum += partial_result[blockIdx.x + i * histogram_bin_count];
            }

            __shared__ unsigned s_data[merge_tile_size];
            s_data[threadIdx.x] = sum;

            // parallel reduce within a tile
            for (int stride = merge_tile_size / 2;
                     stride > 0; stride >>= 1)
            {
#ifndef MUTATION
                 /* BUGINJECT: REMOVE_BARRIER, DOWN */
                __syncthreads();
#endif
                if (threadIdx.x < stride)
                {
                    s_data[threadIdx.x] += s_data[threadIdx.x + stride];
                }
            }

            // tile sum is updated to result array by zero-th thread
            if (threadIdx.x == 0)
            {
                histogram_amp[blockIdx.x] = s_data[0];
            }
        }
}
