//pass
//--blockDim=[128,128] --gridDim=[4,4]

#include <hip/hip_runtime.h>

//////////////////////////////////////////////////////////////////////////////
//// THIS CODE AND INFORMATION IS PROVIDED "AS IS" WITHOUT WARRANTY OF
//// ANY KIND, EITHER EXPRESSED OR IMPLIED, INCLUDING BUT NOT LIMITED TO
//// THE IMPLIED WARRANTIES OF MERCHANTABILITY AND/OR FITNESS FOR A
//// PARTICULAR PURPOSE.
////
//// Copyright (c) Microsoft Corporation. All rights reserved
//////////////////////////////////////////////////////////////////////////////

//----------------------------------------------------------------------------
// File: Convolution.cpp
// 
// Implement C++ AMP based simple and tiled version of Convolution filter used in 
// image processing.
//----------------------------------------------------------------------------

#define DEFAULT_WIDTH   512
#define DEFAULT_HEIGHT  512
// TILE_SIZE should be multiple of both DEFAULT_WIDTH and DEFAULT_HEIGHT
#define TILE_SIZE		128

#define radius 7

#define width DEFAULT_WIDTH
#define height DEFAULT_HEIGHT

#define clamp(a, b, c) ((a) < (b) ? (b) : ((a) > (c) ? (c) : (a)))

//----------------------------------------------------------------------------
// Simple implementation of convolution filter along different dimension
//----------------------------------------------------------------------------
static __attribute__((always_inline)) __device__ float convolution_dim_simple(const float* img, const float* filter)
{
    float sum = 0.0f;
    for (int k = -radius; k <= radius; k++)
    {
        int dim = clamp((blockDim.y*blockIdx.y + threadIdx.y) + k, 0, height-1);

        int aIdxX = (blockDim.x*blockIdx.x + threadIdx.x);
        int aIdxY = dim;

        int kidx = k + radius;
        sum += img[aIdxY*width + aIdxX]*filter[kidx];
    }
    return sum;
}

//----------------------------------------------------------------------------
// Simple implementation of convolution separable filter 
//----------------------------------------------------------------------------
__global__ void convolution_simple(float* v_img, float* v_filter, float* v_result)
{
  v_result[(blockDim.y*blockIdx.y + threadIdx.y)*width + (blockDim.x*blockIdx.x + threadIdx.x)] = convolution_dim_simple(v_img, v_filter);
#ifdef MUTATION
  v_result[(blockDim.y*blockIdx.y + threadIdx.y)*width + (blockDim.x*blockIdx.x + threadIdx.x) + 1] = v_result[(blockDim.y*blockIdx.y + threadIdx.y)*width + (blockDim.x*blockIdx.x + threadIdx.x) + 1];
   /* BUGINJECT: ADD_ACCESS, UP */
#endif
}
