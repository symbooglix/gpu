//pass
//--blockDim=[8,8] --gridDim=[1,1]

#include <hip/hip_runtime.h>

#define _2D_ACCESS(A, y, x, X_DIM) A[(y)*(X_DIM)+(x)]

//////////////////////////////////////////////////////////////////////////////
//// THIS CODE AND INFORMATION IS PROVIDED "AS IS" WITHOUT WARRANTY OF
//// ANY KIND, EITHER EXPRESSED OR IMPLIED, INCLUDING BUT NOT LIMITED TO
//// THE IMPLIED WARRANTIES OF MERCHANTABILITY AND/OR FITNESS FOR A
//// PARTICULAR PURPOSE.
////
//// Copyright (c) Microsoft Corporation. All rights reserved
//////////////////////////////////////////////////////////////////////////////

//----------------------------------------------------------------------------
// File: TransitiveClosure.cpp
//
// Contains the implementation of algorithms which explores connectivity between 
// nodes in a graph and determine shortest path.
// This is based on paper http://www.seas.upenn.edu/~kiderj/research/papers/APSP-gh08-fin-T.pdf
//----------------------------------------------------------------------------

// Defines to help with AMP->OpenCL translation
#define X_DIMENSION 0
#define Y_DIMENSION 1

// Constants - specifies tile size
#define TILE_SIZE (1 << 3)

// State of connection
#define UNCONNECTED 0
#define DIRECTLY_CONNECTED 1
#define INDIRECTLY_CONNECTED 2


#define num_vertices (1 << 6)

//----------------------------------------------------------------------------
// Stage3 - determine connectivity between vertexs' between 3 TILE 
// 1. primary block, 2. block made of row af current and column of primary 
// 3. block made of column of current and row of primary
//----------------------------------------------------------------------------
__global__ void transitive_closure_stage3_kernel(unsigned int* graph, int passnum)
{
    unsigned int group_id0, group_id1;
    if (blockIdx.y < passnum)
    {
        group_id0 = blockIdx.y;
    }
    else
    {
        group_id0 = blockIdx.y + 1;
    }

    if (blockIdx.x < passnum)
    {
        group_id1 = blockIdx.x;
    }
    else
    {
        group_id1 = blockIdx.x + 1;
    }

    // Load block with same row as current block and same column as primary block into shared memory (shBuffer1)
    __shared__ unsigned int shbuffer1[TILE_SIZE][TILE_SIZE];

    int idxY = group_id0 * TILE_SIZE + threadIdx.y;
    int idxX = passnum * TILE_SIZE + threadIdx.x;
    shbuffer1[threadIdx.y][threadIdx.x] = _2D_ACCESS(graph, idxY, idxX, num_vertices);

    // Load block with same column as current block and same row as primary block into shared memory (shBuffer2)
    __shared__ unsigned int shBuffer2[TILE_SIZE][TILE_SIZE];
    idxY = passnum * TILE_SIZE + threadIdx.y;
    idxX = group_id1 * TILE_SIZE + threadIdx.x;
    shBuffer2[threadIdx.y][threadIdx.x] = _2D_ACCESS(graph, idxY, idxX, num_vertices);

    //  Load the current block into shared memory (shbuffer3)
    __shared__ unsigned int curr_block_buffer[TILE_SIZE][TILE_SIZE];
    idxY = group_id0 * TILE_SIZE + threadIdx.y;
    idxX = group_id1 * TILE_SIZE + threadIdx.x;
    curr_block_buffer[threadIdx.y][threadIdx.x] = _2D_ACCESS(graph, idxY, idxX, num_vertices);

#ifndef MUTATION
     /* BUGINJECT: REMOVE_BARRIER, DOWN */
    __syncthreads();
#endif

    // Now perform the actual Floyd-Warshall algorithm on this block
    for (unsigned int k = 0; k < TILE_SIZE; ++k)
    {
        if ( curr_block_buffer[threadIdx.y][threadIdx.x] == UNCONNECTED)
        {
            if ( (shbuffer1[threadIdx.y][k] != UNCONNECTED) && (shBuffer2[k][threadIdx.x] != UNCONNECTED) )
            {
                curr_block_buffer[threadIdx.y][threadIdx.x] = passnum*TILE_SIZE + k + INDIRECTLY_CONNECTED;
            }
        }

        __syncthreads();
    }

    _2D_ACCESS(graph, idxY, idxX, num_vertices) = curr_block_buffer[threadIdx.y][threadIdx.x];
}
