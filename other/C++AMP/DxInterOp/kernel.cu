//pass
//--blockDim=1024 --gridDim=128

#include <hip/hip_runtime.h>

//--------------------------------------------------------------------------------------
// File: ComputeEngine.h
//
// This is an AMPC++ implementation of a compute shader. It transforms a shape with a
// rotation of an angle THETA. 
//
// Copyright (c) Microsoft Corporation. All rights reserved.
//--------------------------------------------------------------------------------------


#define THETA 3.1415f/1024  

__global__ void run(float* data_refY, float* data_refX)
{
  // Rotate the vertex by angle THETA
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  data_refY[idx] = data_refY[idx] * cos(THETA) - data_refX[idx] * sin(THETA);
  data_refX[idx] = data_refY[idx] * sin(THETA) + data_refX[idx] * cos(THETA);
#ifdef MUTATION
  data_refX[idx+1] = data_refX[idx+1];
   /* BUGINJECT: ADD_ACCESS, UP */
#endif
}


