
#include <hip/hip_runtime.h>
//pass
//--gridDim=[64,1,1]       --blockDim=[128,1,1]

__global__ void kernelAddConstant(int *g_a, const int b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    g_a[idx] += b;
}
