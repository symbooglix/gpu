
#include <hip/hip_runtime.h>
//pass
//--gridDim=1                --blockDim=32

#ifndef IMPLEMENT_VOTE_INTRINSICS
__device__ unsigned int all(unsigned int);
__device__ unsigned int any(unsigned int);
#endif

__global__ void VoteAnyKernel3(bool *info, int warp_size)
{
    int tx = threadIdx.x;
    bool *offs = info + (tx * 3);

    // The following should hold true for the second and third warp
    *offs = any((tx >= (warp_size * 3) / 2));
    // The following should hold true for the "upper half" of the second warp,
    // and all of the third warp
    *(offs + 1) = (tx >= (warp_size * 3) / 2? true: false);

    // The following should hold true for the third warp only
    if (all((tx >= (warp_size * 3) / 2)))
    {
        *(offs + 2) = true;
    }
}
