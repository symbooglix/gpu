#include "hip/hip_runtime.h"
//pass
//--gridDim=[8,8,1]        --blockDim=[8,8,1]

texture<float, hipTextureTypeCubemap> tex;

__global__ void
transformKernel(float *g_odata, int width)
{
    __requires(width == 8*8 /*gridDim.x * blockDim.x*/);

    // calculate this thread's data point
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // 0.5f offset and division are necessary to access the original data points
    // in the texture (such that bilinear interpolation will not be activated).
    // For details, see also CUDA Programming Guide, Appendix D

    float u = ((x+0.5f) / (float) width) * 2.f - 1.f;
    float v = ((y+0.5f) / (float) width) * 2.f - 1.f;

    float cx, cy, cz;

    for (unsigned int face = 0; face < 6; face ++)
    {
        //Layer 0 is positive X face
        if (face == 0)
        {
            cx = 1;
            cy = -v;
            cz = -u;
        }
        //Layer 1 is negative X face
        else if (face == 1)
        {
            cx = -1;
            cy = -v;
            cz = u;
        }
        //Layer 2 is positive Y face
        else if (face == 2)
        {
            cx = u;
            cy = 1;
            cz = v;
        }
        //Layer 3 is negative Y face
        else if (face == 3)
        {
            cx = u;
            cy = -1;
            cz = -v;
        }
        //Layer 4 is positive Z face
        else if (face == 4)
        {
            cx = u;
            cy = -v;
            cz = 1;
        }
        //Layer 4 is negative Z face
        else if (face == 5)
        {
            cx = -u;
            cy = -v;
            cz = -1;
        }

        // read from texture, do expected transformation and write to global memory
        g_odata[face*width*width + y*width + x] = -texCubemap(tex, cx, cy, cz);
    }
}
