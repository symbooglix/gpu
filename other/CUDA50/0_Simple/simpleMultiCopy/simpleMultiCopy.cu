
#include <hip/hip_runtime.h>
//pass
//--gridDim=[8192,1,1]     --blockDim=[512,1,1]

__global__ void incKernel(int *g_out, int *g_in, int N, int inner_reps)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N)
    {
        for (int i=0; i<inner_reps; ++i)
        {
            g_out[idx] = g_in[idx] + 1;
        }
    }
}
