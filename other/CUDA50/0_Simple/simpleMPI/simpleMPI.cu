
#include <hip/hip_runtime.h>
//pass
//--gridDim=10000            --blockDim=256

__global__ void simpleMPIKernel(float *input, float *output)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    output[tid] = sqrt(input[tid]);
}
