
#include <hip/hip_runtime.h>
//pass
//--gridDim=8 --blockDim=512

__global__ void simpleKernel(int *dst, int *src, int num)
{
    // Dummy kernel
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    dst[idx] = src[idx] / num;
}
