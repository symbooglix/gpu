
#include <hip/hip_runtime.h>
//pass
//--gridDim=[4,1,1]        --blockDim=[256,1,1]

__global__ void sequence_gpu(int *d_ptr, int length)
{
    int elemID = blockIdx.x * blockDim.x + threadIdx.x;

    if (elemID < length)
    {
        unsigned int laneid;
        //This command gets the lane ID within the current warp
        asm("mov.u32 %0, %%laneid;" : "=r"(laneid));
        d_ptr[elemID] = laneid;
    }
}
