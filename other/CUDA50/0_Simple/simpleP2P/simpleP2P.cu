
#include <hip/hip_runtime.h>
//pass
//--gridDim=[32768,1,1] --blockDim=[512,1,1]

__global__ void SimpleKernel(float *src, float *dst)
{
    // Just a dummy kernel, doing enough for us to verify that everything
    // worked
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    dst[idx] = src[idx] * 2.0f;
}
