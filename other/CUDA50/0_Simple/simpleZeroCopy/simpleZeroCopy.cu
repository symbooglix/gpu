
#include <hip/hip_runtime.h>
//pass
//--gridDim=[4096,1,1]     --blockDim=[256,1,1]

__global__ void vectorAddGPU(float *a, float *b, float *c, int N)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx < N)
    {
        c[idx] = a[idx] + b[idx];
    }
}
