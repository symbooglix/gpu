
#include <hip/hip_runtime.h>
//pass
//--gridDim=64               --blockDim=256
  
__global__ void
testKernel(int *g_odata)
{
    // access thread id
    const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Test various atomic instructions

    // Arithmetic atomic instructions

    // Atomic addition
    atomicAdd(&g_odata[0], 10);

    // Atomic subtraction (final should be 0)
    atomicSub(&g_odata[1], 10);

    // Atomic exchange
    atomicExch(&g_odata[2], tid);

    // Atomic maximum
    atomicMax(&g_odata[3], tid);

    // Atomic minimum
    atomicMin(&g_odata[4], tid);

    // Atomic increment (modulo 17+1)
    atomicInc((unsigned int *)&g_odata[5], 17);

    // Atomic decrement
    atomicDec((unsigned int *)&g_odata[6], 137);

    // Atomic compare-and-swap
    atomicCAS(&g_odata[7], tid-1, tid);

    // Bitwise atomic instructions

    // Atomic AND
    atomicAnd(&g_odata[8], 2*tid+7);

    // Atomic OR
    atomicOr(&g_odata[9], 1 << tid);

    // Atomic XOR
    atomicXor(&g_odata[10], tid);
}
