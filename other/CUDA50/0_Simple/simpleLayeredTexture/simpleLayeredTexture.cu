#include "hip/hip_runtime.h"
//pass
//--gridDim=[64,64,1]      --blockDim=[8,8,1]

texture<float, hipTextureType2DLayered> tex;

__global__ void
transformKernel(float *g_odata, int width, int height, int layer)
{
    __requires(width  == 64*8 /*gridDim.x * blockDim.x*/);
    __requires(height == 64*8 /*gridDim.y * blockDim.y*/);
    __requires(layer  == 1);

    // calculate this thread's data point
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // 0.5f offset and division are necessary to access the original data points
    // in the texture (such that bilinear interpolation will not be activated).
    // For details, see also CUDA Programming Guide, Appendix D
    float u = (x+0.5f) / (float) width;
    float v = (y+0.5f) / (float) height;

    // read from texture, do expected transformation and write to global memory
    g_odata[layer*width*height + y*width + x] = -tex2DLayered(tex, u, v, layer) + layer;
}
