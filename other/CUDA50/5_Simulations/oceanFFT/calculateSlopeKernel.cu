#include "hip/hip_runtime.h"
//pass
//--gridDim=[32,32,1] --blockDim=[8,8,1]

__global__ void calculateSlopeKernel(float *h, float2 *slopeOut, unsigned int width, unsigned int height)
{
    __requires(width == 256);
    __requires(height == 256);

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int i = y*width+x;

    float2 slope = make_float2(0.0f, 0.0f);

    if ((x > 0) && (y > 0) && (x < width-1) && (y < height-1))
    {
        slope.x = h[i+1] - h[i-1];
        slope.y = h[i+width] - h[i-width];
    }

    slopeOut[i] = slope;
}
