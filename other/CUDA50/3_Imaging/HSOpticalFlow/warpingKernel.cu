#include "hip/hip_runtime.h"
//pass
//--gridDim=[10,40]      --blockDim=[32,6]

texture<float, 2, hipReadModeElementType> texToWarp;

__global__ void WarpingKernel(int width, int height, int stride,
                              const float *u, const float *v, float *out)
{
    __requires(width == 320);
    __requires(height == 240);
    __requires(stride == 320);
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;

    const int pos = ix + iy * stride;

    if (ix >= width || iy >= height) return;

    float x = ((float)ix + u[pos] + 0.5f) / (float)width;
    float y = ((float)iy + v[pos] + 0.5f) / (float)height;

    out[pos] = tex2D(texToWarp, x, y);
}
