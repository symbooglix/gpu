#include "hip/hip_runtime.h"
//pass
//--gridDim=[10,30]      --blockDim=[32,8]

texture<float, 2, hipReadModeElementType> texCoarse;

__global__ void UpscaleKernel(int width, int height, int stride, float scale, float *out)
{
    __requires(width == 320);
    __requires(height == 240);
    __requires(stride == 320);
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;

    if (ix >= width || iy >= height) return;

    float x = ((float)ix + 0.5f) / (float)width;
    float y = ((float)iy + 0.5f) / (float)height;

    // exploit hardware interpolation
    // and scale interpolated vector to match next pyramid level resolution
    out[ix + iy * stride] = tex2D(texCoarse, x, y) * scale;
}
