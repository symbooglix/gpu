#include "hip/hip_runtime.h"
//pass
//--gridDim=8 --blockDim=64

#include "common.h"

__global__ void
d_recursiveGaussian_rgba(uint *id, uint *od, int w, int h, float a0, float a1, float a2, float a3, float b1, float b2, float coefp, float coefn)
{
    __requires(w == 512);
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

    if (x >= w) return;

    id += x;    // advance pointers to correct column
    od += x;

    // forward pass
    float4 xp = make_float4(0.0f);  // previous input
    float4 yp = make_float4(0.0f);  // previous output
    float4 yb = make_float4(0.0f);  // previous output by 2
#if CLAMP_TO_EDGE
    xp = rgbaIntToFloat(*id);
    yb = coefp*xp;
    yp = yb;
#endif

    for (int y = 0;
         __global_invariant(__implies(x < w, (__ptr_offset_bytes(od)/sizeof(uint))%w == x)),
         __global_invariant(__write_implies(od, (__write_offset_bytes(od)/sizeof(uint))%w == x)),
         y < h; y++)
    {
        float4 xc = rgbaIntToFloat(*id);
        float4 yc = a0*xc + a1*xp - b1*yp - b2*yb;
        *od = rgbaFloatToInt(yc);
        id += w;
        od += w;    // move to next row
        xp = xc;
        yb = yp;
        yp = yc;
    }

    // reset pointers to point to last element in column
    id -= w;
    od -= w;

    // reverse pass
    // ensures response is symmetrical
    float4 xn = make_float4(0.0f);
    float4 xa = make_float4(0.0f);
    float4 yn = make_float4(0.0f);
    float4 ya = make_float4(0.0f);
#if CLAMP_TO_EDGE
    xn = xa = rgbaIntToFloat(*id);
    yn = coefn*xn;
    ya = yn;
#endif

    for (int y = h-1;
         __global_invariant(__implies(x < w, (__ptr_offset_bytes(od)/sizeof(uint))%w == x)),
         __global_invariant(__write_implies(od, (__write_offset_bytes(od)/sizeof(uint))%w == x)),
         __global_invariant(__read_implies(od, (__read_offset_bytes(od)/sizeof(uint))%w == x)),
         y >= 0; y--)
    {
        float4 xc = rgbaIntToFloat(*id);
        float4 yc = a2*xn + a3*xa - b1*yn - b2*ya;
        xa = xn;
        xn = xc;
        ya = yn;
        yn = yc;
        *od = rgbaFloatToInt(rgbaIntToFloat(*od) + yc);
        id -= w;
        od -= w;  // move to previous row
    }
}
