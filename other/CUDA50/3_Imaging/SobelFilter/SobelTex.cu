#include "hip/hip_runtime.h"
//pass
//--gridDim=512              --blockDim=384

#include "common.h"

__global__ void
SobelTex(Pixel *pSobelOriginal, unsigned int Pitch,
         int w, int h, float fScale)
{
    __requires(w == 512);
    __requires(Pitch == 512);

    unsigned char *pSobel =
        (unsigned char *)(((char *) pSobelOriginal)+blockIdx.x*Pitch);

    for (int i = threadIdx.x;
         __global_invariant(i % blockDim.x == threadIdx.x),
         __global_invariant(__write_implies(pSobelOriginal, __write_offset_bytes(pSobelOriginal)/Pitch == blockIdx.x)),
         __global_invariant(__write_implies(pSobelOriginal, __write_offset_bytes(pSobelOriginal)%Pitch%blockDim.x == threadIdx.x)),
         i < w; i += blockDim.x)
    {
        unsigned char pix00 = tex2D(tex, (float) i-1, (float) blockIdx.x-1);
        unsigned char pix01 = tex2D(tex, (float) i+0, (float) blockIdx.x-1);
        unsigned char pix02 = tex2D(tex, (float) i+1, (float) blockIdx.x-1);
        unsigned char pix10 = tex2D(tex, (float) i-1, (float) blockIdx.x+0);
        unsigned char pix11 = tex2D(tex, (float) i+0, (float) blockIdx.x+0);
        unsigned char pix12 = tex2D(tex, (float) i+1, (float) blockIdx.x+0);
        unsigned char pix20 = tex2D(tex, (float) i-1, (float) blockIdx.x+1);
        unsigned char pix21 = tex2D(tex, (float) i+0, (float) blockIdx.x+1);
        unsigned char pix22 = tex2D(tex, (float) i+1, (float) blockIdx.x+1);
        pSobel[i] = ComputeSobel(pix00, pix01, pix02,
                                 pix10, pix11, pix12,
                                 pix20, pix21, pix22, fScale);
    }
}
