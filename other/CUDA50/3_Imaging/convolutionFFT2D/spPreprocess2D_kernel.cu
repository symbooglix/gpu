#include "hip/hip_runtime.h"
//pass
//--gridDim=4096             --blockDim=256

#include "common.h"

__global__ void spPreprocess2D_kernel(
    fComplex *d_Dst,
    fComplex *d_Src,
    uint DY,
    uint DX,
    uint threadCount,
    uint padding,
    float phaseBase
)
{
    __requires(DY == 2048);
    __requires(DX == 1024);
    __requires(threadCount == 1048576);
    __requires(padding == 16);

    const uint threadId = blockIdx.x *  blockDim.x + threadIdx.x;

    if (threadId >= threadCount)
    {
        return;
    }

    uint x, y, i = threadId;
    udivmod(i, DX / 2, x);
    udivmod(i, DY, y);

    //Avoid overwrites in columns 0 and DX / 2 by different threads (lower and upper halves)
    if ((x == 0) && (y > DY / 2))
    {
        return;
    }

    const uint srcOffset = i * DY * (DX + padding);
    const uint dstOffset = i * DY * DX;

    //Process x = [0 .. DX / 2 - 1] U [DX / 2 + 1 .. DX]
    {
        const uint  loadPos1 = srcOffset +          y * (DX + padding) +        x;
        const uint  loadPos2 = srcOffset + mod(y, DY) * (DX + padding) + (DX - x);
        const uint storePos1 = dstOffset +          y * DX +          x;
        const uint storePos2 = dstOffset + mod(y, DY) * DX + mod(x, DX);

        fComplex D1 = LOAD_FCOMPLEX(loadPos1);
        fComplex D2 = LOAD_FCOMPLEX(loadPos2);

        fComplex twiddle;
        getTwiddle(twiddle, phaseBase * (float)x);
        spPreprocessC2C(D1, D2, twiddle);

        d_Dst[storePos1] = D1;
        d_Dst[storePos2] = D2;
    }

    //Process x = DX / 2
    if (x == 0)
    {
        const uint  loadPos1 = srcOffset +          y * (DX + padding) + DX / 2;
        const uint  loadPos2 = srcOffset + mod(y, DY) * (DX + padding) + DX / 2;
        const uint storePos1 = dstOffset +          y * DX + DX / 2;
        const uint storePos2 = dstOffset + mod(y, DY) * DX + DX / 2;

        fComplex D1 = LOAD_FCOMPLEX(loadPos1);
        fComplex D2 = LOAD_FCOMPLEX(loadPos2);

        //twiddle = getTwiddle(phaseBase * (DX / 2)) = exp(-dir * j * PI / 2)
        fComplex twiddle = {0, (phaseBase > 0) ? 1.0f : -1.0f};
        spPreprocessC2C(D1, D2, twiddle);

        d_Dst[storePos1] = D1;
        d_Dst[storePos2] = D2;
    }
}
