#include "hip/hip_runtime.h"
//pass
//--gridDim=4096             --blockDim=256

#include "common.h"

__global__ void spPostprocess2D_kernel(
    fComplex *d_Dst,
    fComplex *d_Src,
    uint DY,
    uint DX,
    uint threadCount,
    uint padding,
    float phaseBase
)
{
    __requires(DY == 2048);
    __requires(DX == 1024);
    __requires(threadCount == 1048576);
    __requires(padding == 16);

    const uint threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId >= threadCount)
    {
        return;
    }

    uint x, y, i = threadId;
    udivmod(i, DX / 2, x);
    udivmod(i, DY, y);

#ifndef KERNEL_BUG
    //Avoid overwrites in columns DX / 2 by different threads
    if ((x == 0) && (y > DY / 2))
    {
        return;
    }
#endif

    const uint srcOffset = i * DY * DX;
    const uint dstOffset = i * DY * (DX + padding);

    //Process x = [0 .. DX / 2 - 1] U [DX / 2 + 1 .. DX]
    {
        const uint  loadPos1 = srcOffset +          y * DX +          x;
        const uint  loadPos2 = srcOffset + mod(y, DY) * DX + mod(x, DX);
        const uint storePos1 = dstOffset +          y * (DX + padding) +        x;
        const uint storePos2 = dstOffset + mod(y, DY) * (DX + padding) + (DX - x);

        fComplex D1 = LOAD_FCOMPLEX(loadPos1);
        fComplex D2 = LOAD_FCOMPLEX(loadPos2);

        fComplex twiddle;
        getTwiddle(twiddle, phaseBase * (float)x);
        spPostprocessC2C(D1, D2, twiddle);

        d_Dst[storePos1] = D1;
        d_Dst[storePos2] = D2;
    }

    //Process x = DX / 2
    if (x == 0)
    {
        const uint  loadPos1 = srcOffset +          y * DX + DX / 2;
        const uint  loadPos2 = srcOffset + mod(y, DY) * DX + DX / 2;
        const uint storePos1 = dstOffset +          y * (DX + padding) + DX / 2;
        const uint storePos2 = dstOffset + mod(y, DY) * (DX + padding) + DX / 2;

        fComplex D1 = LOAD_FCOMPLEX(loadPos1);
        fComplex D2 = LOAD_FCOMPLEX(loadPos2);

        //twiddle = getTwiddle(phaseBase * (DX / 2)) = exp(dir * j * PI / 2)
        fComplex twiddle = {0, (phaseBase > 0) ? 1.0f : -1.0f};
        spPostprocessC2C(D1, D2, twiddle);

        d_Dst[storePos1] = D1;
        d_Dst[storePos2] = D2;
    }
}
