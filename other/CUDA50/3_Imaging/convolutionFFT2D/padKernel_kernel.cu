#include "hip/hip_runtime.h"
//pass
//--gridDim=[1,1,1]        --blockDim=[32,8,1]

#include "common.h"

__global__ void padKernel_kernel(
    float *d_Dst,
    float *d_Src,
    int fftH,
    int fftW,
    int kernelH,
    int kernelW,
    int kernelY,
    int kernelX
)
{
    __requires(fftH == 2048);
    __requires(fftW == 2048);

    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    const int x = blockDim.x * blockIdx.x + threadIdx.x;

    if (y < kernelH && x < kernelW)
    {
        int ky = y - kernelY;

        if (ky < 0)
        {
            ky += fftH;
        }

        int kx = x - kernelX;

        if (kx < 0)
        {
            kx += fftW;
        }

        d_Dst[ky * fftW + kx] = LOAD_FLOAT(y * kernelW + x);
    }
}
