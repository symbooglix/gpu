#include "hip/hip_runtime.h"
//pass
//--gridDim=[32,32,1] --blockDim=[16,16,1]

#include "common.h"

__global__ void
d_renderFastBicubic(uchar4 *d_output, uint width, uint height, float tx, float ty, float scale, float cx, float cy)
{
    __requires(width == 512);
    __requires(height == 512);

    uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    uint i = __umul24(y, width) + x;

    float u = (x-cx)*scale+cx + tx;
    float v = (y-cy)*scale+cy + ty;

    if ((x < width) && (y < height))
    {
        // write output color
        float c = tex2DFastBicubic<uchar, float>(tex, u, v);
        d_output[i] = make_uchar4(c * 0xff, c * 0xff, c * 0xff, 0);
    }
}
