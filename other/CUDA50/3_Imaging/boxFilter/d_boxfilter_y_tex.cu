#include "hip/hip_runtime.h"
//pass
//--gridDim=16 --blockDim=64

#include "common.h"
  
__global__ void
d_boxfilter_y_tex(float *od, int w, int h, int r)
{
    __requires(w == 1024);
    __requires(h == 1024);

    float scale = 1.0f / (float)((r << 1) + 1);
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

    float t = 0.0f;

    for (int y = -r; y <= r; y++)
    {
        t += tex2D(tex, x, y);
    }

    od[x] = t * scale;

    for (int y = 1; y < h; y++)
    {
        t += tex2D(tex, x, y + r);
        t -= tex2D(tex, x, y - r - 1);
        od[y * w + x] = t * scale;
    }
}
