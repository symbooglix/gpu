#include "hip/hip_runtime.h"
//pass
//--gridDim=16 --blockDim=64

#include "common.h"
  
__global__ void
d_boxfilter_y_global(float *id, float *od, int w, int h, int r)
{
    __requires(w == 1024);
    __requires(h == 1024);
    __requires(r ==   14);

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    d_boxfilter_y(&id[x], &od[x], w, h, r);
}
