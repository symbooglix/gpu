#include "hip/hip_runtime.h"
//pass
//--gridDim=16 --blockDim=64

#include "common.h"
  
__global__ void
d_boxfilter_x_tex(float *od, int w, int h, int r)
{
    __requires(w == 1024);
    __requires(h == 1024);

    float scale = 1.0f / (float)((r << 1) + 1);
    unsigned int y = blockIdx.x*blockDim.x + threadIdx.x;

    float t = 0.0f;

    for (int x =- r; x <= r; x++)
    {
        t += tex2D(tex, x, y);
    }

    od[y * w] = t * scale;

    for (int x = 1;
         x < w; x++)
    {
        t += tex2D(tex, x + r, y);
        t -= tex2D(tex, x - r - 1, y);
        od[y * w + x] = t * scale;
    }
}
