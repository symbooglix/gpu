#include "hip/hip_runtime.h"
//pass
//--gridDim=16 --blockDim=64

#include "common.h"

__global__ void
d_boxfilter_rgba_y(unsigned int *id, unsigned int *od, int w, int h, int r)
{
    __requires(w == 1024);
    __requires(h == 1024);

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    id = &id[x];
    od = &od[x];

    float scale = 1.0f / (float)((r << 1) + 1);

    float4 t;
    // do left edge
    t = rgbaIntToFloat(id[0]) * r;

    for (int y = 0; y < (r + 1); y++)
    {
        t += rgbaIntToFloat(id[y*w]);
    }

    od[0] = rgbaFloatToInt(t * scale);

    for (int y = 1; y < (r + 1); y++)
    {
        t += rgbaIntToFloat(id[(y + r) * w]);
        t -= rgbaIntToFloat(id[0]);
        od[y * w] = rgbaFloatToInt(t * scale);
    }

    // main loop
    for (int y = (r + 1); y < (h - r); y++)
    {
        t += rgbaIntToFloat(id[(y + r) * w]);
        t -= rgbaIntToFloat(id[((y - r) * w) - w]);
        od[y * w] = rgbaFloatToInt(t * scale);
    }

    // do right edge
    for (int y = h - r; y < h; y++)
    {
        t += rgbaIntToFloat(id[(h - 1) * w]);
        t -= rgbaIntToFloat(id[((y - r) * w) - w]);
        od[y * w] = rgbaFloatToInt(t * scale);
    }
}
