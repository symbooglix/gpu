#include "hip/hip_runtime.h"
//pass
//--gridDim=16 --blockDim=64

#include "common.h"
  
__global__ void
d_boxfilter_x_global(float *id, float *od, int w, int h, int r)
{
    __requires(w == 1024);
    __requires(h == 1024);
    __requires(r ==   14);

    unsigned int y = blockIdx.x*blockDim.x + threadIdx.x;
    d_boxfilter_x(&id[y * w], &od[y * w], w, h, r);
}
