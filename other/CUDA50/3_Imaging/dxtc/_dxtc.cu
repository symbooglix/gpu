#include "hip/hip_runtime.h"
//pass
//--gridDim=6400  --blockDim=64 --warp-sync=32

typedef unsigned int uint;
typedef unsigned short ushort;
#define FLT_MAX 0x1.fffffep127f

                   __device__ static __attribute__((always_inline)) void colorSums(const float3 *colors, float3 *sums);
                   __device__ static __attribute__((always_inline)) float3 firstEigenVector(float matrix[6]);
                   __device__ static __attribute__((always_inline)) float3 bestFitLine(const float3 *colors, float3 color_sum);
template <class T> __device__ static __attribute__((always_inline)) void swap(T &a, T &b);
                   __device__ static __attribute__((always_inline)) void sortColors(const float *values, int *ranks);
                   __device__ static __attribute__((always_inline)) void loadColorBlock(const uint *image, float3 colors[16], float3 sums[16], int xrefs[16], int blockOffset);
                   __device__ static __attribute__((always_inline)) float3 roundAndExpand(float3 v, ushort *w);
                   __device__ static __attribute__((always_inline)) float evalPermutation4(const float3 *colors, uint permutation, ushort *start, ushort *end, float3 color_sum);
                   __device__ static __attribute__((always_inline)) float evalPermutation3(const float3 *colors, uint permutation, ushort *start, ushort *end, float3 color_sum);
                   __device__ static __attribute__((always_inline)) void evalAllPermutations(const float3 *colors, const uint *permutations, ushort &bestStart, ushort &bestEnd, uint &bestPermutation, float *errors, float3 color_sum);
                   __device__ static __attribute__((always_inline)) int findMinError(float *errors);
                   __device__ static __attribute__((always_inline)) void saveBlockDXT1(ushort start, ushort end, uint permutation, int xrefs[16], uint2 *result, int blockOffset);

#define NUM_THREADS 64        // Number of threads per block.

__device__ static __attribute__((always_inline)) void colorSums(const float3 *colors, float3 *sums)
{
    const int idx = threadIdx.x;

    sums[idx] = colors[idx];
    sums[idx] += sums[idx^8];
    sums[idx] += sums[idx^4];
    sums[idx] += sums[idx^2];
    sums[idx] += sums[idx^1];
}

__device__ static __attribute__((always_inline)) float3 firstEigenVector(float matrix[6])
{
    // 8 iterations seems to be more than enough.

    float3 v = make_float3(1.0f, 1.0f, 1.0f);

    for (int i = 0;
         __global_invariant(__implies(threadIdx.x >= 16, !__enabled())),
         i < 8; i++)
    {
        float x = v.x * matrix[0] + v.y * matrix[1] + v.z * matrix[2];
        float y = v.x * matrix[1] + v.y * matrix[3] + v.z * matrix[4];
        float z = v.x * matrix[2] + v.y * matrix[4] + v.z * matrix[5];
        float m = max(max(x, y), z);
        float iv = 1.0f / m;
        v = make_float3(x*iv, y*iv, z*iv);
    }

    return v;
}

__device__ static __attribute__((always_inline)) float3 bestFitLine(const float3 *colors, float3 color_sum)
{
    // Compute covariance matrix of the given colors.
    const int idx = threadIdx.x;

    float3 diff = colors[idx] - color_sum * (1.0f / 16.0f);

    // @@ Eliminate two-way bank conflicts here.
    // @@ It seems that doing that and unrolling the reduction doesn't help...
    __shared__ float covariance[16*6];

    covariance[6 * idx + 0] = diff.x * diff.x;    // 0, 6, 12, 2, 8, 14, 4, 10, 0
    covariance[6 * idx + 1] = diff.x * diff.y;
    covariance[6 * idx + 2] = diff.x * diff.z;
    covariance[6 * idx + 3] = diff.y * diff.y;
    covariance[6 * idx + 4] = diff.y * diff.z;
    covariance[6 * idx + 5] = diff.z * diff.z;

    for (int d = 8;
         __global_invariant(__implies(idx >= 16, !__enabled())),
         __global_invariant(__implies(idx >= 16, !__write(covariance))),
         __global_invariant(__implies(idx >= 16, !__read(covariance))),
         __global_invariant(__implies(idx/32 == __other_int(idx)/32 & blockIdx.x == __other_int(blockIdx.x), !__write(covariance))),
         d > 0; d >>= 1)
    {
        if (idx < d)
        {
            covariance[6 * idx + 0] += covariance[6 * (idx+d) + 0];
            covariance[6 * idx + 1] += covariance[6 * (idx+d) + 1];
            covariance[6 * idx + 2] += covariance[6 * (idx+d) + 2];
            covariance[6 * idx + 3] += covariance[6 * (idx+d) + 3];
            covariance[6 * idx + 4] += covariance[6 * (idx+d) + 4];
            covariance[6 * idx + 5] += covariance[6 * (idx+d) + 5];
        }
    }

    // Compute first eigen vector.
    return firstEigenVector(covariance);
}

template <class T>
__device__ static __attribute__((always_inline)) void swap(T &a, T &b)
{
    T tmp = a;
    a = b;
    b = tmp;
}

//__constant__ float3 kColorMetric = { 0.2126f, 0.7152f, 0.0722f };
__constant__ float3 kColorMetric = { 1.0f, 1.0f, 1.0f };

__device__ static __attribute__((always_inline)) void sortColors(const float *values, int *ranks)
{
    const int tid = threadIdx.x;

    int rank = 0;

#pragma unroll

    for (int i = 0;
         __global_invariant(__implies(tid >= 16, !__enabled())),
         i < 16; i++)
    {
        rank += (values[i] < values[tid]);
    }

    ranks[tid] = rank;

    // Resolve elements with the same index.
#pragma unroll

    for (int i = 0;
         __global_invariant(__implies(tid >= 16, !__enabled())),
         __global_invariant(__implies(tid >= 16, !__read(ranks))),
         __global_invariant(__implies(tid/32 == __other_int(tid)/32 & blockIdx.x == __other_int(blockIdx.x), !__write(ranks))),
         i < 15; i++)
    {
        if (tid > i && ranks[tid] == ranks[i])
        {
            ++ranks[tid];
        }
    }

    // IMPERIAL EDIT: post condition of the above code
    __assume(__implies(tid < 16 & __other_int(tid) < 16 & blockIdx.x == __other_int(blockIdx.x), ranks[tid] != ranks[__other_int(tid)]));
}

__device__ static __attribute__((always_inline)) void loadColorBlock(const uint *image, float3 colors[16], float3 sums[16], int xrefs[16], int blockOffset)
{
    const int bid = blockIdx.x + blockOffset;
    const int idx = threadIdx.x;

    __shared__ float dps[16];

    float3 tmp;

    if (idx < 16)
    {
        // Read color and copy to shared mem.
        uint c = image[(bid) * 16 + idx];

        colors[idx].x = ((c >> 0) & 0xFF) * (1.0f / 255.0f);
        colors[idx].y = ((c >> 8) & 0xFF) * (1.0f / 255.0f);
        colors[idx].z = ((c >> 16) & 0xFF) * (1.0f / 255.0f);

        // Sort colors along the best fit line.
        colorSums(colors, sums);
        float3 axis = bestFitLine(colors, sums[0]);

        dps[idx] = dot(colors[idx], axis);

        sortColors(dps, xrefs);

        tmp = colors[idx];

        colors[xrefs[idx]] = tmp;
    }
}

__device__ static __attribute__((always_inline)) float3 roundAndExpand(float3 v, ushort *w)
{
    v.x = rintf(__saturatef(v.x) * 31.0f);
    v.y = rintf(__saturatef(v.y) * 63.0f);
    v.z = rintf(__saturatef(v.z) * 31.0f);

    *w = ((ushort)v.x << 11) | ((ushort)v.y << 5) | (ushort)v.z;
    v.x *= 0.03227752766457f; // approximate integer bit expansion.
    v.y *= 0.01583151765563f;
    v.z *= 0.03227752766457f;
    return v;
}

__constant__ float alphaTable4[4] = { 9.0f, 0.0f, 6.0f, 3.0f };
__constant__ float alphaTable3[4] = { 4.0f, 0.0f, 2.0f, 2.0f };
__constant__ const int prods4[4] = { 0x090000,0x000900,0x040102,0x010402 };
__constant__ const int prods3[4] = { 0x040000,0x000400,0x040101,0x010401 };

#define USE_TABLES 1

__device__ static __attribute__((always_inline)) float evalPermutation4(const float3 *colors, uint permutation, ushort *start, ushort *end, float3 color_sum)
{
    // Compute endpoints using least squares.
#if USE_TABLES
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

    int akku = 0;

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        alphax_sum += alphaTable4[bits & 3] * colors[i];
        akku += prods4[bits & 3];
    }

    float alpha2_sum = float(akku >> 16);
    float beta2_sum = float((akku >> 8) & 0xff);
    float alphabeta_sum = float((akku >> 0) & 0xff);
    float3 betax_sum = (9.0f * color_sum) - alphax_sum;
#else
    float alpha2_sum = 0.0f;
    float beta2_sum = 0.0f;
    float alphabeta_sum = 0.0f;
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        float beta = (bits & 1);

        if (bits & 2)
        {
            beta = (1 + beta) * (1.0f / 3.0f);
        }

        float alpha = 1.0f - beta;

        alpha2_sum += alpha * alpha;
        beta2_sum += beta * beta;
        alphabeta_sum += alpha * beta;
        alphax_sum += alpha * colors[i];
    }

    float3 betax_sum = color_sum - alphax_sum;
#endif

    // alpha2, beta2, alphabeta and factor could be precomputed for each permutation, but it's faster to recompute them.
    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;

    // Round a, b to the closest 5-6-5 color and expand...
    a = roundAndExpand(a, start);
    b = roundAndExpand(b, end);

    // compute the error
    float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    return (0.111111111111f) * dot(e, kColorMetric);
}

__device__ static __attribute__((always_inline)) float evalPermutation3(const float3 *colors, uint permutation, ushort *start, ushort *end, float3 color_sum)
{
    // Compute endpoints using least squares.
#if USE_TABLES
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

    int akku = 0;

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        alphax_sum += alphaTable3[bits & 3] * colors[i];
        akku += prods3[bits & 3];
    }

    float alpha2_sum = float(akku >> 16);
    float beta2_sum = float((akku >> 8) & 0xff);
    float alphabeta_sum = float((akku >> 0) & 0xff);
    float3 betax_sum = (4.0f * color_sum) - alphax_sum;
#else
    float alpha2_sum = 0.0f;
    float beta2_sum = 0.0f;
    float alphabeta_sum = 0.0f;
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        float beta = (bits & 1);

        if (bits & 2)
        {
            beta = 0.5f;
        }

        float alpha = 1.0f - beta;

        alpha2_sum += alpha * alpha;
        beta2_sum += beta * beta;
        alphabeta_sum += alpha * beta;
        alphax_sum += alpha * colors[i];
    }

    float3 betax_sum = color_sum - alphax_sum;
#endif

    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;

    // Round a, b to the closest 5-6-5 color and expand...
    a = roundAndExpand(a, start);
    b = roundAndExpand(b, end);

    // compute the error
    float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    return (0.25f) * dot(e, kColorMetric);
}

__device__ static __attribute__((always_inline)) void evalAllPermutations(const float3 *colors, const uint *permutations, ushort &bestStart, ushort &bestEnd, uint &bestPermutation, float *errors, float3 color_sum)
{
    const int idx = threadIdx.x;

    float bestError = FLT_MAX;

    __shared__ uint s_permutations[160];

    for (int i = 0; i < 16; i++)
    {
        int pidx = idx + NUM_THREADS * i;

        if (pidx >= 992)
        {
            break;
        }

        ushort start, end;
        uint permutation = permutations[pidx];

        if (pidx < 160)
        {
            s_permutations[pidx] = permutation;
        }

        float error = evalPermutation4(colors, permutation, &start, &end, color_sum);

        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;
        }
    }

    if (bestStart < bestEnd)
    {
        swap(bestEnd, bestStart);
        bestPermutation ^= 0x55555555;    // Flip indices.
    }

    for (int i = 0; i < 3; i++)
    {
        int pidx = idx + NUM_THREADS * i;

        if (pidx >= 160)
        {
            break;
        }

        ushort start, end;
        uint permutation = s_permutations[pidx];
        float error = evalPermutation3(colors, permutation, &start, &end, color_sum);

        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;

            if (bestStart > bestEnd)
            {
                swap(bestEnd, bestStart);
                bestPermutation ^= (~bestPermutation >> 1) & 0x55555555;    // Flip indices.
            }
        }
    }

    errors[idx] = bestError;
}

__device__ static __attribute__((always_inline)) int findMinError(float *errors)
{
    const int idx = threadIdx.x;

    __shared__ int indices[NUM_THREADS];
    indices[idx] = idx;

    for (int d = NUM_THREADS/2; d > 32; d >>= 1)
    {
        __syncthreads();

        if (idx < d)
        {
            float err0 = errors[idx];
            float err1 = errors[idx + d];

            if (err1 < err0)
            {
                errors[idx] = err1;
                indices[idx] = indices[idx + d];
            }
        }
    }

    __syncthreads();

    // unroll last 6 iterations
    if (idx < 32)
    {
        if (errors[idx + 32] < errors[idx])
        {
            errors[idx] = errors[idx + 32];
            indices[idx] = indices[idx + 32];
        }

        if (errors[idx + 16] < errors[idx])
        {
            errors[idx] = errors[idx + 16];
            indices[idx] = indices[idx + 16];
        }

        if (errors[idx + 8] < errors[idx])
        {
            errors[idx] = errors[idx + 8];
            indices[idx] = indices[idx + 8];
        }

        if (errors[idx + 4] < errors[idx])
        {
            errors[idx] = errors[idx + 4];
            indices[idx] = indices[idx + 4];
        }

        if (errors[idx + 2] < errors[idx])
        {
            errors[idx] = errors[idx + 2];
            indices[idx] = indices[idx + 2];
        }

        if (errors[idx + 1] < errors[idx])
        {
            errors[idx] = errors[idx + 1];
            indices[idx] = indices[idx + 1];
        }
    }

    __syncthreads();

    return indices[0];
}

__device__ static __attribute__((always_inline)) void saveBlockDXT1(ushort start, ushort end, uint permutation, int xrefs[16], uint2 *result, int blockOffset)
{
    const int bid = blockIdx.x + blockOffset;

    if (start == end)
    {
        permutation = 0;
    }

    // Reorder permutation.
    uint indices = 0;

    for (int i = 0; i < 16; i++)
    {
        int ref = xrefs[i];
        indices |= ((permutation >> (2 * ref)) & 3) << (2 * i);
    }

    // Write endpoints.
    result[bid].x = (end << 16) | start;

    // Write palette indices.
    result[bid].y = indices;
}

__global__ void compress(const uint *permutations, const uint *image, uint2 *result, int blockOffset)
{
    const int idx = threadIdx.x;

    __shared__ float3 colors[16];
    __shared__ float3 sums[16];
    __shared__ int xrefs[16];

    loadColorBlock(image, colors, sums, xrefs, blockOffset);

    __syncthreads();

    ushort bestStart, bestEnd;
    uint bestPermutation;

    __shared__ float errors[NUM_THREADS];

    evalAllPermutations(colors, permutations, bestStart, bestEnd, bestPermutation, errors, sums[0]);

    // Use a parallel reduction to find minimum error.
    const int minIdx = findMinError(errors);

    __syncthreads();

    // Only write the result of the winner thread.
    if (idx == minIdx)
    {
        saveBlockDXT1(bestStart, bestEnd, bestPermutation, xrefs, result, blockOffset);
    }
}
