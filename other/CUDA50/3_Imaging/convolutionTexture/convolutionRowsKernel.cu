#include "hip/hip_runtime.h"
//pass
//--gridDim=[192,128,1]    --blockDim=[16,12,1]

#include "common.h"

__global__ void convolutionRowsKernel(
    float *d_Dst,
    int imageW,
    int imageH
)
{
    __requires(imageW == 3072);
    const   int ix = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
    const   int iy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);
    const float  x = (float)ix + 0.5f;
    const float  y = (float)iy + 0.5f;

    if (ix >= imageW || iy >= imageH)
    {
        return;
    }

    float sum = 0;

#if(UNROLL_INNER)
    sum = convolutionRow<2 *KERNEL_RADIUS>(x, y);
#else

    for (int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++)
    {
        sum += tex2D(texSrc, x + (float)k, y) * c_Kernel[KERNEL_RADIUS - k];
    }

#endif

    d_Dst[IMAD(iy, imageW, ix)] = sum;
}
