
#include <hip/hip_runtime.h>
//pass
//--gridDim=[4,1,1]        --blockDim=[512,1,1]

__global__ void
initValue(float *od, float value)
{
    // position of write into global memory
    unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    od[index] = value;

    // sync after each decomposition step
    __syncthreads();
}
