#include "hip/hip_runtime.h"
//pass
//--gridDim=[32,32,1] --blockDim=[16,16,1]

#include "common.h"
  
__global__ void
d_render_regular(uint *d_output, uint imageW, uint imageH,
                 float density, float brightness,
                 float transferOffset, float transferScale, float transferWeight = 0.0f)
{
    __requires(imageW == 32*16 /*gridDim.x*blockDim.x*/);

    d_render<TF_SINGLE_1D>(d_output,imageW,imageH,density,brightness,transferOffset,transferScale,transferWeight);
}
