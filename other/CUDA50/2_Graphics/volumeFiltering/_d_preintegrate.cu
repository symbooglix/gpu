#include "hip/hip_runtime.h"
//pass
//--gridDim=[64,64,1]      --blockDim=[16,16,1]

//REQUIRES: hipExtent
//REQUIRES: SURFACE

#include "common.h"

__global__ void
d_preintegrate(int layer, float steps, hipExtent extent)
{
    uint x = blockIdx.x*blockDim.x + threadIdx.x;
    uint y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x >= extent.width || y >= extent.height)
    {
        return;
    }

    float sx = float(x)/float(extent.width);
    float sy = float(y)/float(extent.height);

    float smax = max(sx,sy);
    float smin = min(sx,sy);

    float4 iv;

    if (x != y)
    {
        // assumes square textures!
        float fracc = smax - smin;
        fracc = 1.0 /(fracc*steps);

        float4 intmax = tex1D(transferIntegrateTex,smax);
        float4 intmin = tex1D(transferIntegrateTex,smin);
        iv.x = (intmax.x - intmin.x)*fracc;
        iv.y = (intmax.y - intmin.y)*fracc;
        iv.z = (intmax.z - intmin.z)*fracc;
        //iv.w = (intmax.w - intmin.w)*fracc;
        iv.w   = (1.0 - exp(-(intmax.w - intmin.w) * fracc));
    }
    else
    {
        float4 sample = tex1D(transferTex,smin);
        iv.x = sample.x;
        iv.y = sample.y;
        iv.z = sample.z;
        //iv.w = sample.w;
        iv.w   = (1.0 - exp(-sample.w));
    }

    iv.x =  __saturatef(iv.x);
    iv.y =  __saturatef(iv.y);
    iv.z =  __saturatef(iv.z);
    iv.w =  __saturatef(iv.w);

    // surface writes need byte offsets for x!
    surf2DLayeredwrite(iv,transferLayerPreintSurf, x * sizeof(float4), y, layer);
}
