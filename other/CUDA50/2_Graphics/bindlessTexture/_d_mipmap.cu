#include "hip/hip_runtime.h"
//pass
//--gridDim=[19,15] --blockDim=[16,16]

//REQUIRES:SURFACE

typedef unsigned int uchar;
typedef unsigned int uint;

__device__ float4 fminf(float4, float4);

__device__ static __attribute__((always_inline)) uchar4 to_uchar4(float4 vec)
{
    return make_uchar4((uchar)vec.x, (uchar)vec.y, (uchar)vec.z, (uchar)vec.w);
}

__global__ void
d_mipmap(hipSurfaceObject_t mipOutput, hipTextureObject_t mipInput, uint imageW, uint imageH)
{
    __requires(imageW == 16*32 /*blockDim.x * gridDim.x*/);

    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    float px = 1.0/float(imageW);
    float py = 1.0/float(imageH);

    if ((x < imageW) && (y < imageH))
    {
        // take the average of 4 samples

        // we are using the normalized access to make sure non-power-of-two textures
        // behave well when downsized.
        float4 color = 
            (tex2D<float4>(mipInput,(x + 0) * px, (y + 0) * py)) +
            (tex2D<float4>(mipInput,(x + 1) * px, (y + 0) * py)) +
            (tex2D<float4>(mipInput,(x + 1) * px, (y + 1) * py)) +
            (tex2D<float4>(mipInput,(x + 0) * px, (y + 1) * py));

        color /= 4.0;
        color *= 255.0;
        color = fminf(color,make_float4(255.0));

        surf2Dwrite(to_uchar4(color),mipOutput,x * sizeof(uchar4),y);
    }
}
