#include "hip/hip_runtime.h"
//pass
//--gridDim=[32,32,1]      --blockDim=[16,16,1]

typedef unsigned char uchar;
texture<uchar, 3, hipReadModeNormalizedFloat> tex;  // 3D texture

typedef unsigned int uint;
#define __umul24(x,y) (x*y)

__global__ void
d_render(uint *d_output, uint imageW, uint imageH, float w)
{
    __requires(imageW == 32*16 /*gridDim.x*blockDim.x*/);

    uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;

    float u = x / (float) imageW;
    float v = y / (float) imageH;
    // read from 3D texture
    float voxel = tex3D(tex, u, v, w);

    if ((x < imageW) && (y < imageH))
    {
        // write output color
        uint i = __umul24(y, imageW) + x;
        d_output[i] = voxel*255;
    }
}
