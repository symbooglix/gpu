
#include <hip/hip_runtime.h>
//pass
//--gridDim=40 --blockDim=256

typedef unsigned char Bool;
typedef unsigned int uint;

__global__ void computeVisibilities_kernel(const float *angles,
                                           const float *scannedAngles,
                                           int numAngles,
                                           Bool *visibilities)
{
    uint i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numAngles)
    {
        visibilities[i] = scannedAngles[i] <= angles[i];
    }
}
