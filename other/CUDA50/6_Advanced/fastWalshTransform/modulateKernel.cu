
#include <hip/hip_runtime.h>
//pass
//--gridDim=128              --blockDim=256

__global__ void modulateKernel(float *d_A, float *d_B, int N)
{
    int        tid = blockIdx.x * blockDim.x + threadIdx.x;
    int numThreads = blockDim.x * gridDim.x;
    float     rcpN = 1.0f / (float)N;

    for (int pos = tid; pos < N; pos += numThreads)
    {
        d_A[pos] *= d_B[pos] * rcpN;
    }
}
