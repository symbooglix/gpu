#include "hip/hip_runtime.h"
//pass
//--gridDim=[11377,1,1]    --blockDim=[256,1,1]

#include "common.h"

__global__ void makeNewEdges(const uint *survivedEdgesIDs,
                             const uint *verticesMapping,
                             const uint *edges,
                             const float *weights,
                             uint *newEdges,
                             float *newWeights,
                             uint edgesCount)
{
    uint tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < edgesCount)
    {
        uint edgeID = survivedEdgesIDs[tid];
        uint oldEdge = edges[edgeID];

        newEdges[tid] = verticesMapping[oldEdge];
        newWeights[tid] = weights[edgeID];
    }
}
