#include "hip/hip_runtime.h"
//pass
//--gridDim=[4800,1,1]     --blockDim=[256,1,1]

#include "common.h"

__global__ void markSegments(const uint *verticesOffsets,
                             uint *flags,
                             uint verticesCount)
{
    uint tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < verticesCount)
    {
        flags[verticesOffsets[tid]] = 1;
    }
}
