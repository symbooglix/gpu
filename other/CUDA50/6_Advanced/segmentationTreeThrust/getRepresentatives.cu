#include "hip/hip_runtime.h"
//pass
//--gridDim=[1322,1,1]     --blockDim=[256,1,1]

#include "common.h"

__global__ void getRepresentatives(const uint *successors,
                                   uint *representatives,
                                   uint verticesCount)
{
    uint tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < verticesCount)
    {
        uint successor = successors[tid];
        uint nextSuccessor = successors[successor];

        while (successor != nextSuccessor)
        {
            successor = nextSuccessor;
            nextSuccessor = successors[nextSuccessor];
        }

        representatives[tid] = successor;
    }
}
