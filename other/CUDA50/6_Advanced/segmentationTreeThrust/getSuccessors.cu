#include "hip/hip_runtime.h"
//pass
//--gridDim=[1322,1,1]     --blockDim=[256,1,1]

#include "common.h"

__global__ void getSuccessors(const uint *verticesOffsets,
                              const uint *minScannedEdges,
                              uint *successors,
                              uint verticesCount,
                              uint edgesCount)
{
    uint tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < verticesCount)
    {
        uint successorPos = (tid < verticesCount - 1) ?
                            (verticesOffsets[tid + 1] - 1) :
                            (edgesCount - 1);

        successors[tid] = minScannedEdges[successorPos];
    }
}
