#include "hip/hip_runtime.h"
//pass
//--gridDim=[1322,1,1]     --blockDim=[256,1,1]

#include "common.h"

__global__ void getVerticesMapping(const uint *clusteredVerticesIDs,
                                   const uint *newVerticesIDs,
                                   uint *verticesMapping,
                                   uint verticesCount)
{
    __requires(
     clusteredVerticesIDs[blockIdx.x * blockDim.x + threadIdx.x] !=
     clusteredVerticesIDs[__other_int(blockIdx.x * blockDim.x + threadIdx.x)]);

    uint tid = blockIdx.x * blockDim.x + threadIdx.x;
    __requires(clusteredVerticesIDs[tid] != clusteredVerticesIDs[__other_int(tid)]);

    if (tid < verticesCount)
    {
        uint vertexID = clusteredVerticesIDs[tid];
        verticesMapping[vertexID] = newVerticesIDs[tid];
    }
}
