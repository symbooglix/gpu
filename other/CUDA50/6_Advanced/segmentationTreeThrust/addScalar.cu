#include "hip/hip_runtime.h"
//pass
//--gridDim=[11377,1,1]    --blockDim=[256,1,1]

#include "common.h"

__global__ void addScalar(uint *array, int scalar, uint size)
{
    uint tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size)
    {
        array[tid] += scalar;
    }
}
