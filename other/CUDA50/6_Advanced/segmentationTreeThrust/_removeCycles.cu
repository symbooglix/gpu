#include "hip/hip_runtime.h"
//pass
//--gridDim=[1322,1,1]     --blockDim=[256,1,1]

#include "common.h"

__global__ void removeCycles(uint *successors,
                             uint verticesCount)
{
    uint tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < verticesCount)
    {
        uint successor = successors[tid];
        uint nextSuccessor = successors[successor];

        if (tid == nextSuccessor)
        {
            if (tid < successor)
            {
                successors[tid] = tid;
            }
            else
            {
                successors[successor] = successor;
            }
        }
    }
}
