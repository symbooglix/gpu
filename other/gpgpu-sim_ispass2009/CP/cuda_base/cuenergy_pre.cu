#include "hip/hip_runtime.h"
//pass
//--blockDim=[16,8] --gridDim=4

/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

//#include <stdio.h>

#include "cuenergy.h"

#if UNROLLX != 1
# error "UNROLLX must be 1"
#endif

// IMPERIAL EDIT
//__constant__ float4 atominfo[MAXATOMS];
__constant__ float4 atominfo[1];

/* This is a reference version of the kernel.  It is simpler and slower
 * than the optimzed version. */

__global__ void cenergy(int numatoms, float gridspacing, float * energygrid) {
  unsigned int xindex  = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
  unsigned int yindex  = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
  unsigned int outaddr = __umul24(gridDim.x, blockDim.x) * yindex + xindex;

  float coorx = gridspacing * xindex;
  float coory = gridspacing * yindex;

  int atomid;
  float energyval=0.0f;

  /* For each atom, compute and accumulate its contribution to energyval
   * for this thread's grid point */
  for (atomid=0; atomid<numatoms; atomid++) {
    float dx = coorx - atominfo[atomid].x;
    float dy = coory - atominfo[atomid].y;
    float r_1 = 1.0f / sqrtf(dx*dx + dy*dy + atominfo[atomid].z);
    energyval += atominfo[atomid].w * r_1;
  }

  energygrid[outaddr] += energyval;
}
