
#include <hip/hip_runtime.h>
//pass
//--blockDim=96 --gridDim=96

// N-queen for CUDA
//
// Copyright(c) 2008 Ping-Che Chen

#define THREAD_NUM		96

/* --------------------------------------------------------------------------
 * This is a non-recursive version of n-queen backtracking solver for CUDA.
 * It receives multiple initial conditions from a CPU iterator, and count
 * each conditions.
 * --------------------------------------------------------------------------
 */

__global__ void solve_nqueen_cuda_kernel(int n, int mark, unsigned int* total_masks, unsigned int* total_l_masks, unsigned int* total_r_masks, unsigned int* results, int total_conditions)
{
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	const int idx = bid * blockDim.x + tid;

	__shared__ unsigned int mask[THREAD_NUM][10];
	__shared__ unsigned int l_mask[THREAD_NUM][10];
	__shared__ unsigned int r_mask[THREAD_NUM][10];
	__shared__ unsigned int m[THREAD_NUM][10];

	__shared__ unsigned int sum[THREAD_NUM];

	const unsigned int t_mask = (1 << n) - 1;
	int total = 0;
	int i = 0;
	unsigned int index;

	if(idx < total_conditions) {
		mask[tid][i] = total_masks[idx];
		l_mask[tid][i] = total_l_masks[idx];
		r_mask[tid][i] = total_r_masks[idx];
		m[tid][i] = mask[tid][i] | l_mask[tid][i] | r_mask[tid][i];

		while(i >= 0) {
			if((m[tid][i] & t_mask) == t_mask) {
				i--;
			}
			else {
				index = (m[tid][i] + 1) & ~m[tid][i];
				m[tid][i] |= index;
				if((index & t_mask) != 0) {
					if(i + 1 == mark) {
						total++;
						i--;
					}
					else {
						mask[tid][i + 1] = mask[tid][i] | index;
						l_mask[tid][i + 1] = (l_mask[tid][i] | index) << 1;
						r_mask[tid][i + 1] = (r_mask[tid][i] | index) >> 1;
						m[tid][i + 1] = (mask[tid][i + 1] | l_mask[tid][i + 1] | r_mask[tid][i + 1]);
						i++;
					}
				}
				else {
					i --;
				}
			}
		}

		sum[tid] = total;
	}
	else {
		sum[tid] = 0;
	}

	__syncthreads();

	// reduction
	if(tid < 64 && tid + 64 < THREAD_NUM) { sum[tid] += sum[tid + 64]; } __syncthreads();
	if(tid < 32) { sum[tid] += sum[tid + 32]; } __syncthreads();
	if(tid < 16) { sum[tid] += sum[tid + 16]; } __syncthreads();
	if(tid < 8) { sum[tid] += sum[tid + 8]; } __syncthreads();
	if(tid < 4) { sum[tid] += sum[tid + 4]; } __syncthreads();
	if(tid < 2) { sum[tid] += sum[tid + 2]; } __syncthreads();
	if(tid < 1) { sum[tid] += sum[tid + 1]; } __syncthreads();

	if(tid == 0) {
		results[bid] = sum[0];
	}
}
