
#include <hip/hip_runtime.h>
//pass
//--gridDim=[6,10] --blockDim=[13,13]

__constant__ int kernelTemplate[25] = {
        0,  1,  2,  3,  4,
        29, 30, 31, 32, 33,
        58, 59, 60, 61, 62,
        87, 88, 89, 90, 91,
        116,117,118,119,120 };

__global__ void executeFirstLayer(float *Layer1_Neurons_GPU,float *Layer1_Weights_GPU,float *Layer2_Neurons_GPU)
{
	int blockID=blockIdx.x;
	int pixelX=threadIdx.x;
	int pixelY=threadIdx.y;


	int weightBegin=blockID*26;
	int windowX=pixelX*2;
	int windowY=pixelY*2;

	float result=0;

	result+=Layer1_Weights_GPU[weightBegin];

	++weightBegin;

//for(int i=0;i<25;++i)
//{
//	result+=Layer1_Neurons_GPU[(windowY*29+windowX+kernelTemplate[i])+(29*29*blockIdx.y)]*Layer1_Weights_GPU[weightBegin+i];
//}

//result=(1.7159*tanhf(0.66666667*result));

	Layer2_Neurons_GPU[(13*13*blockID+pixelY*13+pixelX)+(13*13*6*blockIdx.y)]=result;

}
