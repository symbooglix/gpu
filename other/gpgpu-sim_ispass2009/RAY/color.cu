#include "hip/hip_runtime.h"
//pass
//--blockDim=32 --gridDim=32

// IMPERIAL EDIT: this kernel was commented out
#include "common.h"

__global__  __device__ void color(uint * result, uint * Obj, float * prof, float3 * A, float3 * u, uint imageW, uint imageH, float pas)
{
	uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	uint id = x + y * imageW;

	if( x < imageW && y < imageH )
	{
		float t(prof[id]);
		if( t > 0.0f  && t < 1000.0f ) {
			Rayon R;
			R.A = A[id];
			R.u = u[id];
			Sphere s(cnode[Obj[id]].s);
			s.C.x += pas;
			float4 f = make_float4(s.R,s.V,s.B,s.A)*(dot(getNormale(R.A+R.u*t,s.C),(-1.0f)*R.u));
			result[id] = rgbaFloatToInt(f);
		}
		else {
			result[id] = 0;
		}
		prof[id] = 100000.0f;
	}
}
