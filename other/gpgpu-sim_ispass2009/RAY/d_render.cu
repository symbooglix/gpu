#include "hip/hip_runtime.h"
//pass
//--blockDim=32 --gridDim=32

// IMPERIAL EDIT: this kernel was commented out
#include "common.h"

__global__ void d_render(uint * d_output, uint imageW, uint imageH, float pas, float df, float tPixel)
{
	uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	uint id = x + y * imageW;

	if( x < imageW && y < imageH )
	{
		//float tPixel = 2.0f/(float)min(imageW,imageH);
		matrice3x4 M(MView);
		Rayon R;
		R.A = make_float3(M.m[0].w,M.m[1].w,M.m[2].w);
		R.u = make_float3(M.m[0])*df
			+ make_float3(M.m[2])*(float(x)-float(imageW)*0.5f)*tPixel
			+ make_float3(M.m[1])*(float(y)-float(imageH)*0.5f)*tPixel;
		R.u = normalize(R.u);
		Sphere s(cnode[1].s), s2(cnode[2].s), st(cnode[2].s);
		float t, t2, tt;
		s.C.x += pas, s2.C.x += pas;
		t = intersectionSphere(R,s.C,s.r);
		t2 = intersectionSphere(R,s2.C,s2.r);
		if( !t ) {
			//myswap(s,s2);
			//swap(t,t2);
         tt = t;
			t = t2;
			t2 = tt;
			st = s;
			s = s2;
			s2 = st;
		}
		else if( t2 && t2 < t ) {
			//myswap(s,s2);
			//swap(t,t2);
         tt = t;
			t = t2;
			t2 = tt;
         st = s;
         s = s2;
         s2 = st;
		}
		float4 f = make_float4(0,1,0,1)*(dot(getNormale(R.A+R.u*t,s.C),(-1.0f)*R.u));
		uint n = rgbaFloatToInt(f);
		//printf("%f\n",d_node[0].s.r);
		if( t > 0.0f )
			d_output[id] = n;
		//else d_output[id] = 0;
	}
	__syncthreads();
}
