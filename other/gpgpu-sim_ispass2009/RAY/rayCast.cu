#include "hip/hip_runtime.h"
//pass
//--blockDim=32 --gridDim=32

// IMPERIAL EDIT: this kernel was commented out
#include "common.h"

__global__ void rayCast (uint * d_output, uint * d_temp, uint imageW, uint imageH, float pas, float df)
//(uint * result, uint * temp, uint imageW, uint imageH, float pas, float df)
{
	uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y);
	uint id = x + y * gridDim.x;
	//float tmp= float(imageW)/float(gridDim.x);
	float t;

	//if( x < gridDim.x && y < gridDim.y )
	if( d_temp[id] == 0 )
	{
		float tPixel = 2.0f/float(imageW);
		matrice3x4 M(MView);
		Rayon R;
		R.A = make_float3(M.m[0].w,M.m[1].w,M.m[2].w);
		R.u = make_float3(M.m[0])*df
			+ make_float3(M.m[2])*(float(x)-float(imageW)*0.5f)*tPixel
			+ make_float3(M.m[1])*(float(y)-float(imageH)*0.5f)*tPixel;
		R.u = normalize(R.u);
		Sphere s(cnode[1].s);
		s.C.x += pas;
		t = intersectionSphere(R,s.C,s.r/(imageW/gridDim.x));

		if( t > 0.0f ) {		
			//float4 f = make_float4(0,1,0,1)*(dot(getNormale(R.A+R.u*t,s.C),(-1.0f)*R.u));
			d_output[id] = rgbaFloatToInt(make_float4(0,1,0,1));
			//printf("%d %d\n",int(x*tmp),int((y*tmp)/2));
		}
		else {
//       float tmp= float(imageW)/gridDim.x;
//       d_temp[int(x*tmp+(y*tmp)*imageW)] = 1;
//       d_temp[int(x*tmp+(tmp*(float(y)+0.5f)*imageW))] = 1;
//       d_temp[int(tmp*(float(x)+0.5f)+(y*tmp)*imageW)] = 1;
//       d_temp[int(tmp*(float(x)+0.5f)+(tmp*(float(y)+0.5f)*imageW))] = 1;
			//if(gridDim.x==16) printf("hep %d %f\n",gridDim.x,t);
		}
	}
	else {
//       float tmp= float(imageW)/gridDim.x;
//       d_temp[int(x*tmp+(y*tmp)*imageW)] = 1;
//       d_temp[int(x*tmp+(tmp*(float(y)+0.5f)*imageW))] = 1;
//       d_temp[int(tmp*(float(x)+0.5f)+(y*tmp)*imageW)] = 1;
//       d_temp[int(tmp*(float(x)+0.5f)+(tmp*(float(y)+0.5f)*imageW))] = 1;
			//if(gridDim.x==16) printf("hep %d %f\n",gridDim.x,t);
	}
	//__syncthreads();
}
