#include "hip/hip_runtime.h"
//pass
//--blockDim=32 --gridDim=32

// IMPERIAL EDIT: this kernel was commented out
#include "common.h"

__global__  __device__ void rayTrace(uint * Obj, float * prof, float3 * A, float3 * u, uint imageW, uint imageH, float pas, float df, uint nObj)
{
	uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	uint id = x + y * imageW;

	if( x < imageW && y < imageH )
	{
		Sphere s(cnode[nObj].s);
		float t;
		s.C.x += pas;
		Rayon R;
		R.A = A[id];
		R.u = u[id];
		t = intersectionSphere(R,s.C,s.r);

		if( t > 0.0f && t < prof[id] ) {
			prof[id] = t;
			Obj[id] = nObj;
		}
	}
}
