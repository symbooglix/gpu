#include "hip/hip_runtime.h"
//pass
//--blockDim=32 --gridDim=1

#include "common.h"

__global__  __device__ void render(uint * result, Node * dnode, uint imageW, uint imageH, float pas, float df)
{
    __requires(imageW == 64);
    __requires(imageH == 64);
	uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	uint tid(__umul24(threadIdx.y, blockDim.x) + threadIdx.x);

	uint id(x + y * imageW);
	float4 pile[5];
	uint Obj, nRec(5), n(0);
	//__shared__ Node node[numObj];
	float prof, tmp;

	//if( tid < numObj ) node[tid] = cnode[tid];

	for( int i(0); i < nRec; ++i )
		pile[i] = make_float4(0.0f,0.0f,0.0f,1.0f);

	if( x < imageW && y < imageH )
	{
		prof = 10000.0f;
		result[id] = 0;
		float tPixel(2.0f/float(min(imageW,imageH)));
		float4 f(make_float4(0.0f,0.0f,0.0f,1.0f));
		matrice3x4 M(MView);
		Rayon R;
		R.A = make_float3(M.m[0].w,M.m[1].w,M.m[2].w);
		R.u = make_float3(M.m[0])*df
			+ make_float3(M.m[2])*(float(x)-float(imageW)*0.5f)*tPixel
			+ make_float3(M.m[1])*(float(y)-float(imageH)*0.5f)*tPixel;
		R.u = normalize(R.u);
#ifdef DEVICE_EMU
//    printf("%d: R.A = %e %e %e\n", threadIdx.x, R.A.x, R.A.y, R.A.z);
//    printf("%d: R.u = %e %e %e\n", threadIdx.x, R.u.x, R.u.y, R.u.z);
#endif
#ifdef DEBUG_RT_CUDA
//    d_debug_float4[threadIdx.x*2+0].x= R.A.x;
//    d_debug_float4[threadIdx.x*2+0].y= R.A.y;
//    d_debug_float4[threadIdx.x*2+0].z= R.A.z;
//    d_debug_float4[threadIdx.x*2+1].x= R.u.x;
//    d_debug_float4[threadIdx.x*2+1].y= R.u.y;
//    d_debug_float4[threadIdx.x*2+1].z= R.u.z;
#endif
		__syncthreads();

		for( int i(0); i < nRec && n == i; i++ ) {

			for( int j(0); j < numObj; j++ ) {
				Node nod(cnode[j]);
				Sphere s(nod.s);
				float t;
				s.C.x += pas;
				if( nod.fg )
					t = intersectionPlan(R,s.C,s.C);
				else
					t = intersectionSphere(R,s.C,s.r);

				if( t > 0.0f && t < prof ) {
					prof = t;
					Obj = j;
				}
			}
#ifdef DEBUG_RT_CUDA
         //d_debug_float4[threadIdx.x*5+i].x= prof;
#endif
#ifdef DEVICE_EMU
//       printf("%d: i=%d, t=%e\n", threadIdx.x, i, prof);
#endif
			float t = prof;
			if( t > 0.0f && t < 10000.0f ) {
				n++;
				Node nod(cnode[Obj]);
				Sphere s(nod.s);
				s.C.x += pas;
				float4 color(make_float4(s.R,s.V,s.B,s.A));
				float3 P(R.A+R.u*t), L(normalize(make_float3(10.0f,10.0f,10.0f)-P)), V(normalize(R.A-P));
				float3 N(nod.fg?getNormaleP(P):getNormale(P,s.C));
				float3 Np(dot(V,N)<0.0f?(-1*N):N);
				pile[i] = 0.05f * color;
            #ifdef DEVICE_EMU
//          printf("%d: i=%d, pile[i] = %e %e %e %e\n", threadIdx.x, i, pile[i].x, pile[i].y, pile[i].z, pile[i].w);
//          printf("%d: i=%d, color = %e %e %e %e\n", threadIdx.x, i, color.x, color.y, color.z, color.w);
//          printf("%d: i=%d, P = %e %e %e\n", threadIdx.x, i, P.x, P.y, P.z);
//          printf("%d: i=%d, L = %e %e %e\n", threadIdx.x, i, L.x, L.y, L.z);
//          printf("%d: i=%d, V = %e %e %e\n", threadIdx.x, i, V.x, V.y, V.z);
//          printf("%d: i=%d, N = %e %e %e\n", threadIdx.x, i, N.x, N.y, N.z);
//          printf("%d: i=%d, Np = %e %e %e\n", threadIdx.x, i, Np.x, Np.y, Np.z);
//          printf("%d: i=%d, dot(Np,L) = %e\n", threadIdx.x, i, dot(Np,L));
            //printf("%d: i=%d, notShadowRay(cnode,P,L,pas) = %d\n", threadIdx.x, i, (int) notShadowRay(cnode,P,L,pas));

            #endif
            #ifdef DEBUG_RT_CUDA
            //d_debug_float4[threadIdx.x*16+i*3+0]= pile[i];
//          d_debug_float4[threadIdx.x*16+i*8+0]= color;
//          d_debug_float4[threadIdx.x*16+i*8+1].x= P.x;d_debug_float4[threadIdx.x*16+i*8+1].y= P.y;d_debug_float4[threadIdx.x*16+i*8+1].z= P.z;
//          d_debug_float4[threadIdx.x*16+i*8+2].x= L.x;d_debug_float4[threadIdx.x*16+i*8+2].y= L.y;d_debug_float4[threadIdx.x*16+i*8+2].z= L.z;
//          d_debug_float4[threadIdx.x*16+i*8+3].x= V.x;d_debug_float4[threadIdx.x*16+i*8+3].y= V.y;d_debug_float4[threadIdx.x*16+i*8+3].z= V.z;
//          d_debug_float4[threadIdx.x*16+i*8+4].x= N.x;d_debug_float4[threadIdx.x*16+i*8+4].y= N.y;d_debug_float4[threadIdx.x*16+i*8+4].z= N.z;
//          d_debug_float4[threadIdx.x*16+i*8+5].x= Np.x;d_debug_float4[threadIdx.x*16+i*8+5].y= Np.y;d_debug_float4[threadIdx.x*16+i*8+5].z= Np.z;
//          d_debug_float4[threadIdx.x*16+i*8+6].x= dot(Np,L);
            //d_debug_float4[threadIdx.x*16+i*8+7].x= (float) notShadowRay(cnode,P,L,pas);
            #endif
            #ifdef DEBUG_RT_CUDA
            if( dot(Np,L) > 0.0f && notShadowRay(d_debug_float4, d_debug_uint, i, cnode,P,L,pas) ) {
            #else
            if( dot(Np,L) > 0.0f && notShadowRay(cnode,P,L,pas) ) {
            #endif
               //float3 Ri(2.0f*Np*dot(Np,L) - L);
					float3 Ri(normalize(L+V));
					//Ri = (L+V)/normalize(L+V);
					pile[i] += 0.3f * color* (min(1.0f,dot(Np,L)));
               #ifdef DEVICE_EMU
//             printf("%d: i=%d, pile[i] = %e %e %e %e\n", threadIdx.x, i, pile[i].x, pile[i].y, pile[i].z, pile[i].w);
               #endif
               #ifdef DEBUG_RT_CUDA
               //d_debug_float4[threadIdx.x*16+i*3+1]= pile[i];
               #endif
               #ifdef FIXED_CONST_PARSE
					tmp = 0.8f * pow(max(0.0f,min(1.0f,dot(Np,Ri))),50.0f);
               #else
               tmp = 0.8f * float2int_pow50(max(0.0f,min(1.0f,dot(Np,Ri))));
               #endif
					pile[i].x += tmp;
					pile[i].y += tmp;
					pile[i].z += tmp;
               #ifdef DEVICE_EMU
//             printf("%d: i=%d, pile[i] = %e %e %e %e\n", threadIdx.x, i, pile[i].x, pile[i].y, pile[i].z, pile[i].w);
               #endif
               #ifdef DEBUG_RT_CUDA
               //d_debug_float4[threadIdx.x*16+i*3+2]= pile[i];
               #endif
				}

				R.u = 2.0f*N*dot(N,V) - V;
				R.u = normalize(R.u);
				R.A = P+R.u*0.0001f;
			}
			prof = 10000.0f;
		}
      #ifdef DEBUG_RT_CUDA
      /*d_debug_float4[threadIdx.x*5+0]= pile[0];
      d_debug_float4[threadIdx.x*5+1]= pile[1];
      d_debug_float4[threadIdx.x*5+2]= pile[2];
      d_debug_float4[threadIdx.x*5+3]= pile[3];
      d_debug_float4[threadIdx.x*5+4]= pile[4];*/
      #endif
#ifdef DEVICE_EMU
//    printf("%d: pile[0] = %e %e %e %e\n", threadIdx.x, pile[0].x, pile[0].y, pile[0].z, pile[0].w);
//    printf("%d: pile[1] = %e %e %e %e\n", threadIdx.x, pile[1].x, pile[1].y, pile[1].z, pile[1].w);
//    printf("%d: pile[2] = %e %e %e %e\n", threadIdx.x, pile[2].x, pile[2].y, pile[2].z, pile[2].w);
//    printf("%d: pile[3] = %e %e %e %e\n", threadIdx.x, pile[3].x, pile[3].y, pile[3].z, pile[3].w);
//    printf("%d: pile[4] = %e %e %e %e\n", threadIdx.x, pile[4].x, pile[4].y, pile[4].z, pile[4].w);
#endif
      for( int i(n-1); i > 0; i-- )
				pile[i-1] = pile[i-1] + 0.8f*pile[i];
#ifdef DEVICE_EMU
//    printf("%d: pile[0] = %e %e %e %e\n", threadIdx.x, pile[0].x, pile[0].y, pile[0].z, pile[0].w);
#endif
      result[id] += rgbaFloatToInt(pile[0]);
	}
}
