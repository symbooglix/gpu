#include "hip/hip_runtime.h"
//pass
//--blockDim=256 --gridDim=2

#include "common.h"

///////////////////////////////////////
//// Compute reverse substring matches
///////////////////////////////////////

__global__ void
mummergpuRCKernel(MatchCoord* match_coords,
               char* queries, 
               const int* queryAddrs,
			   const int* queryLengths,
               const int numQueries,
			   const int min_match_len) 
{

   int qryid = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
   if (qryid >= numQueries) { return; }
   int qlen = queryLengths[qryid];

   XPRINTF("> rc qryid: %d\n", qryid);

   queries++; // skip the 'q' character


   // start at root for first query character
   TextureAddress cur;

   int mustmatch = 0;
   int qry_match_len = 0;

   int qryAddr=queryAddrs[qryid];
   MatchCoord * result = match_coords + qryAddr - __umul24(qryid, min_match_len + 1);
   queries += qryAddr;

   for (int qrystart = qlen;
       qrystart >= min_match_len ;
       qrystart--, result++)
   {
      #ifdef VERBOSE
      queries[qrystart] = '\0';
	  XPRINTF("qry: ", queries);
      for (int j = qrystart-1; j >= 0; j--)
      { XPRINTF("%c", rc(queries[j])); }
      XPRINTF("\n");
      #endif

	  PixelOfNode node;
      TextureAddress prev;

      if (((cur.data == 0)) || (qry_match_len < 1))
      {
	    // start at root of tree
	    cur.x = 0; cur.y = 1;
	    qry_match_len = 1; 
        mustmatch = 0;
      }

	  char c = rc(queries[qrystart-qry_match_len]);

	  XPRINTF("In node (%d,%d): starting with %c [%d] =>  \n", cur.x, cur.y, c, qry_match_len);

	  int refpos = 0;
	  while ((c != '\0'))
	  {
		 XPRINTF("Next edge to follow: %c (%d)\n", c, qry_match_len);

	     PixelOfChildren children;
		 children.data = tex2D(childrentex,cur.x, cur.y);
		 prev = cur;

		 switch(c)
		 {
			case 'A': cur=children.children[0]; break;
			case 'C': cur=children.children[1]; break;
			case 'G': cur=children.children[2]; break;
			case 'T': cur=children.children[3]; break;
            default: cur.data = 0; break;
		 };		 

		 XPRINTF(" In node: (%d,%d)\n", cur.x, cur.y);

		 // No edge to follow out of the node
         if (cur.data == 0)
		 {
			XPRINTF(" no edge\n");
	        set_result(prev, result, 0, qry_match_len, min_match_len, 
                       REVERSE);

            qry_match_len -= 1;
            mustmatch = 0;

			goto NEXT_SUBSTRING;
		 }

         {
           unsigned short xval = cur.data & 0xFFFF;
           unsigned short yval = (cur.data & 0xFFFF0000) >> 16;
		   node.data = tex2D(nodetex, xval, yval);
         }

		 XPRINTF(" Edge coordinates: %d - %d\n", node.start, node.end);

         if (mustmatch)
         {
           int edgelen = node.end - node.start+1;
           if (mustmatch >= edgelen)
           {
             XPRINTF(" mustmatch(%d) >= edgelen(%d), skipping edge\n", mustmatch, edgelen);

             refpos = node.end+1;
             qry_match_len += edgelen;
             mustmatch -= edgelen;
           }
           else
           {
             XPRINTF(" mustmatch(%d) < edgelen(%d), skipping to:%d\n", 
                     mustmatch, edgelen, node.start+mustmatch);

             qry_match_len += mustmatch;
             refpos = node.start + mustmatch;
             mustmatch = 0;
           }
         }
         else
         {
           // Try to walk the edge, the first char definitely matches
           qry_match_len++;
           refpos = node.start+1;
         }

		 c = rc(queries[qrystart-qry_match_len]);

		 while (refpos <= node.end && c != '\0')
		 { 
            char r = getRef(refpos);

			XPRINTF(" Edge cmp ref: %d %c, qry: %d %c\n", refpos, r, qry_match_len, c);
						
			if (r != c)
			{
			   // mismatch on edge
			   XPRINTF("mismatch on edge: %d, edge_pos: %d\n", qry_match_len,refpos - (node.start));
               goto RECORD_RESULT;
			}

	        qry_match_len++;
			refpos++;
			c = rc(queries[qrystart-qry_match_len]);
		 }
	  }

	  XPRINTF("end of string\n");

      RECORD_RESULT:
	
      set_result(cur, result, refpos - node.start, qry_match_len, 
                 min_match_len, REVERSE);

      mustmatch = refpos - node.start;
      qry_match_len -= mustmatch + 1;

      NEXT_SUBSTRING:

      node.data = tex2D(nodetex, prev.x, prev.y);
      cur = node.suffix;

      XPRINTF(" following suffix link. mustmatch:%d qry_match_len:%d sl:(%d,%d)\n", 
              mustmatch, qry_match_len, cur.x, cur.y);

      do {} while(0);
   }
	
   return;
}
