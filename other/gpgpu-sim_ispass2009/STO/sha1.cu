#include "hip/hip_runtime.h"
//pass
//--blockDim=32 --gridDim=2

#include "sha1_common.h"

/*==========================================================================
                                SHA1 KERNEL

* Copyright (c) 2008, NetSysLab at the University of British Columbia
* All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions are met:
*     * Redistributions of source code must retain the above copyright
*       notice, this list of conditions and the following disclaimer.
*     * Redistributions in binary form must reproduce the above copyright
*       notice, this list of conditions and the following disclaimer in the
*       documentation and/or other materials provided with the distribution.
*     * Neither the name of the University nor the
*       names of its contributors may be used to endorse or promote products
*       derived from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY NetSysLab ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
* WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
* DISCLAIMED. IN NO EVENT SHALL NetSysLab BE LIABLE FOR ANY
* DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
* (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
* LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
* ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
* SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

DESCRIPTION
  CPU version of the storeGPU library.


==========================================================================*/

/*===========================================================================

FUNCTION SHA1

DESCRIPTION
  Main sha1 hash function

DEPENDENCIES
  GPU must be initialized

RETURN VALUE
  output: the hash result

===========================================================================*/
__global__
void sha1( unsigned char *input, int chunkSize, int totalThreads,
	   int padSize, unsigned char *scratch ) {
  __requires(chunkSize == 1012);
  __requires(totalThreads == 98304);
  __requires(padSize == 0);
  
  // get the current thread index
  int threadIndex = threadIdx.x + blockDim.x * blockIdx.x;
  int chunkIndex = threadIndex * chunkSize;
  int hashIndex  = threadIndex * SHA1_HASH_SIZE;

  if(threadIndex >= totalThreads)
    return;
  
  if ((threadIndex == (totalThreads - 1)) && (padSize > 0)) {
    for(int i = 0 ; i < padSize ; i++)
      input[chunkIndex + chunkSize - padSize + i] = 0;	
  }
  
#ifdef FEATURE_SHARED_MEMORY
  
  __shared__ unsigned int sharedMemory[4 * 1024 - 32];
  
  unsigned int *sharedMemoryIndex = sharedMemory + ((threadIdx.x >> 5) * 512);
  unsigned char *tempInput = input + chunkIndex;
  unsigned int *inputIndex = (unsigned int *)(tempInput);
  
  sha1_internal(inputIndex, sharedMemoryIndex, chunkSize, 
	       scratch + hashIndex );

#else
  sha1_internal(input + chunkIndex, chunkSize, scratch + hashIndex );
#endif /* FEATURE_SHARED_MEMORY */

}
