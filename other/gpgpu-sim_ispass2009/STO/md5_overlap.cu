#include "hip/hip_runtime.h"
//pass
//--blockDim=32 --gridDim=2

#include "md5_common.h"

__global__
void md5_overlap( unsigned char *input, int chunkSize, int offset,
		  int totalThreads, int padSize, unsigned char *output ) {
  __requires(chunkSize == 52);
  __requires(offset == 4);
  __requires(totalThreads == 49152);
  __requires(padSize == 0);

  int threadIndex = threadIdx.x + blockDim.x * blockIdx.x;
  int chunkIndex = threadIndex * offset;
  int hashIndex  = threadIndex * MD5_HASH_SIZE;


  if(threadIndex >= totalThreads)
    return;
  
  if ((threadIndex == (totalThreads - 1))) {
    chunkSize-= padSize;
  }


#ifdef FEATURE_SHARED_MEMORY
  
  __shared__ unsigned int sharedMemory[4 * 1024 - 32];
  
  unsigned int *sharedMemoryIndex = sharedMemory + ((threadIdx.x >> 5) * 512);
  unsigned int *inputIndex = (unsigned int *)(input + chunkIndex);
  
  md5_internal_overlap(inputIndex, sharedMemoryIndex, chunkSize, 
	       output + hashIndex );

#else
  md5_internal(input + chunkIndex, chunkSize, output + hashIndex );
#endif /* FEATURE_SHARED_MEMORY */


}
