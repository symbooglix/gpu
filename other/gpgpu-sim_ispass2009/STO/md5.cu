#include "hip/hip_runtime.h"
//pass
//--blockDim=192 --gridDim=512

#include "md5_common.h"

/*===========================================================================

FUNCTION MD5

DESCRIPTION
  Main md5 hash function

DEPENDENCIES
  GPU must be initialized

RETURN VALUE
  output: the hash result

===========================================================================*/

__global__
void md5( unsigned char *input, int chunkSize, int totalThreads,
          int padSize, unsigned char *scratch) {
  __requires(chunkSize == 1012);
  __requires(totalThreads == 98304);
  __requires(padSize == 0);
  
  int threadIndex = threadIdx.x + blockDim.x * blockIdx.x;
  int chunkIndex = threadIndex * chunkSize;
  int hashIndex  = threadIndex * MD5_HASH_SIZE;

  if(threadIndex >= totalThreads)
    return;
  
  if ((threadIndex == (totalThreads - 1)) && (padSize > 0)) {
    for(int i = 0 ; i < padSize ; i++)
      input[chunkIndex + chunkSize - padSize + i] = 0;
  }


#ifdef FEATURE_SHARED_MEMORY
  
  __shared__ unsigned int sharedMemory[4 * 1024 - 32];
  
  // 512 words are allocated for every warp of 32 threads
  unsigned int *sharedMemoryIndex = sharedMemory + ((threadIdx.x >> 5) * 512);
  unsigned int *inputIndex = (unsigned int *)(input + chunkIndex);
  
  md5_internal(inputIndex, sharedMemoryIndex, chunkSize, 
	       scratch + hashIndex );

#else
  md5_internal(input + chunkIndex, chunkSize, scratch + hashIndex );
#endif /* FEATURE_SHARED_MEMORY */

}
