#include "hip/hip_runtime.h"
//pass
//--blockDim=[32,1] --gridDim=[1,1]

#include <hip/hip_runtime.h>

#define N 32

// Define this to more rigorously avoid bank conflicts, 
// even at the lower (root) levels of the tree
// Note that due to the higher addressing overhead, performance 
// is lower with ZERO_BANK_CONFLICTS enabled.  It is provided
// as an example.
//#define ZERO_BANK_CONFLICTS 

// 16 banks on G80
#define NUM_BANKS 16
#define LOG_NUM_BANKS 4

#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(index) ((index) >> LOG_NUM_BANKS + (index) >> (2*LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(index) ((index) >> LOG_NUM_BANKS)
#endif

__global__ void k(float *g_odata, 
                        const float *g_idata, 
                        float *g_blockSums, 
                        int n, 
                        int blockIndex, 
                        int baseIndex,
                        int storeSum, int isNP2)
{
  int ai, bi, mem_ai, mem_bi, bankOffsetA, bankOffsetB;
  /*extern*/ __shared__ float s_data[N*2];

  // ------------------------------------------------------------------------
  // loadSharedChunkFromMem()
  // ------------------------------------------------------------------------
  baseIndex = (baseIndex == 0) ?  blockIdx.x * (blockDim.x << 1) : baseIndex;
  int thid = threadIdx.x;
  mem_ai = baseIndex + threadIdx.x;
  mem_bi = mem_ai + blockDim.x;

  ai = thid;
  bi = thid + blockDim.x;

  // compute spacing to avoid bank conflicts
  bankOffsetA = CONFLICT_FREE_OFFSET(ai);
  bankOffsetB = CONFLICT_FREE_OFFSET(bi);

  // Cache the computational window in shared memory
  // pad values beyond n with zeros

  s_data[ai + bankOffsetA] = g_idata[mem_ai];

  if (isNP2 != 0) // compile-time decision
  {
    s_data[bi + bankOffsetB] = (bi < n) ? g_idata[mem_bi] : 0;
  }
  else
  {
    s_data[bi + bankOffsetB] = g_idata[mem_bi];
  }

  // ------------------------------------------------------------------------
  // prescanBlock()
  // ------------------------------------------------------------------------

  // ------------------------------------------------------------------------
  // -- buildSum()
  // ------------------------------------------------------------------------
  unsigned int stride = 1;
  // build the sum in place up the tree
  for (int d = blockDim.x;
    __invariant(__implies((d == 0) & __write(s_data), thid == 0)),
    __invariant(__implies((d == 0) & __read(s_data), thid == 0)),
          d > 0; d >>= 1)
  {
    __syncthreads();

    stride *= 2;

    if (thid < d)
    {
      int i_1  = stride * thid;
      int ai_1 = i_1 + stride/2 - 1;
      int bi_1 = ai_1 + stride/2;

      ai_1 += CONFLICT_FREE_OFFSET(ai_1);
      bi_1 += CONFLICT_FREE_OFFSET(bi_1);

#ifdef MUTATION
      s_data[0] += s_data[ai_1];
#else
      s_data[bi_1] += s_data[ai_1];
#endif
       /* BUGINJECT: MUTATE_OFFSET, UP, ZERO */
    }
  }

  // ------------------------------------------------------------------------
  // -- clearLastElement()
  // ------------------------------------------------------------------------
  blockIndex = (blockIndex == 0) ? blockIdx.x : blockIndex;
  if (threadIdx.x == 0)
  {
    int index = (blockDim.x << 1) - 1;
    index += CONFLICT_FREE_OFFSET(index);

    if (storeSum != 0) // compile-time decision
    {
      // write this block's total sum to the corresponding index in the blockSums array
      g_blockSums[blockIndex] = s_data[index];
    }

    // zero the last element in the scan so it will propagate back to the front
    s_data[index] = 0;
  }

  // ------------------------------------------------------------------------
  // -- scanRootToLeaves()
  // ------------------------------------------------------------------------
  for (int d = 1; d <= blockDim.x; d *= 2)
  {
    stride >>= 1;

    __syncthreads();

    if (thid < d)
    {
      int i_2  =  2 * stride * thid;
      int ai_2 = i_2 + stride - 1;
      int bi_2 = ai_2 + stride;
      ai_2 += CONFLICT_FREE_OFFSET(ai_2);
      bi_2 += CONFLICT_FREE_OFFSET(bi_2);

      float t      = s_data[ai_2];
      s_data[ai_2] = s_data[bi_2];
      s_data[bi_2] += t; 
    }
  }

  // ------------------------------------------------------------------------
  // storeSharedChunkToMem()
  // ------------------------------------------------------------------------
  __syncthreads();

  // write results to global memory
  g_odata[mem_ai] = s_data[ai + bankOffsetA]; 
  if (isNP2 != 0) // compile-time decision
  {
    if (bi < n)
      g_odata[mem_bi] = s_data[bi + bankOffsetB]; 
  }
  else
  {
    g_odata[mem_bi] = s_data[bi + bankOffsetB]; 
  }
}
