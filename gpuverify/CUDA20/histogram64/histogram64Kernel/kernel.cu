#include "hip/hip_runtime.h"
//pass
//--blockDim=[128,1] --gridDim=[64,1]

#include <hip/hip_runtime.h>

#define BIN_COUNT 64

////////////////////////////////////////////////////////////////////////////////
// GPU-specific definitions
////////////////////////////////////////////////////////////////////////////////
//Fast mul on G8x / G9x / G100
#define IMUL(a, b) a * b

//Threads block size for histogram64Kernel()
//Preferred to be a multiple of 64 (refer to the supplied whitepaper)
//REVISIT: 192 is not a pow2 so is very slow to prove
//#define THREAD_N 192
#define THREAD_N 128


////////////////////////////////////////////////////////////////////////////////
// If threadPos == threadIdx.x, there are always  4-way bank conflicts,
// since each group of 16 threads (half-warp) accesses different bytes,
// but only within 4 shared memory banks. Having shuffled bits of threadIdx.x 
// as in histogram64GPU(), each half-warp accesses different shared memory banks
// avoiding any bank conflicts at all.
// Refer to the supplied whitepaper for detailed explanations.
////////////////////////////////////////////////////////////////////////////////
// REVISIT: this inline syntax does not work
__device__ inline void addData64(unsigned char *s_Hist, int threadPos, unsigned int data) __attribute__((always_inline));
__device__ inline void addData64(unsigned char *s_Hist, int threadPos, unsigned int data) {
    s_Hist[threadPos + IMUL(data, THREAD_N)]++;
}

////////////////////////////////////////////////////////////////////////////////
// Main computation pass: compute gridDim.x partial histograms
////////////////////////////////////////////////////////////////////////////////
__global__ void histogram64Kernel(unsigned int *d_Result, unsigned int *d_Data, int dataN){
    //Encode thread index in order to avoid bank conflicts in s_Hist[] access:
    //each half-warp accesses consecutive shared memory banks
    //and the same bytes within the banks

    const int threadPos =
        //[31 : 6] <== [31 : 6]
        ((threadIdx.x & (~63)) >> 0) |
        //[5  : 2] <== [3  : 0]
        ((threadIdx.x &    15) << 2) |
        //[1  : 0] <== [5  : 4]
        ((threadIdx.x &    48) >> 4);

    //Per-thread histogram storage
    __shared__ unsigned char s_Hist[THREAD_N * BIN_COUNT];

    //Flush shared memory
    for(int i = 0;
             i < BIN_COUNT / 4; i++) {
      //         ((unsigned int *)s_Hist)[threadIdx.x + i * THREAD_N] = 0; 
      s_Hist[threadIdx.x + i * THREAD_N] = 0;
    }

    __syncthreads();

    ////////////////////////////////////////////////////////////////////////////
    // Cycle through current block, update per-thread histograms
    // Since only 64-bit histogram of 8-bit input data array is calculated,
    // only highest 6 bits of each 8-bit data element are extracted,
    // leaving out 2 lower bits.
    ////////////////////////////////////////////////////////////////////////////
    unsigned int data4;
    for(int pos = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
        // These two invariants are strong (fast to prove) but only valid if THREAD_N/blockDim.x is a pow2
        __global_invariant(
          __implies(__is_pow2(THREAD_N),
            __read_implies(s_Hist,
              __mod_pow2( __read_offset_bytes(s_Hist) - (((data4 >>  2) & 63) * THREAD_N), THREAD_N) == threadPos |
              __mod_pow2( __read_offset_bytes(s_Hist) - (((data4 >> 10) & 63) * THREAD_N), THREAD_N) == threadPos |
              __mod_pow2( __read_offset_bytes(s_Hist) - (((data4 >> 18) & 63) * THREAD_N), THREAD_N) == threadPos |
              __mod_pow2( __read_offset_bytes(s_Hist) - (((data4 >> 26) & 63) * THREAD_N), THREAD_N) == threadPos))),
        __global_invariant(
          __implies(__is_pow2(THREAD_N),
            __write_implies(s_Hist,
              __mod_pow2(__write_offset_bytes(s_Hist) - (((data4 >>  2) & 63) * THREAD_N), THREAD_N) == threadPos |
              __mod_pow2(__write_offset_bytes(s_Hist) - (((data4 >> 10) & 63) * THREAD_N), THREAD_N) == threadPos |
              __mod_pow2(__write_offset_bytes(s_Hist) - (((data4 >> 18) & 63) * THREAD_N), THREAD_N) == threadPos |
              __mod_pow2(__write_offset_bytes(s_Hist) - (((data4 >> 26) & 63) * THREAD_N), THREAD_N) == threadPos))),
                  pos < dataN; pos += IMUL(blockDim.x, gridDim.x)){
        data4 = d_Data[pos];
        addData64(s_Hist, threadPos, (data4 >>  2) & 0x3FU);
        addData64(s_Hist, threadPos, (data4 >> 10) & 0x3FU);
        addData64(s_Hist, threadPos, (data4 >> 18) & 0x3FU);
        addData64(s_Hist, threadPos, (data4 >> 26) & 0x3FU);
    }

    __syncthreads();

    ////////////////////////////////////////////////////////////////////////////
    // Merge per-thread histograms into per-block and write to global memory.
    // Start accumulation positions for half-warp each thread are shifted
    // in order to avoid bank conflicts. 
    // See supplied whitepaper for detailed explanations.
    ////////////////////////////////////////////////////////////////////////////
     /* BUGINJECT: ADD_BARRIER, DOWN */
    if(threadIdx.x < BIN_COUNT){
#ifdef MUTATION
        __syncthreads();
#endif

        unsigned int sum = 0;
        const int value = threadIdx.x;

        const int valueBase = IMUL(value, THREAD_N);
        const int  startPos = (threadIdx.x & 15) * 4;

        //Threads with non-zero start positions wrap around the THREAD_N border
        // REVISIT: loop index clash with loop0 rewritten to use 'j' instead
        for(int j = 0, accumPos = startPos; j < THREAD_N; j++){
            sum += s_Hist[valueBase + accumPos];
            accumPos++;
            if(accumPos == THREAD_N) accumPos = 0;
        }

        d_Result[blockIdx.x * BIN_COUNT + value] = sum;
    }
}
