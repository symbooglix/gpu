//pass
//--blockDim=[64,1] --gridDim=[64,1]

#include <hip/hip_runtime.h>

#define BIN_COUNT 64

////////////////////////////////////////////////////////////////////////////////
// GPU-specific definitions
////////////////////////////////////////////////////////////////////////////////
//Fast mul on G8x / G9x / G100
#define IMUL(a, b) a * b

////////////////////////////////////////////////////////////////////////////////
// Merge blockN histograms into gridDim.x histograms
// blockDim.x == BIN_COUNT
// gridDim.x  == BLOCK_N2
////////////////////////////////////////////////////////////////////////////////
#define MERGE_THREADS 64


__global__ void mergeHistogram64Kernel(
    unsigned int *d_Histogram,
    unsigned int *d_PartialHistograms,
    unsigned int blockN
){
    __shared__ unsigned int data[MERGE_THREADS];

    unsigned int sum = 0;
    for(unsigned int i = threadIdx.x; i < blockN; i += MERGE_THREADS) {
        sum += d_PartialHistograms[blockIdx.x + i * BIN_COUNT];
    }
    data[threadIdx.x] = sum;

    for(unsigned int stride = MERGE_THREADS / 2;
        stride > 0; stride >>= 1){
        __syncthreads();
         /* BUGINJECT: ADD_BARRIER, DOWN */
        if(threadIdx.x < stride) {
#ifdef MUTATION
            __syncthreads();
#endif
            data[threadIdx.x] += data[threadIdx.x + stride];
        }
    }

    if(threadIdx.x == 0)
        d_Histogram[blockIdx.x] = data[0];
}
