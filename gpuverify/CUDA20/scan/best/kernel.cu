#include "hip/hip_runtime.h"
//pass
//--blockDim=[32,1] --gridDim=[1,1]

#include <hip/hip_runtime.h>

#define N 32

#define NUM_BANKS 16
#define LOG_NUM_BANKS 4

// Define this to more rigorously avoid bank conflicts, even at the lower (root) levels of the tree
//#define ZERO_BANK_CONFLICTS 

#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(index) ((index) >> LOG_NUM_BANKS + (index) >> (2 * LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(index) ((index) >> LOG_NUM_BANKS)
// #define CONFLICT_FREE_OFFSET(index) (index)
#endif

#ifdef CHECK_BANK_CONFLICTS
#define TEMP(index)   cutilBankChecker(temp, index)
#else
#define TEMP(index)   temp[index]
#endif

///////////////////////////////////////////////////////////////////////////////
// Work-efficient compute implementation of scan, one thread per 2 elements
// Work-efficient: O(log(n)) steps, and O(n) adds.
// Also shared storage efficient: Uses n + n/NUM_BANKS shared memory -- no ping-ponging
// Also avoids most bank conflicts using single-element offsets every NUM_BANKS elements.
//
// In addition, If ZERO_BANK_CONFLICTS is defined, uses 
//     n + n/NUM_BANKS + n/(NUM_BANKS*NUM_BANKS) 
// shared memory. If ZERO_BANK_CONFLICTS is defined, avoids ALL bank conflicts using 
// single-element offsets every NUM_BANKS elements, plus additional single-element offsets 
// after every NUM_BANKS^2 elements.
//
// Uses a balanced tree type algorithm.  See Blelloch, 1990 "Prefix Sums 
// and Their Applications", or Prins and Chatterjee PRAM course notes:
// http://www.cs.unc.edu/~prins/Classes/203/Handouts/pram.pdf
// 
// This work-efficient version is based on the algorithm presented in Guy Blelloch's
// Excellent paper "Prefix sums and their applications".
// http://www-2.cs.cmu.edu/afs/cs.cmu.edu/project/scandal/public/papers/CMU-CS-90-190.html
//
// Pro: Work Efficient, very few bank conflicts (or zero if ZERO_BANK_CONFLICTS is defined)
// Con: More instructions to compute bank-conflict-free shared memory addressing,
// and slightly more shared memory storage used.
//
// @param g_odata  output data in global memory
// @param g_idata  input data in global memory
// @param n        input number of elements to scan from input data
__global__ void scanBestKernel(float *g_odata, float *g_idata, int n)
{
    __requires(n == blockDim.x*2);
    __requires(__is_pow2(n));

    // Dynamically allocated shared memory for scan kernels
    /*extern*/ __shared__  float temp[N*2];

    int thid = threadIdx.x;

#ifdef NORENAME
    int ai = thid;
    int bi = thid + (n >> 1);

    // compute spacing to avoid bank conflicts
    int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    int bankOffsetB = CONFLICT_FREE_OFFSET(bi);

    // Cache the computational window in shared memory
    TEMP(ai + bankOffsetA) = g_idata[ai]; 
    TEMP(bi + bankOffsetB) = g_idata[bi]; 
#else
    int ai_outer = thid;
    int bi_outer = thid + (n >> 1);

    // compute spacing to avoid bank conflicts
    int bankOffsetA = CONFLICT_FREE_OFFSET(ai_outer);
    int bankOffsetB = CONFLICT_FREE_OFFSET(bi_outer);

    // Cache the computational window in shared memory
    TEMP(ai_outer + bankOffsetA) = g_idata[ai_outer]; 
#ifdef MUTATION
    TEMP(0) = g_idata[bi_outer]; 
#else
    TEMP(bi_outer + bankOffsetB) = g_idata[bi_outer]; 
#endif
      /* BUGINJECT: MUTATE_OFFSET, UP, ZERO */
#endif

    __syncthreads();

    int offset = 1;

    // build the sum in place up the tree
    for (int d = n >> 1;
        __invariant(__implies((d == 0) & __write(temp), thid == 0)),
        __invariant(__implies((d == 0) & __read(temp), thid == 0)),
            d > 0; d >>= 1)
    {
        __syncthreads();

	      offset *= 2;

        if (thid < d)      
        {
            int ai = offset/2*(2*thid+1)-1;
            int bi = offset/2*(2*thid+2)-1;

            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            TEMP(bi) += TEMP(ai);
        }

    }

    // scan back down the tree

    // clear the last element
    if (thid == 0)
    {
        int index = n - 1;
        index += CONFLICT_FREE_OFFSET(index);
        TEMP(index) = 0;
    }

    // traverse down the tree building the scan in place
    for (int d = 1; d < n; d *= 2)
    {
        offset /= 2;

        __syncthreads();

        if (thid < d)
        {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;

            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            float t  = TEMP(ai);
            TEMP(ai) = TEMP(bi);
            TEMP(bi) += t;
        }
    }

    __syncthreads();

    // write results to global memory
#ifdef NORENAME
    g_odata[ai] = TEMP(ai + bankOffsetA); 
    g_odata[bi] = TEMP(bi + bankOffsetB); 
#else
    g_odata[ai_outer] = TEMP(ai_outer + bankOffsetA); 
    g_odata[bi_outer] = TEMP(bi_outer + bankOffsetB); 
#endif

    
}
