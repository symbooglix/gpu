#include "hip/hip_runtime.h"
//pass
//--blockDim=[32,1] --gridDim=[1,1]

#include <hip/hip_runtime.h>

#define N 32

///////////////////////////////////////////////////////////////////////////////
//! Work-efficient compute implementation of scan, one thread per 2 elements
//! Work-efficient: O(log(n)) steps, and O(n) adds.
//! Also shared storage efficient: Uses n elements in shared mem -- no ping-ponging
//! Uses a balanced tree type algorithm.  See Blelloch, 1990 "Prefix Sums 
//! and Their Applications", or Prins and Chatterjee PRAM course notes:
//! http://www.cs.unc.edu/~prins/Classes/203/Handouts/pram.pdf
//!
//! Pro: Work Efficient
//! Con: Shared memory bank conflicts due to the addressing used.
//
//! @param g_odata  output data in global memory
//! @param g_idata  input data in global memory
//! @param n        input number of elements to scan from input data
///////////////////////////////////////////////////////////////////////////////

__global__ void scan_workefficient_kernel (float *g_odata, float *g_idata, int n)
{
    __requires(n == blockDim.x*2);
    __requires(__is_pow2(n));

    // Dynamically allocated shared memory for scan kernels
    /*extern*/ __shared__  float temp[N*2];

    int thid = threadIdx.x;

    int offset = 1;

    // Cache the computational window in shared memory
    temp[2*thid]   = g_idata[2*thid];
    temp[2*thid+1] = g_idata[2*thid+1];

    // build the sum in place up the tree
    for (int d = n>>1;
      __invariant(__implies((d == 0) & __write(temp), thid == 0)),
      __invariant(__implies((d == 0) & __read(temp), thid == 0)),
        d > 0; d >>= 1)
    {

      __syncthreads();

      offset *= 2;

      if (thid < d)
      {
        int ai = offset/2*(2*thid+1)-1;
        int bi = offset/2*(2*thid+2)-1;

        temp[bi] = 1;
        temp[bi] += temp[ai];
      }
    }

    // scan back down the tree

    // clear the last element
    if (thid == 0)
    {
        temp[n - 1] = 0;
    }

    // traverse down the tree building the scan in place
    for (int d = 1; d < n; d *= 2)
    {
      __syncthreads();

      offset >>= 1;

      if (thid < d)
      {
        int ai = offset*(2*thid+1)-1;
        int bi = offset*(2*thid+2)-1;

        float t = temp[ai];
        temp[ai]  = temp[bi];
        temp[bi] += t;
      }
    }

#ifndef MUTATION
     /* BUGINJECT: REMOVE_BARRIER, DOWN */
    __syncthreads();
#endif

    // write results to global memory
    g_odata[2*thid]   = temp[2*thid];
    g_odata[2*thid+1] = temp[2*thid+1];
}
