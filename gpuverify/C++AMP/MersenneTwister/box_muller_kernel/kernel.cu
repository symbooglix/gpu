//pass
//--blockDim=1024 --gridDim=4

#include <hip/hip_runtime.h>

//////////////////////////////////////////////////////////////////////////////
//// Copyright (c) Microsoft Corporation. All rights reserved
//// This software contains source code provided by NVIDIA Corporation.
//////////////////////////////////////////////////////////////////////////////

//----------------------------------------------------------------------------
// File: MersenneTwister.cpp
// 
// This sample implements Mersenne Twister random number generator 
// and Cartesian Box-Muller transformation on the GPU.
//----------------------------------------------------------------------------

#define   MT_RNG_COUNT 4096
#define          MT_MM 9
#define          MT_NN 19
#define       MT_WMASK 0xFFFFFFFFU
#define       MT_UMASK 0xFFFFFFFEU
#define       MT_LMASK 0x1U
#define      MT_SHIFT0 12
#define      MT_SHIFTB 7
#define      MT_SHIFTC 15
#define      MT_SHIFT1 18

////////////////////////////////////////////////////////////////////////////////
// Transform each of MT_RNG_COUNT lanes of n_per_RNG uniformly distributed 
// random samples, produced by rand_MT_amp(), to normally distributed lanes
// using Cartesian form of Box-Muller transformation.
// n_per_RNG must be even.
////////////////////////////////////////////////////////////////////////////////
static __attribute__((always_inline)) __device__ void box_muller_transform(float* u1, float* u2)
{
    float r = sqrt(-2.0f * log(*u1));
    float phi = 2.0f * 3.14159265358979f * (*u2);
    *u1 = r * cos(phi);
    *u2 = r * sin(phi);
}

__global__ void box_muller_kernel(float* random_nums, float* normalized_random_nums, int n_per_RNG)
{
    int gid = (blockIdx.x*blockDim.x + threadIdx.x);

    for(int out = 0;
                  out < n_per_RNG; out += 2) 
	{
		float f0 = random_nums[out * MT_RNG_COUNT + gid];
		float f1 = random_nums[(out + 1) * MT_RNG_COUNT + gid];
                box_muller_transform(&f0, &f1);
                normalized_random_nums[out * MT_RNG_COUNT + gid] = f0;
                normalized_random_nums[(out + 1) * MT_RNG_COUNT + gid] = f1;
#ifdef MUTATION
    normalized_random_nums[out * MT_RNG_COUNT + gid + 1] = normalized_random_nums[out * MT_RNG_COUNT + gid + 1];
                 /* BUGINJECT: ADD_ACCESS, UP */
#endif
    }
}

