//pass
//--blockDim=[1024,1] --gridDim=[4,1]

#include <hip/hip_runtime.h>

//////////////////////////////////////////////////////////////////////////////
//// Copyright (c) Microsoft Corporation. All rights reserved
//// This software contains source code provided by NVIDIA Corporation.
//////////////////////////////////////////////////////////////////////////////

//----------------------------------------------------------------------------
// File: MersenneTwister.cpp
// 
// This sample implements Mersenne Twister random number generator 
// and Cartesian Box-Muller transformation on the GPU.
//----------------------------------------------------------------------------

#define   MT_RNG_COUNT 4096
#define          MT_MM 9
#define          MT_NN 19
#define       MT_WMASK 0xFFFFFFFFU
#define       MT_UMASK 0xFFFFFFFEU
#define       MT_LMASK 0x1U
#define      MT_SHIFT0 12
#define      MT_SHIFTB 7
#define      MT_SHIFTC 15
#define      MT_SHIFT1 18


////////////////////////////////////////////////////////////////////////////////
// Write MT_RNG_COUNT vertical lanes of n_per_RNG random numbers to random_nums.
// For coalesced global writes MT_RNG_COUNT should be a multiple of hardware scehduling unit size.
// Hardware scheduling unit is called warp or wave or wavefront
// Initial states for each generator are the same, since the states are
// initialized from the global seed. In order to improve distribution properties
// on small n_per_RNG supply dedicated (local) seed to each twister.
// The local seeds, in their turn, can be extracted from global seed
// by means of any simple random number generator, like LCG.
////////////////////////////////////////////////////////////////////////////////
__global__ void rand_MT_kernel(
			   float* random_nums, 
			   const unsigned int matrix_a, 
			   const unsigned int mask_b, const unsigned int mask_c, 
			   const unsigned int seed, const int n_per_RNG)
{
    int state_1;
    int state_M;
    unsigned int mti, mti_M, x;
    unsigned int mti_1, mt[MT_NN];

    //Bit-vector Mersenne Twister parameters are in matrix_a, mask_b, mask_c, seed
    //Initialize current state
    mt[0] = seed;
    for(int state = 1; state < MT_NN; state++)
        mt[state] = (1812433253U * (mt[state - 1] ^ (mt[state - 1] >> 30)) + state) & MT_WMASK;

    mti_1 = mt[0];
    for(int out = 0, state = 0; 
          out < n_per_RNG; out++) 
	{
        state_1 = state + 1;
        state_M = state + MT_MM;
        if (state_1 >= MT_NN) state_1 -= MT_NN;
        if (state_M >= MT_NN) state_M -= MT_NN;
        mti  = mti_1;
        mti_1 = mt[state_1];
        mti_M = mt[state_M];

        x    = (mti & MT_UMASK) | (mti_1 & MT_LMASK);
        x    =  mti_M ^ (x >> 1) ^ ((x & 1) ? matrix_a : 0);
        mt[state] = x;
        state = state_1;

        //Tempering transformation
        x ^= (x >> MT_SHIFT0);
        x ^= (x << MT_SHIFTB) & mask_b;
        x ^= (x << MT_SHIFTC) & mask_c;
        x ^= (x >> MT_SHIFT1);

        
        //Convert to (0, 1) float and write to global memory
		// Using UINT max, to convert a uniform number in uint range to a uniform range over [-1 ... 1] 
        random_nums[out*MT_RNG_COUNT + (blockIdx.x * blockDim.x + threadIdx.x)] = ((float)x + 1.0f) / 4294967296.0f;
#ifdef MUTATION
        random_nums[out*MT_RNG_COUNT + (blockIdx.x * blockDim.x + threadIdx.x) + 1] = random_nums[out*MT_RNG_COUNT + (blockIdx.x * blockDim.x + threadIdx.x) + 1];
         /* BUGINJECT: ADD_ACCESS, UP */
#endif
    }
}

