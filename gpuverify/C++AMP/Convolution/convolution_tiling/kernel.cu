//pass
//--blockDim=[1,128] --gridDim=[512,6]

#include <hip/hip_runtime.h>

//////////////////////////////////////////////////////////////////////////////
//// THIS CODE AND INFORMATION IS PROVIDED "AS IS" WITHOUT WARRANTY OF
//// ANY KIND, EITHER EXPRESSED OR IMPLIED, INCLUDING BUT NOT LIMITED TO
//// THE IMPLIED WARRANTIES OF MERCHANTABILITY AND/OR FITNESS FOR A
//// PARTICULAR PURPOSE.
////
//// Copyright (c) Microsoft Corporation. All rights reserved
//////////////////////////////////////////////////////////////////////////////

//----------------------------------------------------------------------------
// File: Convolution.cpp
// 
// Implement C++ AMP based simple and tiled version of Convolution filter used in 
// image processing.
//----------------------------------------------------------------------------

#define DEFAULT_WIDTH   512
#define DEFAULT_HEIGHT  512
// TILE_SIZE should be multiple of both DEFAULT_WIDTH and DEFAULT_HEIGHT
#define TILE_SIZE		128

#define width DEFAULT_WIDTH
#define height DEFAULT_HEIGHT

#define clamp(a, b, c) ((a) < (b) ? (b) : ((a) > (c) ? (c) : (a)))

#define dim_to_convolve y

#define radius 7

//----------------------------------------------------------------------------
// Tile implementation of convolution filter along different dimension
//----------------------------------------------------------------------------
__global__ void convolution_tiling(const float* img, const float* filter, float* result)
{

    __shared__ float local_buf[TILE_SIZE];
    
    int idx_convolve = (blockIdx.dim_to_convolve)*(TILE_SIZE - 2 * radius) + (int)(threadIdx.dim_to_convolve) - radius;
    int max_idx_convolve = height;
    float sum = 0.0f;

    int a_idxY = blockIdx.y;
    int a_idxX = blockIdx.x;

    a_idxY = clamp(idx_convolve, 0, max_idx_convolve-1);
    if (idx_convolve < (max_idx_convolve + radius))
    {
        local_buf[threadIdx.dim_to_convolve] = img[a_idxY*width + a_idxX];
    }

#ifndef MUTATION
     /* BUGINJECT: REMOVE_BARRIER, DOWN */
    __syncthreads();
#endif

    if ((int)(threadIdx.dim_to_convolve) >= radius && (int)(threadIdx.dim_to_convolve) < (TILE_SIZE - radius) && idx_convolve < max_idx_convolve)
    {
        for (int k = -radius; k <= radius; k++)
        {
            int k_idx = k + radius;
            sum += local_buf[threadIdx.dim_to_convolve + k]*filter[k_idx];
        }
        result[a_idxY*width + a_idxX] = sum;
    }
}
