//pass
//--blockDim=[8,8] --gridDim=[1,1]

#include <hip/hip_runtime.h>

#define _2D_ACCESS(A, y, x, X_DIM) A[(y)*(X_DIM)+(x)]

//////////////////////////////////////////////////////////////////////////////
//// THIS CODE AND INFORMATION IS PROVIDED "AS IS" WITHOUT WARRANTY OF
//// ANY KIND, EITHER EXPRESSED OR IMPLIED, INCLUDING BUT NOT LIMITED TO
//// THE IMPLIED WARRANTIES OF MERCHANTABILITY AND/OR FITNESS FOR A
//// PARTICULAR PURPOSE.
////
//// Copyright (c) Microsoft Corporation. All rights reserved
//////////////////////////////////////////////////////////////////////////////

//----------------------------------------------------------------------------
// File: TransitiveClosure.cpp
//
// Contains the implementation of algorithms which explores connectivity between 
// nodes in a graph and determine shortest path.
// This is based on paper http://www.seas.upenn.edu/~kiderj/research/papers/APSP-gh08-fin-T.pdf
//----------------------------------------------------------------------------

// Defines to help with AMP->OpenCL translation
#define X_DIMENSION 0
#define Y_DIMENSION 1

// Constants - specifies tile size
#define TILE_SIZE (1 << 3)

// State of connection
#define UNCONNECTED 0
#define DIRECTLY_CONNECTED 1
#define INDIRECTLY_CONNECTED 2


#define num_vertices (1 << 6)

//----------------------------------------------------------------------------
// Stage2 - determine connectivity between vertexs' between 2 TILE - primary 
// and current - current is along row or column of primary
//----------------------------------------------------------------------------
__global__ void transitive_closure_stage2_kernel(unsigned int* graph, int passnum)
{
  // Load primary block into shared memory (primary_block_buffer)
  __shared__ unsigned int primary_block_buffer[TILE_SIZE][TILE_SIZE];
  int idxY = passnum * TILE_SIZE + threadIdx.y;
  int idxX = passnum * TILE_SIZE + threadIdx.x;

  primary_block_buffer[threadIdx.y][threadIdx.x] = _2D_ACCESS(graph, idxY, idxX, num_vertices);

  // Load the current block into shared memory (curr_block_buffer)
  __shared__ unsigned int curr_block_buffer[TILE_SIZE][TILE_SIZE];
  unsigned int group_id0, group_id1;
  if (blockIdx.y == 0)
  {
    group_id0 = passnum;
    if (blockIdx.x < passnum)
    {
      group_id1 = blockIdx.x;
    }
    else
    {
      group_id1 = blockIdx.x + 1;
    }
  }
  else
  {
    group_id1 = passnum;
    if (blockIdx.x < passnum)
    {
      group_id0 = blockIdx.x;
    }
    else
    {
      group_id0 = blockIdx.x + 1;
    }
  }

  idxY = group_id0 * TILE_SIZE + threadIdx.y;
  idxX = group_id1 * TILE_SIZE + threadIdx.x;
  curr_block_buffer[threadIdx.y][threadIdx.x] = _2D_ACCESS(graph, idxY, idxX, num_vertices);

#ifndef MUTATION
   /* BUGINJECT: REMOVE_BARRIER, DOWN */
  __syncthreads();
#endif

  // Now perform the actual Floyd-Warshall algorithm on this block
  for (unsigned int k = 0;
                k < TILE_SIZE; ++k)
  {
    
    if ( curr_block_buffer[threadIdx.y][threadIdx.x] == UNCONNECTED)
    {
      if (blockIdx.y == 0)
      {
        if ( (primary_block_buffer[threadIdx.y][k] != UNCONNECTED) && (curr_block_buffer[k][threadIdx.x] != UNCONNECTED) )
        {
          curr_block_buffer[threadIdx.y][threadIdx.x] = passnum*TILE_SIZE + k + INDIRECTLY_CONNECTED;
        }
      }
      else
      {
        if ( (curr_block_buffer[threadIdx.y][k] != UNCONNECTED) && (primary_block_buffer[k][threadIdx.x] != UNCONNECTED) )
        {
          curr_block_buffer[threadIdx.y][threadIdx.x] = passnum*TILE_SIZE + k + INDIRECTLY_CONNECTED;
        }
      }
    }

    __syncthreads();
  }

  _2D_ACCESS(graph, idxY, idxX, num_vertices) = curr_block_buffer[threadIdx.y][threadIdx.x];
}
