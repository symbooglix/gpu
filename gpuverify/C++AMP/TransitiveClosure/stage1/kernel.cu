//pass
//--blockDim=[8,8] --gridDim=[1,1]

#include <hip/hip_runtime.h>

#define _2D_ACCESS(A, y, x, X_DIM) A[(y)*(X_DIM)+(x)]

//////////////////////////////////////////////////////////////////////////////
//// THIS CODE AND INFORMATION IS PROVIDED "AS IS" WITHOUT WARRANTY OF
//// ANY KIND, EITHER EXPRESSED OR IMPLIED, INCLUDING BUT NOT LIMITED TO
//// THE IMPLIED WARRANTIES OF MERCHANTABILITY AND/OR FITNESS FOR A
//// PARTICULAR PURPOSE.
////
//// Copyright (c) Microsoft Corporation. All rights reserved
//////////////////////////////////////////////////////////////////////////////

//----------------------------------------------------------------------------
// File: TransitiveClosure.cpp
//
// Contains the implementation of algorithms which explores connectivity between 
// nodes in a graph and determine shortest path.
// This is based on paper http://www.seas.upenn.edu/~kiderj/research/papers/APSP-gh08-fin-T.pdf
//----------------------------------------------------------------------------

// Defines to help with AMP->OpenCL translation
#define X_DIMENSION 0
#define Y_DIMENSION 1

// Constants - specifies tile size
#define TILE_SIZE (1 << 3)


#define num_vertices (1 << 6)

// State of connection
#define UNCONNECTED 0
#define DIRECTLY_CONNECTED 1
#define INDIRECTLY_CONNECTED 2

//----------------------------------------------------------------------------
// Stage1 - determine connectivity between vertexs' within a TILE - primary
//----------------------------------------------------------------------------

__global__ void transitive_closure_stage1_kernel(unsigned int* graph, int passnum)
{
    
    // Load primary block into shared memory (primary_block_buffer)
    __shared__ unsigned int primary_block_buffer[TILE_SIZE][TILE_SIZE];

    // TODO: check that in OpenCL the order is 0=x, 1=y, 2=z (in AMP it is reversed)
    int idxY = passnum * TILE_SIZE + threadIdx.y;
    int idxX = passnum * TILE_SIZE + threadIdx.x;

    primary_block_buffer[threadIdx.y][threadIdx.x] = _2D_ACCESS(graph, idxY, idxX, num_vertices);

#ifndef MUTATION
     /* BUGINJECT: REMOVE_BARRIER, DOWN */
    __syncthreads();
#endif

    // Now perform the actual Floyd-Warshall algorithm on this block
    for (unsigned int k = 0;
         k < TILE_SIZE; ++k)
    {
        if ( primary_block_buffer[threadIdx.y][threadIdx.x] == UNCONNECTED)
        {
            if ( (primary_block_buffer[threadIdx.y][k] != UNCONNECTED) && (primary_block_buffer[k][threadIdx.x] != UNCONNECTED) )
            {
                primary_block_buffer[threadIdx.y][threadIdx.x] = passnum*TILE_SIZE + k + INDIRECTLY_CONNECTED;
            }
        }

        __syncthreads();
    }

    _2D_ACCESS(graph, idxY, idxX, num_vertices) = primary_block_buffer[threadIdx.y][threadIdx.x];
}

