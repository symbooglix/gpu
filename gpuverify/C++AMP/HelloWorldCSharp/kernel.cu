//pass
//--gridDim=1024 --blockDim=1024

#include <hip/hip_runtime.h>

__global__ void square_array(float* dataView)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  dataView[idx] = dataView[idx] * dataView[idx];
#ifdef MUTATION
  dataView[idx+1] = dataView[idx+1];
#endif
   /* BUGINJECT: ADD_ACCESS, UP */
}
