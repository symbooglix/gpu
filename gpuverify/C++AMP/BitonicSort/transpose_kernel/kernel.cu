#include "hip/hip_runtime.h"
//pass
//--blockDim=[16,16] --gridDim=[32,32]

#include <hip/hip_runtime.h>

//////////////////////////////////////////////////////////////////////////////
//// THIS CODE AND INFORMATION IS PROVIDED "AS IS" WITHOUT WARRANTY OF
//// ANY KIND, EITHER EXPRESSED OR IMPLIED, INCLUDING BUT NOT LIMITED TO
//// THE IMPLIED WARRANTIES OF MERCHANTABILITY AND/OR FITNESS FOR A
//// PARTICULAR PURPOSE.
////
//// Copyright (c) Microsoft Corporation. All rights reserved
//////////////////////////////////////////////////////////////////////////////

#define X_DIMENSION 0
#define Y_DIMENSION 1

// Original kernels are templated.  We will check the float case.
#define _type float

//----------------------------------------------------------------------------
// File: BitonicSort.cpp
// 
// Implements Bitonic sort in C++ AMP
// Supports only int, unsigned, long and unsigned long
//----------------------------------------------------------------------------

#define BITONIC_TILE_SIZE          512
// Should be a square matrix
#define NUM_ELEMENTS                (BITONIC_TILE_SIZE * BITONIC_TILE_SIZE) 
#define MATRIX_WIDTH                BITONIC_TILE_SIZE
#define MATRIX_HEIGHT               BITONIC_TILE_SIZE
// Should be divisible by MATRIX_WIDTH and MATRIX_HEIGHT
// else parallel_for_each will crash
#define TRANSPOSE_TILE_SIZE        16

//----------------------------------------------------------------------------
// Kernel implements 2D matrix transpose
//----------------------------------------------------------------------------
__global__ void transpose_kernel(_type* data_in, _type* data_out, unsigned width, unsigned height)
{
  __requires(width == 512 /*MATRIX_WIDTH*/);

  __shared__ _type transpose_shared_data[TRANSPOSE_TILE_SIZE][TRANSPOSE_TILE_SIZE];

  transpose_shared_data[threadIdx.y][threadIdx.x] = data_in[(blockDim.y*blockIdx.y + threadIdx.y)*width + (blockDim.x*blockIdx.x + threadIdx.x)];

#ifndef MUTATION
   /* BUGINJECT: REMOVE_BARRIER, DOWN */
  __syncthreads();
#endif

  data_out[(blockDim.x*blockIdx.x + threadIdx.x)*width + (blockDim.y*blockIdx.y + threadIdx.y)] = transpose_shared_data[threadIdx.y][threadIdx.x];
}
