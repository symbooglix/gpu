//pass
//--blockDim=512 --gridDim=512

#include <hip/hip_runtime.h>

//////////////////////////////////////////////////////////////////////////////
//// THIS CODE AND INFORMATION IS PROVIDED "AS IS" WITHOUT WARRANTY OF
//// ANY KIND, EITHER EXPRESSED OR IMPLIED, INCLUDING BUT NOT LIMITED TO
//// THE IMPLIED WARRANTIES OF MERCHANTABILITY AND/OR FITNESS FOR A
//// PARTICULAR PURPOSE.
////
//// Copyright (c) Microsoft Corporation. All rights reserved
//////////////////////////////////////////////////////////////////////////////

// Original kernels are templated.  We will check the float case.
#define _type float

//----------------------------------------------------------------------------
// File: BitonicSort.cpp
// 
// Implements Bitonic sort in C++ AMP
// Supports only int, unsigned, long and unsigned long
//----------------------------------------------------------------------------

#define BITONIC_TILE_SIZE          512
// Should be a square matrix
#define NUM_ELEMENTS                (BITONIC_TILE_SIZE * BITONIC_TILE_SIZE) 
#define MATRIX_WIDTH                BITONIC_TILE_SIZE
#define MATRIX_HEIGHT               BITONIC_TILE_SIZE
// Should be divisible by MATRIX_WIDTH and MATRIX_HEIGHT
// else parallel_for_each will crash
#define TRANSPOSE_TILE_SIZE        16

//----------------------------------------------------------------------------
// Kernel implements partial sorting on accelerator, BITONIC_TILE_SIZE at a time
//----------------------------------------------------------------------------
__global__ void bitonic_sort_kernel(_type* data, unsigned ulevel, unsigned ulevelmask)
{
    __shared__ _type sh_data[BITONIC_TILE_SIZE];

    int local_idx = threadIdx.x;
    int global_idx = blockIdx.x*blockDim.x + threadIdx.x;

    // Cooperatively load data - each thread will load data from global memory
    // into tile_static
    sh_data[local_idx] = data[global_idx];

    // Wait till all threads have loaded their portion of data
#ifndef MUTATION
     /* BUGINJECT: REMOVE_BARRIER, DOWN */
    __syncthreads();
#endif
    
    // Sort data in tile_static memory
    for (unsigned int j = ulevel >> 1 ;
        j > 0 ; j >>= 1)
    {
        _type result = ((sh_data[local_idx & ~j] <= sh_data[local_idx | j]) == (bool)(ulevelmask & global_idx)) ? sh_data[local_idx ^ j] : sh_data[local_idx];
        __syncthreads();
        sh_data[local_idx] = result;
        __syncthreads();
    }
    
    // Store shared data
    data[global_idx] = sh_data[local_idx];
}

