//pass
//--blockDim=[16,16] --gridDim=[16,16]

#include <hip/hip_runtime.h>


//////////////////////////////////////////////////////////////////////////////
//// THIS CODE AND INFORMATION IS PROVIDED "AS IS" WITHOUT WARRANTY OF
//// ANY KIND, EITHER EXPRESSED OR IMPLIED, INCLUDING BUT NOT LIMITED TO
//// THE IMPLIED WARRANTIES OF MERCHANTABILITY AND/OR FITNESS FOR A
//// PARTICULAR PURPOSE.
////
//// Copyright (c) Microsoft Corporation. All rights reserved
//////////////////////////////////////////////////////////////////////////////

//----------------------------------------------------------------------------
// File: Matrixmult.cpp
// 
// Implement GPU based matrix multiplication
//----------------------------------------------------------------------------

#define _type float

#define M 256
#define N 256
#define W 256

#define tile_size 16

#define X_DIMENSION 0
#define Y_DIMENSION 1

//----------------------------------------------------------------------------
// Implement tiled version of matrix multiplication
// M, N and W are sizes of matrix
// input matrix - va is of size (M * N), vb is (N * W) 
// output matrix - vresult (M * W)
//----------------------------------------------------------------------------
__global__ void mxm_amp_tiled(const _type * va, const _type * vb, _type * vresult)
{

	{
		__shared__ _type localB[tile_size][tile_size];
		__shared__ _type localA[tile_size][tile_size];

		_type temp_c = 0;

		int localIdxX = threadIdx.x;
                int localIdxY = threadIdx.y;
		int globalIdxX = blockIdx.x*blockDim.x + threadIdx.x;
                int globalIdxY = blockIdx.y*blockDim.y + threadIdx.y;
  
		for (int i = 0;
                       i < N; i += tile_size)
		{

			localA[localIdxY][localIdxX] = va[globalIdxY*M + i + localIdxX];
			localB[localIdxY][localIdxX] = vb[(i + localIdxY)*N + globalIdxX];
#ifndef MUTATION
    /* BUGINJECT: REMOVE_BARRIER, DOWN */
			__syncthreads();
#endif
        
			for (unsigned int k = 0; k < tile_size; k++)
			{
				temp_c += localA[localIdxY][k] * localB[k][localIdxX];
			}

                        __syncthreads();       
		}

		vresult[globalIdxY*M + globalIdxX] = temp_c;
	}
}
