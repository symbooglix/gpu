//pass
//--blockDim=[64,64] --gridDim=[4,4]

#include <hip/hip_runtime.h>

//////////////////////////////////////////////////////////////////////////////
//// THIS CODE AND INFORMATION IS PROVIDED "AS IS" WITHOUT WARRANTY OF
//// ANY KIND, EITHER EXPRESSED OR IMPLIED, INCLUDING BUT NOT LIMITED TO
//// THE IMPLIED WARRANTIES OF MERCHANTABILITY AND/OR FITNESS FOR A
//// PARTICULAR PURPOSE.
////
//// Copyright (c) Microsoft Corporation. All rights reserved
//////////////////////////////////////////////////////////////////////////////

//----------------------------------------------------------------------------
// File: Matrixmult.cpp
// 
// Implement GPU based matrix multiplication
//----------------------------------------------------------------------------

#define _type float

#define M 256
#define N 256
#define W 256

#define X_DIMENSION 0
#define Y_DIMENSION 1


//----------------------------------------------------------------------------
// Implement simple matrix multiplication on GPU using C++ AMP
// M, N and W are sizes of matrix
// input matrix - va is of size (M * N), vb is (N * W) 
// output matrix - vresult (M * W)
//----------------------------------------------------------------------------
__global__ void mxm_amp_simple(const _type * va, const _type * vb, _type * vresult)
{
    // Compute - outer 2 for loops of CPU are replaced by a parallel_for_each
        {
            _type result = 0.0f;

            for(int i = 0; i < N; ++i)
            {
                int idx_a_X = i;
                int idx_a_Y = blockIdx.y*blockDim.y + threadIdx.y;

                int idx_b_X = blockIdx.x*blockDim.x + threadIdx.x;
                int idx_b_Y = i; 

                result += va[idx_a_Y*M + idx_a_X] * vb[idx_b_Y*N + idx_b_X];
            }

            vresult[(blockIdx.y*blockDim.y + threadIdx.y)*M + (blockIdx.x*blockDim.x + threadIdx.x)] = result;
#ifdef MUTATION
            vresult[(blockIdx.y*blockDim.y + threadIdx.y)*M + (blockIdx.x*blockDim.x + threadIdx.x) + 1] = vresult[(blockIdx.y*blockDim.y + threadIdx.y)*M + (blockIdx.x*blockDim.x + threadIdx.x) + 1];
             /* BUGINJECT: ADD_ACCESS, UP */
#endif
        }
}
