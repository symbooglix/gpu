//pass
//--blockDim=256 --gridDim=512

//ALTERNATIVELY:
//--blockDim=16 --gridDim=8 -DSMALL


#include <hip/hip_runtime.h>

//#define SMALL

#define fast_min(x, y) ((x) < (y) ? (x) : (y))

//////////////////////////////////////////////////////////////////////////////
//// THIS CODE AND INFORMATION IS PROVIDED "AS IS" WITHOUT WARRANTY OF
//// ANY KIND, EITHER EXPRESSED OR IMPLIED, INCLUDING BUT NOT LIMITED TO
//// THE IMPLIED WARRANTIES OF MERCHANTABILITY AND/OR FITNESS FOR A
//// PARTICULAR PURPOSE.
////
//// Copyright (c) Microsoft Corporation. All rights reserved
//////////////////////////////////////////////////////////////////////////////

//----------------------------------------------------------------------------
// File: BinomialOptions.cpp
// 
// Implement GPU based binomial option pricing. Verify correctness with CPU 
// implementation
//----------------------------------------------------------------------------

#ifdef SMALL

// Date set - small and normal
// small problem size
#define  MAX_OPTIONS    (32)
#define  NUM_STEPS      (64)
#define  TIME_STEPS     (2)
#define  CACHE_DELTA    (2 * TIME_STEPS)
#define  CACHE_SIZE     (16)
#define  CACHE_STEP     (CACHE_SIZE - CACHE_DELTA)

#else

// normal problem size
#define  MAX_OPTIONS    (512)
#define  NUM_STEPS      (2048)
#define  TIME_STEPS     (16)
#define  CACHE_DELTA    (2 * TIME_STEPS)
#define  CACHE_SIZE     (256)
#define  CACHE_STEP     (CACHE_SIZE - CACHE_DELTA)

#endif

#if NUM_STEPS % CACHE_DELTA
    #error Bad constants
#endif


//----------------------------------------------------------------------------
// GPU implementation - Call value at period t : V(t) = S(t) - X
//----------------------------------------------------------------------------
static __attribute__((always_inline)) __device__ float expiry_call_value(float s, float x, float vdt, int t)
{
    float d = s * exp(vdt * (2.0f * t - NUM_STEPS)) - x;
    return (d > 0) ? d : 0;
}

//----------------------------------------------------------------------------
// GPU implementation of binomial options tree walking to calculate option pricing
// Refer README.txt for more details on algorithm
//----------------------------------------------------------------------------
// Using #ifdef to workaround an exception on Window 7 and Debug build
// Runtime throw's an exception:
//		ID3D11DeviceContext::Dispatch: The Shader Resource View in slot 0 of the Compute 
//  Shader unit is a Structured Buffer while the shader expects a typed Buffer.  This 
// mismatch is invalid if the shader actually uses the view (e.g. it is not skipped due to shader code branching).
// This issue will be fixed in next release.
__global__ void binomial_options_kernel(
                   const float* s, const float* x, 
                   const float* vdt, const float* pu_by_df, 
                   const float* pd_by_df,
                   float* call_value, 
                   float* call_buffer) 
{
  int tile_idx = blockIdx.x;
  int local_idx = threadIdx.x;

  __shared__ float call_a[CACHE_SIZE+1];
  __shared__ float call_b[CACHE_SIZE+1];

  //Global memory frame for current option (thread group)
  int tid = local_idx;

  // CACHE_SIZE number of thread are operating, hence steping by CACHE_SIZE
  // below for loop is similar to first inner loop of binomial_options_cpu
  //Compute values at expiry date
  for(int index = tid; index <= NUM_STEPS; index += CACHE_SIZE)
  {
    int idxA = tile_idx * (NUM_STEPS + 16) + (index);
    call_buffer[idxA] = expiry_call_value(s[tile_idx], x[tile_idx], vdt[tile_idx], index);
  }

  // Walk down binomial tree - equivalent to 2nd inner loop of binomial_options_cpu
  //                              Additional boundary checking 
  // So double-buffer and synchronize to avoid read-after-write hazards.
  for(int i = NUM_STEPS; i > 0; i -= CACHE_DELTA)
  {

    for(int c_base = 0; c_base < i; c_base += CACHE_STEP)
    {
      // Start and end positions within shared memory cache
      int c_start = fast_min(CACHE_SIZE - 1, i - c_base);
      int c_end   = c_start - CACHE_DELTA;

      // Read data(with apron) to shared memory
#ifndef MUTATION
       /* BUGINJECT: REMOVE_BARRIER, DOWN */
      __syncthreads();
#endif
      if(tid <= c_start)
      {
        int idxB = tile_idx * (NUM_STEPS + 16) + (c_base + tid);
        call_a[tid] = call_buffer[idxB];
      }

      // Calculations within shared memory
      for(int k = c_start - 1; 
        k >= c_end;)
      {
        // Compute discounted expected value
        __syncthreads();
        call_b[tid] = pu_by_df[tile_idx] * call_a[tid + 1] + pd_by_df[tile_idx] * call_a[tid];
        k--;

        // Compute discounted expected value
        __syncthreads();
        call_a[tid] = pu_by_df[tile_idx] * call_b[tid + 1] + pd_by_df[tile_idx] * call_b[tid];
        k--;
      }

      // Flush shared memory cache
      __syncthreads();
      if(tid <= c_end)
      {
        int idxC = tile_idx * (NUM_STEPS + 16) + (c_base + tid);
        call_buffer[idxC] = call_a[tid];
      }
    }
  }

  // Write the value at the top of the tree to destination buffer
  if (tid == 0) 
    call_value[tile_idx] = call_a[0];
}
