#include "hip/hip_runtime.h"
//pass
//--blockDim=32 --gridDim=2

#include "sha1_common.h"

/*==========================================================================
                                SHA1 KERNEL

* Copyright (c) 2008, NetSysLab at the University of British Columbia
* All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions are met:
*     * Redistributions of source code must retain the above copyright
*       notice, this list of conditions and the following disclaimer.
*     * Redistributions in binary form must reproduce the above copyright
*       notice, this list of conditions and the following disclaimer in the
*       documentation and/or other materials provided with the distribution.
*     * Neither the name of the University nor the
*       names of its contributors may be used to endorse or promote products
*       derived from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY NetSysLab ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
* WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
* DISCLAIMED. IN NO EVENT SHALL NetSysLab BE LIABLE FOR ANY
* DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
* (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
* LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
* ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
* SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

DESCRIPTION
  CPU version of the storeGPU library.


==========================================================================*/


__global__
void sha1_overlap( unsigned char *input, int chunkSize, int offset,
		   int totalThreads, int padSize, unsigned char *output ) {
  __requires(chunkSize == 52);
  __requires(offset == 4);
  __requires(totalThreads == 49152);
  __requires(padSize == 0);

  int threadIndex = threadIdx.x + blockDim.x * blockIdx.x;
  int chunkIndex = threadIndex * offset;
  int hashIndex  = threadIndex * SHA1_HASH_SIZE;

  if(threadIndex >= totalThreads)
    return;
  
  if ((threadIndex == (totalThreads - 1))) {
    chunkSize-= padSize;
  }

#ifdef FEATURE_SHARED_MEMORY
  
  __shared__ unsigned int sharedMemory[4 * 1024 - 32];
  
    //NOTE : SAMER : this can exceed the size of the shared memory 
  unsigned int *sharedMemoryIndex = sharedMemory + ((threadIdx.x >> 5) * 512);
  unsigned int *inputIndex = (unsigned int *)(input + chunkIndex);
  
  sha1_internal_overlap(inputIndex, sharedMemoryIndex, chunkSize, 
	       output + hashIndex );

#else
  sha1_internal(input + chunkIndex, chunkSize, output + hashIndex );
#endif /* FEATURE_SHARED_MEMORY */


}
