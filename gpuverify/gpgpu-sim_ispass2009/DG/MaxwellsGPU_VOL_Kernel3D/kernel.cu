#include "hip/hip_runtime.h"
//pass
//--blockDim=32 --gridDim=2

#include "../common.h"

__global__ void MaxwellsGPU_VOL_Kernel3D(float *g_rhsQ){

  /* fastest */
  __device__ __shared__ float s_Q[p_Nfields*BSIZE];
  __device__ __shared__ float s_facs[12];

  const int n = threadIdx.x;
  const int k = blockIdx.x;
  
  /* "coalesced"  */
  int m = n+k*p_Nfields*BSIZE;
  int id = n;
  s_Q[id] = tex1Dfetch(t_Q, m); m+=BSIZE; id+=BSIZE;
  s_Q[id] = tex1Dfetch(t_Q, m); m+=BSIZE; id+=BSIZE;
  s_Q[id] = tex1Dfetch(t_Q, m); m+=BSIZE; id+=BSIZE;
  s_Q[id] = tex1Dfetch(t_Q, m); m+=BSIZE; id+=BSIZE;
  s_Q[id] = tex1Dfetch(t_Q, m); m+=BSIZE; id+=BSIZE;
  s_Q[id] = tex1Dfetch(t_Q, m); 

  if(p_Np<12 && n==0)
    for(m=0;m<12;++m)
      s_facs[m] = tex1Dfetch(t_vgeo, 12*k+m);
  else if(n<12 && p_Np>=12)
    s_facs[n] = tex1Dfetch(t_vgeo, 12*k+n);

  __syncthreads();

  float dHxdr=0,dHxds=0,dHxdt=0;
  float dHydr=0,dHyds=0,dHydt=0;
  float dHzdr=0,dHzds=0,dHzdt=0;
  float dExdr=0,dExds=0,dExdt=0;
  float dEydr=0,dEyds=0,dEydt=0;
  float dEzdr=0,dEzds=0,dEzdt=0;
  float Q;

  for(m=0;p_Np-m;){
    float4 D = tex1Dfetch(t_DrDsDt, n+m*BSIZE);

    id = m;
    Q = s_Q[id]; dHxdr += D.x*Q; dHxds += D.y*Q; dHxdt += D.z*Q; id += BSIZE;
    Q = s_Q[id]; dHydr += D.x*Q; dHyds += D.y*Q; dHydt += D.z*Q; id += BSIZE;
    Q = s_Q[id]; dHzdr += D.x*Q; dHzds += D.y*Q; dHzdt += D.z*Q; id += BSIZE;
    Q = s_Q[id]; dExdr += D.x*Q; dExds += D.y*Q; dExdt += D.z*Q; id += BSIZE;
    Q = s_Q[id]; dEydr += D.x*Q; dEyds += D.y*Q; dEydt += D.z*Q; id += BSIZE;
    Q = s_Q[id]; dEzdr += D.x*Q; dEzds += D.y*Q; dEzdt += D.z*Q; 

    ++m;
#if ( (p_Np) % 2 )==0
    D = tex1Dfetch(t_DrDsDt, n+m*BSIZE);

    id = m;
    Q = s_Q[id]; dHxdr += D.x*Q; dHxds += D.y*Q; dHxdt += D.z*Q; id += BSIZE;
    Q = s_Q[id]; dHydr += D.x*Q; dHyds += D.y*Q; dHydt += D.z*Q; id += BSIZE;
    Q = s_Q[id]; dHzdr += D.x*Q; dHzds += D.y*Q; dHzdt += D.z*Q; id += BSIZE;
    Q = s_Q[id]; dExdr += D.x*Q; dExds += D.y*Q; dExdt += D.z*Q; id += BSIZE;
    Q = s_Q[id]; dEydr += D.x*Q; dEyds += D.y*Q; dEydt += D.z*Q; id += BSIZE;
    Q = s_Q[id]; dEzdr += D.x*Q; dEzds += D.y*Q; dEzdt += D.z*Q; 

    ++m;

#if ( (p_Np)%3 )==0
    D = tex1Dfetch(t_DrDsDt, n+m*BSIZE);

    id = m;
    Q = s_Q[id]; dHxdr += D.x*Q; dHxds += D.y*Q; dHxdt += D.z*Q; id += BSIZE;
    Q = s_Q[id]; dHydr += D.x*Q; dHyds += D.y*Q; dHydt += D.z*Q; id += BSIZE;
    Q = s_Q[id]; dHzdr += D.x*Q; dHzds += D.y*Q; dHzdt += D.z*Q; id += BSIZE;
    Q = s_Q[id]; dExdr += D.x*Q; dExds += D.y*Q; dExdt += D.z*Q; id += BSIZE;
    Q = s_Q[id]; dEydr += D.x*Q; dEyds += D.y*Q; dEydt += D.z*Q; id += BSIZE;
    Q = s_Q[id]; dEzdr += D.x*Q; dEzds += D.y*Q; dEzdt += D.z*Q; 

    ++m;
#endif
#endif
  }
  
  const float drdx= s_facs[0];
  const float drdy= s_facs[1];
  const float drdz= s_facs[2];
  const float dsdx= s_facs[4];
  const float dsdy= s_facs[5];
  const float dsdz= s_facs[6];
  const float dtdx= s_facs[8];
  const float dtdy= s_facs[9];
  const float dtdz= s_facs[10];
  
  m = n+p_Nfields*BSIZE*k;

  g_rhsQ[m] = -(drdy*dEzdr+dsdy*dEzds+dtdy*dEzdt - drdz*dEydr-dsdz*dEyds-dtdz*dEydt); m += BSIZE;
  g_rhsQ[m] = -(drdz*dExdr+dsdz*dExds+dtdz*dExdt - drdx*dEzdr-dsdx*dEzds-dtdx*dEzdt); m += BSIZE;
  g_rhsQ[m] = -(drdx*dEydr+dsdx*dEyds+dtdx*dEydt - drdy*dExdr-dsdy*dExds-dtdy*dExdt); m += BSIZE;
  g_rhsQ[m] =  (drdy*dHzdr+dsdy*dHzds+dtdy*dHzdt - drdz*dHydr-dsdz*dHyds-dtdz*dHydt); m += BSIZE;
  g_rhsQ[m] =  (drdz*dHxdr+dsdz*dHxds+dtdz*dHxdt - drdx*dHzdr-dsdx*dHzds-dtdx*dHzdt); m += BSIZE;
  g_rhsQ[m] =  (drdx*dHydr+dsdx*dHyds+dtdx*dHydt - drdy*dHxdr-dsdy*dHxds-dtdy*dHxdt); 
}
