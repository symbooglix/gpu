#include "hip/hip_runtime.h"
//pass
//--blockDim=32 --gridDim=2

#include "../common.h"

__global__ void MaxwellsGPU_SURF_Kernel3D(float *g_Q, float *g_rhsQ){

  __device__ __shared__ float s_fluxQ[p_Nfields*p_Nfp*p_Nfaces];

  const int n = threadIdx.x;
  const int k = blockIdx.x;
  int m;

  /* grab surface nodes and store flux in shared memory */
  if(n< (p_Nfp*p_Nfaces) ){
    /* coalesced reads (maybe) */
    m = 7*(k*p_Nfp*p_Nfaces)+n;
    const  int idM   = tex1Dfetch(t_surfinfo, m); m += p_Nfp*p_Nfaces;
           int idP   = tex1Dfetch(t_surfinfo, m); m += p_Nfp*p_Nfaces;
    const  float Fsc = tex1Dfetch(t_surfinfo, m); m += p_Nfp*p_Nfaces;
    const  float Bsc = tex1Dfetch(t_surfinfo, m); m += p_Nfp*p_Nfaces;
    const  float nx  = tex1Dfetch(t_surfinfo, m); m += p_Nfp*p_Nfaces;
    const  float ny  = tex1Dfetch(t_surfinfo, m); m += p_Nfp*p_Nfaces;
    const  float nz  = tex1Dfetch(t_surfinfo, m);

    /* check if idP<0  */
    double dHx, dHy, dHz, dEx, dEy, dEz;
    if(idP<0){
      idP = p_Nfields*(-1-idP);
      
      dHx = Fsc*(tex1Dfetch(t_partQ, idP+0) - tex1Dfetch(t_Q, idM+0*BSIZE));
      dHy = Fsc*(tex1Dfetch(t_partQ, idP+1) - tex1Dfetch(t_Q, idM+1*BSIZE));
      dHz = Fsc*(tex1Dfetch(t_partQ, idP+2) - tex1Dfetch(t_Q, idM+2*BSIZE));
      
      dEx = Fsc*(tex1Dfetch(t_partQ, idP+3) - tex1Dfetch(t_Q, idM+3*BSIZE));
      dEy = Fsc*(tex1Dfetch(t_partQ, idP+4) - tex1Dfetch(t_Q, idM+4*BSIZE));
      dEz = Fsc*(tex1Dfetch(t_partQ, idP+5) - tex1Dfetch(t_Q, idM+5*BSIZE));
    }
    else{
      dHx = Fsc*(tex1Dfetch(t_Q, idP+0*BSIZE) - tex1Dfetch(t_Q, idM+0*BSIZE));
      dHy = Fsc*(tex1Dfetch(t_Q, idP+1*BSIZE) - tex1Dfetch(t_Q, idM+1*BSIZE));
      dHz = Fsc*(tex1Dfetch(t_Q, idP+2*BSIZE) - tex1Dfetch(t_Q, idM+2*BSIZE));
      
      dEx = Fsc*(Bsc*tex1Dfetch(t_Q, idP+3*BSIZE) - tex1Dfetch(t_Q, idM+3*BSIZE));
      dEy = Fsc*(Bsc*tex1Dfetch(t_Q, idP+4*BSIZE) - tex1Dfetch(t_Q, idM+4*BSIZE));
      dEz = Fsc*(Bsc*tex1Dfetch(t_Q, idP+5*BSIZE) - tex1Dfetch(t_Q, idM+5*BSIZE));
    }

    const double ndotdH = nx*dHx + ny*dHy + nz*dHz;
    const double ndotdE = nx*dEx + ny*dEy + nz*dEz;

    m = n;
    s_fluxQ[m] = -ny*dEz + nz*dEy + dHx - ndotdH*nx; m += p_Nfp*p_Nfaces;
    s_fluxQ[m] = -nz*dEx + nx*dEz + dHy - ndotdH*ny; m += p_Nfp*p_Nfaces;
    s_fluxQ[m] = -nx*dEy + ny*dEx + dHz - ndotdH*nz; m += p_Nfp*p_Nfaces;

    s_fluxQ[m] =  ny*dHz - nz*dHy + dEx - ndotdE*nx; m += p_Nfp*p_Nfaces;
    s_fluxQ[m] =  nz*dHx - nx*dHz + dEy - ndotdE*ny; m += p_Nfp*p_Nfaces;
    s_fluxQ[m] =  nx*dHy - ny*dHx + dEz - ndotdE*nz; 
  }

  /* make sure all element data points are cached */
  __syncthreads();

  if(n< (p_Np))
  {
    float rhsHx = 0, rhsHy = 0, rhsHz = 0;
    float rhsEx = 0, rhsEy = 0, rhsEz = 0;
    
    int sk = n;
    /* can manually unroll to 4 because there are 4 faces */
    for(m=0;p_Nfaces*p_Nfp-m;){
      const float4 L = tex1Dfetch(t_LIFT, sk); sk+=p_Np;

      /* broadcast */
      int sk1 = m;
      rhsHx += L.x*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsHy += L.x*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsHz += L.x*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsEx += L.x*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsEy += L.x*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsEz += L.x*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      ++m;

      /* broadcast */
      sk1 = m;
      rhsHx += L.y*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsHy += L.y*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsHz += L.y*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsEx += L.y*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsEy += L.y*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsEz += L.y*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      ++m;

      /* broadcast */
      sk1 = m;
      rhsHx += L.z*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsHy += L.z*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsHz += L.z*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsEx += L.z*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsEy += L.z*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsEz += L.z*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      ++m;

      /* broadcast */
      sk1 = m;
      rhsHx += L.w*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsHy += L.w*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsHz += L.w*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsEx += L.w*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsEy += L.w*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      rhsEz += L.w*s_fluxQ[sk1]; sk1 += p_Nfp*p_Nfaces;
      ++m;

    }
    
    m = n+p_Nfields*k*BSIZE;
    g_rhsQ[m] += rhsHx; m += BSIZE;
    g_rhsQ[m] += rhsHy; m += BSIZE;
    g_rhsQ[m] += rhsHz; m += BSIZE;
    g_rhsQ[m] += rhsEx; m += BSIZE;
    g_rhsQ[m] += rhsEy; m += BSIZE;
    g_rhsQ[m] += rhsEz; m += BSIZE;

  }
}
