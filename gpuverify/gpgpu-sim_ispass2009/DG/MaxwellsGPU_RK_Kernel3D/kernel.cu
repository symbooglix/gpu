#include "hip/hip_runtime.h"
//pass
//--blockDim=32 --gridDim=2

#include "../common.h"

__global__ void MaxwellsGPU_RK_Kernel3D(int Ntotal, float *g_resQ, float *g_rhsQ, float *g_Q, float fa, float fb, float fdt){
  
  int n = blockIdx.x * blockDim.x + threadIdx.x;
    
  if(n<Ntotal){
    float rhs = g_rhsQ[n];
    float res = g_resQ[n];
    res = fa*res + fdt*rhs;
    
    g_resQ[n] = res;
    g_Q[n]    += fb*res;
  }

} 
