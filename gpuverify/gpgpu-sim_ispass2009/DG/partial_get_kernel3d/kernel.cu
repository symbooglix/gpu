#include "hip/hip_runtime.h"
//pass
//--blockDim=32 --gridDim=2

#include "../common.h"
__global__ void partial_get_kernel3d(int Ntotal, int *g_index, float *g_partQ){
  
  int n = blockIdx.x * blockDim.x + threadIdx.x;
    
  if(n<Ntotal)
    g_partQ[n] = tex1Dfetch(t_Q, g_index[n]);
  
} 
