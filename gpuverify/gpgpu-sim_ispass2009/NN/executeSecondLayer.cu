
#include <hip/hip_runtime.h>
//pass
//--gridDim=[50,10] --blockDim=[5,5]

__constant__ int kernelTemplate2[25] = {
        0,  1,  2,  3,  4,
        13, 14, 15, 16, 17, 
        26, 27, 28, 29, 30,
        39, 40, 41, 42, 43, 
        52, 53, 54, 55, 56   };

__global__ void executeSecondLayer(float *Layer2_Neurons_GPU, float *Layer2_Weights_GPU,float *Layer3_Neurons_GPU)
{
	int blockID=blockIdx.x;
	int pixelX=threadIdx.x;
	int pixelY=threadIdx.y;


	int weightBegin=blockID*26*6;
	int windowX=pixelX*2;
	int windowY=pixelY*2;
    
	float result=0;

	
	result+=Layer2_Weights_GPU[weightBegin];
	
	if(blockID==1 && pixelX==0 && pixelY==0)
	{
		result+=0;
	}

	++weightBegin;

	for (int i=0; i<25; ++i )
    {
        result+=Layer2_Neurons_GPU[(windowX + 13*windowY +kernelTemplate2[i])+(13*13*6*blockIdx.y)]*Layer2_Weights_GPU[weightBegin+i*6];
        result+=Layer2_Neurons_GPU[(169 + windowX + 13*windowY +kernelTemplate2[i])+(13*13*6*blockIdx.y)]*Layer2_Weights_GPU[weightBegin+i*6+1];
	result+=Layer2_Neurons_GPU[(338 + windowX + 13*windowY + kernelTemplate2[i])+(13*13*6*blockIdx.y)]*Layer2_Weights_GPU[weightBegin+i*6+2];
        result+=Layer2_Neurons_GPU[(507 + windowX + 13*windowY + kernelTemplate2[i])+(13*13*6*blockIdx.y)]*Layer2_Weights_GPU[weightBegin+i*6+3];
        result+=Layer2_Neurons_GPU[(676 + windowX + 13*windowY + kernelTemplate2[i])+(13*13*6*blockIdx.y)]*Layer2_Weights_GPU[weightBegin+i*6+4];
        result+=Layer2_Neurons_GPU[(845 + windowX + 13*windowY + kernelTemplate2[i])+(13*13*6*blockIdx.y)]*Layer2_Weights_GPU[weightBegin+i*6+5];
	}

	result=(1.7159*tanhf(0.66666667*result));

	Layer3_Neurons_GPU[(5*5*blockID+pixelY*5+pixelX)+(1250*blockIdx.y)]=result;
}
