#include "hip/hip_runtime.h"
//pass
//--blockDim=[4,1] --gridDim=[64,64]

// IMPERIAL EDIT: this kernel was commented out
#include "common.h"

__global__  __device__ void renderPixel(uint * result, Node * dnode, uint imageW, uint imageH, float pas, float df)
{
    __requires(imageW == 64);
    __requires(imageH == 64);
	uint id(blockIdx.x + __umul24(blockIdx.y, imageW));
	uint tid(threadIdx.x), x(blockIdx.x), y(blockIdx.y);
	Node node;
	float t(0.0f), tPixel;
	float4 Color(make_float4(0.0f,0.0f,0.0f,1.0f));
	matrice3x4 M(MView);
	Rayon R;
	Sphere s;
	__shared__ float T[numObj];
	__shared__ uint Obj;

	T[tid] = 10000.0f;
	
	if( x < imageW && y < imageH && tid < numObj ) {
		node = dnode[tid];
		if( tid == 0 ) result[id] = 0;
		tPixel = 2.0f/float(min(imageW,imageH));
		R.A = make_float3(M.m[0].w,M.m[1].w,M.m[2].w);
		R.u = make_float3(M.m[0])*df
			+ make_float3(M.m[2])*(float(x)-float(imageW)*0.5f)*tPixel
			+ make_float3(M.m[1])*(float(y)-float(imageH)*0.5f)*tPixel;
		R.u = normalize(R.u);
		
		s = node.s;
		s.C.x += pas;

		if( node.fg )
			t = intersectionPlan(R,s.C,s.C);
		else
			t = intersectionSphere(R,s.C,s.r);

		T[tid] = t;

		__syncthreads();

		if( tid == 0 ) {
			float tmp(t);
			Obj = 0;
			for( int i(1); i < numObj; i++ ) {
				if( T[i] > 0.0f && ( tmp == 0.0f || T[i] < tmp ) ) {
					tmp = T[i];
					Obj = i;
				}
			}
		}

		__syncthreads();

		if( tid == Obj && t > 0.0f ) {
			s = node.s;
			s.C.x += pas;
			float3 P(R.A+R.u*t), L(normalize(make_float3(0,1,2)-P)), V(-1*R.u);
			float3 N(node.fg?getNormaleP(P):getNormale(P,s.C));
			if( dot(N,L) > 0.0f ) {
				Color = 0.5f*make_float4(s.R,s.V,s.B,s.A)*(max(0.0f,dot(N,L)));
            #ifdef FIXED_CONST_PARSE
				Color += 0.8f*make_float4(1.0f,1.0f,1.0f,1.0f)*pow(max(0.0f,min(1.0f,dot(2.0f*N*dot(N,L)-L,V))),20.0f);
            #else
            Color += 0.8f*make_float4(1.0f,1.0f,1.0f,1.0f)*float2int_pow20(max(0.0f,min(1.0f,dot(2.0f*N*dot(N,L)-L,V))));
            #endif
			}
			result[id] = rgbaFloatToInt(Color);
		}
	}

}
