#include "hip/hip_runtime.h"
//pass
//--gridDim=128              --blockDim=[128,3,1]

#define MUL(a, b) __umul24(a, b)
#define QRNG_DIMENSIONS 3
#define QRNG_RESOLUTION 31
#define INT_SCALE (1.0f / (float)0x80000001U)

static __constant__ unsigned int c_Table[QRNG_DIMENSIONS][QRNG_RESOLUTION];

__global__ void quasirandomGeneratorKernel(
    float *d_Output,
    unsigned int seed,
    unsigned int N
)
{
    __requires(N == 1048576);
    unsigned int *dimBase = &c_Table[threadIdx.y][0];
    unsigned int      tid = MUL(blockDim.x, blockIdx.x) + threadIdx.x;
    unsigned int  threadN = MUL(blockDim.x, gridDim.x);

    for (unsigned int pos = tid; pos < N; pos += threadN)
    {
        unsigned int result = 0;
        unsigned int data = seed + pos;

        for (int bit = 0; bit < QRNG_RESOLUTION; bit++, data >>= 1)
            if (data & 1)
            {
                result ^= dimBase[bit];
            }

        d_Output[MUL(threadIdx.y, N) + pos] = (float)(result + 1) * INT_SCALE;
    }
}
