#include "hip/hip_runtime.h"
//pass
//--gridDim=256              --blockDim=256

__global__ void rngSetupStates(
    hiprandState *rngState,
    unsigned long long seed,
    unsigned long long offset)
{
    // determine global thread id
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Each thread gets the same seed, a different
    // sequence number. A different offset is used for
    // each device.
    hiprand_init(seed, tid, offset, &rngState[tid]);
}
