#include "hip/hip_runtime.h"
//pass
//--gridDim=256 --blockDim=256 --warp-sync=32 -DUNROLL_REDUCTION

// Notes:
// There are two reductions given in reduction.h
// If the unrolled reduction is used then we need to rely on implicit warpsyncs

//Loop unrolled
//--gridDim=256 --blockDim=256 --warp-sync=32 -DUNROLL_REDUCTION
//Nested loops
//--gridDim=256 --blockDim=256

//REQUIRES: const array as formal (imperial edit)

#ifndef DOUBLE_PRECISION
typedef float real;
#else
typedef double real;
#endif

#include "reduction.h"
#define MAX_OPTIONS 512
#define THREAD_N 256

typedef struct
{
    real S;
    real X;
    real MuByT;
    real VBySqrtT;
} __TOptionData;

typedef struct {
    real Expected;
    real Confidence;
} __TOptionValue;

#if 0 // imperial edit
static __device__ __constant__ __TOptionData d_OptionData[MAX_OPTIONS];
static __device__ __TOptionValue d_CallValue[MAX_OPTIONS];
#endif

__device__ static __attribute__((always_inline)) float endCallValue(float S, float X, float r, float MuByT, float VBySqrtT)
{
    float callValue = S * __expf(MuByT + VBySqrtT * r) - X;
    return (callValue > 0) ? callValue : 0;
}

__global__ void MonteCarloOneBlockPerOption(
    __TOptionData *d_OptionData,   // imperial edit
    __TOptionValue * d_CallValue,  // imperial edit
    hiprandState *rngStates,
    int pathN)
{
    const int SUM_N = THREAD_N;
    __shared__ real s_SumCall[SUM_N];
    __shared__ real s_Sum2Call[SUM_N];

    const int optionIndex = blockIdx.x;
    const real        S = d_OptionData[optionIndex].S;
    const real        X = d_OptionData[optionIndex].X;
    const real    MuByT = d_OptionData[optionIndex].MuByT;
    const real VBySqrtT = d_OptionData[optionIndex].VBySqrtT;

    // determine global thread id
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Copy random number state to local memory for efficiency
    hiprandState localState = rngStates[tid];

    //Cycle through the entire samples array:
    //derive end stock price for each path
    //accumulate partial integrals into intermediate shared memory buffer
    for (int iSum = threadIdx.x;
         __invariant(threadIdx.x <= iSum),
         __invariant(               iSum < SUM_N + blockDim.x),
         __invariant(iSum % blockDim.x == threadIdx.x),
         __global_invariant(__write_implies(s_SumCall, __write_offset_bytes(s_SumCall)/sizeof(real) % blockDim.x == threadIdx.x)),
         __global_invariant(__write_implies(s_Sum2Call, __write_offset_bytes(s_Sum2Call)/sizeof(real) % blockDim.x == threadIdx.x)),
         iSum < SUM_N; iSum += blockDim.x)
    {
        __TOptionValue sumCall = {0, 0};

        for (int i = iSum; i < pathN; i += SUM_N)
        {
            real              r = hiprand_normal(&localState);
            real      callValue = endCallValue(S, X, r, MuByT, VBySqrtT);
            sumCall.Expected   += callValue;
            sumCall.Confidence += callValue * callValue;
        }

        s_SumCall[iSum]  = sumCall.Expected;
        s_Sum2Call[iSum] = sumCall.Confidence;
    }

    // store random number state back to global memory
    rngStates[tid] = localState;

    //Reduce shared memory accumulators
    //and write final result to global memory
    sumReduce<real, SUM_N, THREAD_N>(s_SumCall, s_Sum2Call);

    if (threadIdx.x == 0)
    {
        __TOptionValue t = {s_SumCall[0], s_Sum2Call[0]};
        d_CallValue[optionIndex] = t;
    }
}
