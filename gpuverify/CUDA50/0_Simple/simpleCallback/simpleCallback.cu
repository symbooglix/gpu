
#include <hip/hip_runtime.h>
//pass
//--gridDim=[196,1,1]      --blockDim=[512,1,1]
  
__global__
void incKernel(int *data, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N)
        data[i]++;
}
