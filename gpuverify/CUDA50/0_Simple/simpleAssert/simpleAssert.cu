#include "hip/hip_runtime.h"
//xfail:NOT_ALL_VERIFIED
//--gridDim=[2,1,1]        --blockDim=[32,1,1]

#define assert __assert

__global__ void testKernel(int N)
{
    int gtid = blockIdx.x*blockDim.x + threadIdx.x ;
    assert(gtid < N) ;
}
