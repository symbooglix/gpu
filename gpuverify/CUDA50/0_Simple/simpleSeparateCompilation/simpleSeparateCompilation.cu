#include "hip/hip_runtime.h"
//pass
//--gridDim=[1,1,1]        --blockDim=[1024,1,1] --no-inline

__device__ float multiplyByTwo(float number)
{
    return number * 2.0f;
}

__device__ float divideByTwo(float number)
{
    return number * 0.5f;
}

typedef unsigned int uint;
typedef float(*deviceFunc)(float);

__global__ void transformVector(float *v, deviceFunc f, uint size)
{
    __requires(f == multiplyByTwo | f == divideByTwo);
    uint tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size)
    {
        v[tid] = (*f)(v[tid]);
    }
}
