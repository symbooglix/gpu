
#include <hip/hip_runtime.h>
//pass
//--gridDim=[1,1,1]        --blockDim=[4,1,1]

__global__ void kernel(int *g_data)
{
    // write data to global memory
    const unsigned int tid = threadIdx.x;
    int data = g_data[tid];

    // use integer arithmetic to process all four bytes with one thread
    // this serializes the execution, but is the simplest solutions to avoid
    // bank conflicts for this very low number of threads
    // in general it is more efficient to process each byte by a separate thread,
    // to avoid bank conflicts the access pattern should be
    // g_data[4 * wtid + wid], where wtid is the thread id within the half warp
    // and wid is the warp id
    // see also the programming guide for a more in depth discussion.
    g_data[tid] = ((((data <<  0) >> 24) - 10) << 24)
                  | ((((data <<  8) >> 24) - 10) << 16)
                  | ((((data << 16) >> 24) - 10) <<  8)
                  | ((((data << 24) >> 24) - 10) <<  0);
}
