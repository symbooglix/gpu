
#include <hip/hip_runtime.h>
//pass
//--gridDim=[32768,1,1]    --blockDim=[512,1,1]

__global__ void increment_kernel(int *g_data, int inc_value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    g_data[idx] = g_data[idx] + inc_value;
}
