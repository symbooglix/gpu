
#include <hip/hip_runtime.h>
//pass
//--gridDim=[4,1,1]        --blockDim=[32,1,1]

__global__ void sequence_gpu(int *d_ptr, int length)
{
    int elemID = blockIdx.x * blockDim.x + threadIdx.x;

    if (elemID < length)
    {
        d_ptr[elemID] = elemID;
    }
}
