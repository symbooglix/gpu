
#include <hip/hip_runtime.h>
//pass
//--gridDim=[1,1,1]        --blockDim=[128,1,1]

#ifndef IMPLEMENT_VOTE_INTRINSICS
__device__ unsigned int any(unsigned int);
#endif

__global__ void VoteAnyKernel1(unsigned int *input, unsigned int *result, int size)
{
    int tx = threadIdx.x;

    result[tx] = any(input[tx]);
}

