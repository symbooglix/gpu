
#include <hip/hip_runtime.h>
//pass
//--gridDim=[1,1,1]        --blockDim=[128,1,1]

#ifndef IMPLEMENT_VOTE_INTRINSICS
__device__ unsigned int all(unsigned int);
#endif

__global__ void VoteAllKernel2(unsigned int *input, unsigned int *result, int size)
{
    int tx = threadIdx.x;

    result[tx] = all(input[tx]);
}
