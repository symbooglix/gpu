#include "hip/hip_runtime.h"
//pass
//--gridDim=[64,64,1]      --blockDim=[8,8,1]

texture<float, 2, hipReadModeElementType> tex;

__global__ void transformKernel(float *outputData,
                                int width,
                                int height,
                                float theta)
{
    __requires(width == 64*8 /*gridDim.x * blockDim.x*/);

    // calculate normalized texture coordinates
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    float u = x / (float) width;
    float v = y / (float) height;

    // transform coordinates
    u -= 0.5f;
    v -= 0.5f;
    float tu = u*cosf(theta) - v*sinf(theta) + 0.5f;
    float tv = v*cosf(theta) + u*sinf(theta) + 0.5f;

    // read from texture and write to global memory
    outputData[y*width + x] = tex2D(tex, tu, tv);
}
