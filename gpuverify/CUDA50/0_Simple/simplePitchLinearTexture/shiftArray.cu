#include "hip/hip_runtime.h"
//pass
//--gridDim=[128,128,1]    --blockDim=[16,16,1]

texture<float, 2, hipReadModeElementType> texRefArray;

__global__ void shiftArray(float *odata,
                           int pitch,
                           int width,
                           int height,
                           int shiftX,
                           int shiftY)
{
    __requires(pitch == 2048);
    __requires(width == 2048);
    __requires(height == 2048);

    int xid = blockIdx.x * blockDim.x + threadIdx.x;
    int yid = blockIdx.y * blockDim.y + threadIdx.y;

    odata[yid * pitch + xid] = tex2D(texRefArray,
                                     (xid + shiftX) / (float) width,
                                     (yid + shiftY) / (float) height);
}
