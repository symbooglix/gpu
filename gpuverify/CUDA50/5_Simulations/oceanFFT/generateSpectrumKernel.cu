#include "hip/hip_runtime.h"
//pass
//--gridDim=[32,32,1] --blockDim=[8,8,1]

__device__ static __attribute__((always_inline)) float2 conjugate(float2 arg);
__device__ static __attribute__((always_inline)) float2 complex_exp(float arg);
__device__ static __attribute__((always_inline)) float2 complex_add(float2 a, float2 b);
__device__ static __attribute__((always_inline)) float2 complex_mult(float2 ab, float2 cd);

__device__ static __attribute__((always_inline))
float2 conjugate(float2 arg)
{
    return make_float2(arg.x, -arg.y);
}

__device__ static __attribute__((always_inline))
float2 complex_exp(float arg)
{
    return make_float2(cosf(arg), sinf(arg));
}

__device__ static __attribute__((always_inline))
float2 complex_add(float2 a, float2 b)
{
    return make_float2(a.x + b.x, a.y + b.y);
}

__device__ static __attribute__((always_inline))
float2 complex_mult(float2 ab, float2 cd)
{
    return make_float2(ab.x * cd.x - ab.y * cd.y, ab.x * cd.y + ab.y * cd.x);
}

__global__ void generateSpectrumKernel(float2 *h0,
                                       float2 *ht,
                                       unsigned int in_width,
                                       unsigned int out_width,
                                       unsigned int out_height,
                                       float t,
                                       float patchSize)
{
    __requires(out_width == 256);
    __requires(out_height == 256);

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int in_index = y*in_width+x;
    unsigned int in_mindex = (out_height - y)*in_width + (out_width - x); // mirrored
    unsigned int out_index = y*out_width+x;

    // calculate wave vector
    float2 k;
    k.x = (-(int)out_width / 2.0f + x) * (2.0f * HIP_PI_F / patchSize);
    k.y = (-(int)out_width / 2.0f + y) * (2.0f * HIP_PI_F / patchSize);

    // calculate dispersion w(k)
    float k_len = sqrtf(k.x*k.x + k.y*k.y);
    float w = sqrtf(9.81f * k_len);

    if ((x < out_width) && (y < out_height))
    {
        float2 h0_k = h0[in_index];
        float2 h0_mk = h0[in_mindex];

        // output frequency-space complex values
        ht[out_index] = complex_add(complex_mult(h0_k, complex_exp(w * t)), complex_mult(conjugate(h0_mk), complex_exp(-w * t)));
        //ht[out_index] = h0_k;
    }
}
