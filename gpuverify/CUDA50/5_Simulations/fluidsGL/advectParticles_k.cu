#include "hip/hip_runtime.h"
//pass
//--gridDim=[8,8] --blockDim=[64,4]

#include "common.h"

__global__ void
advectParticles_k(cData *part, cData *v, int dx, int dy,
                  float dt, int lb, size_t pitch)
{
    __requires(dx == 512);
    __requires(dy == 512);
    __requires(lb == 16);

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    int p;

    // gtidx is the domain location in x for this thread
    cData pterm, vterm;

    if (gtidx < dx)
    {
        for (p = 0; p < lb; p++)
        {
            // fi is the domain location in y for this thread
            int fi = gtidy + p;

            if (fi < dy)
            {
                int fj = fi * dx + gtidx;
                pterm = part[fj];

                int xvi = ((int)(pterm.x * dx));
                int yvi = ((int)(pterm.y * dy));
                vterm = *((cData *)((char *)v + yvi * pitch) + xvi);

                pterm.x += dt * vterm.x;
                pterm.x = pterm.x - (int)pterm.x;
                pterm.x += 1.f;
                pterm.x = pterm.x - (int)pterm.x;
                pterm.y += dt * vterm.y;
                pterm.y = pterm.y - (int)pterm.y;
                pterm.y += 1.f;
                pterm.y = pterm.y - (int)pterm.y;

                part[fj] = pterm;
            }
        } // If this thread is inside the domain in Y
    } // If this thread is inside the domain in X
}
