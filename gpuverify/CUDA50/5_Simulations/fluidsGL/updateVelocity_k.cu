#include "hip/hip_runtime.h"
//pass
//--gridDim=[8,8,1] --blockDim=[64,4,1]

#include "common.h"
  
__global__ void
updateVelocity_k(cData *v, float *vx, float *vy,
                 int dx, int pdx, int dy, int lb, size_t pitch)
{
    __requires(dx == 512);
    __requires(dy == 512);
    __requires(lb == 16);
    __requires(pitch == 4096);

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    int p;

    float vxterm, vyterm;
    cData nvterm;

    // gtidx is the domain location in x for this thread
    if (gtidx < dx)
    {
        for (p = 0;
           __global_invariant(__write_implies(v, __write_offset_bytes(v)/pitch/lb%blockDim.y == threadIdx.y)),
           __global_invariant(__write_implies(v, __write_offset_bytes(v)/pitch/lb/blockDim.y == blockIdx.y)),
             p < lb; p++)
        {
            // fi is the domain location in y for this thread
            int fi = gtidy + p;

            if (fi < dy)
            {
                int fjr = fi * pdx + gtidx;
                vxterm = vx[fjr];
                vyterm = vy[fjr];

                // Normalize the result of the inverse FFT
                float scale = 1.f / (dx * dy);
                nvterm.x = vxterm * scale;
                nvterm.y = vyterm * scale;

                cData *fj = (cData *)((char *)v + fi * pitch) + gtidx;
                *fj = nvterm;
            }
        } // If this thread is inside the domain in Y
    } // If this thread is inside the domain in X
}
