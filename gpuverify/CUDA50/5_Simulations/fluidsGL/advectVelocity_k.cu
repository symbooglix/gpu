#include "hip/hip_runtime.h"
//pass
//--gridDim=[8,8,1] --blockDim=[64,4,1]

#include "common.h"

texture<float2, 2> texref;

__global__ void
advectVelocity_k(cData *v, float *vx, float *vy,
                 int dx, int pdx, int dy, float dt, int lb)
{
    __requires(dx == 512);
    __requires(dy == 512);
    __requires(lb == 16);
    __requires(pdx == 514);

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    int p;

    cData vterm, ploc;
    float vxterm, vyterm;

    // gtidx is the domain location in x for this thread
    if (gtidx < dx)
    {
        for (p = 0;
             __global_invariant(__write_implies(vx, ((__write_offset_bytes(vx)/sizeof(int))%pdx == gtidx))),
             __global_invariant(__write_implies(vx, ((__write_offset_bytes(vx)/sizeof(int)) - gtidx)/pdx < dy)),
             __global_invariant(__write_implies(vy, ((__write_offset_bytes(vy)/sizeof(int))%pdx == gtidx))),
             __global_invariant(__write_implies(vy, ((__write_offset_bytes(vy)/sizeof(int)) - gtidx)/pdx < dy)),
             p < lb; p++)
        {
            // fi is the domain location in y for this thread
            int fi = gtidy + p;

            if (fi < dy)
            {
                int fj = fi * pdx + gtidx;
                vterm = tex2D(texref, (float)gtidx, (float)fi);
                ploc.x = (gtidx + 0.5f) - (dt * vterm.x * dx);
                ploc.y = (fi + 0.5f) - (dt * vterm.y * dy);
                vterm = tex2D(texref, ploc.x, ploc.y);
                vxterm = vterm.x;
                vyterm = vterm.y;
                vx[fj] = vxterm;
                vy[fj] = vyterm;
            }
        }
    }
}
