#include "hip/hip_runtime.h"
//pass
//--gridDim=1 --blockDim=[9,9]

#include "common.h"

__global__ void
addForces_k(cData *v, int dx, int dy, int spx, int spy, float fx, float fy, int r, size_t pitch)
{
    __requires(dx == 512);
    __requires(dy == 512);
    __requires(spx == 1);
    __requires(spy == 1);
    __requires(r == 4);
    __requires(pitch == 4096);

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    cData *fj = (cData *)((char *)v + (ty + spy) * pitch) + tx + spx;

    cData vterm = *fj;
    tx -= r;
    ty -= r;
    float s = 1.f / (1.f + tx*tx*tx*tx + ty*ty*ty*ty);
    vterm.x += s * fx;
    vterm.y += s * fy;
    *fj = vterm;
}
