#include "hip/hip_runtime.h"
//pass
//--gridDim=[40,51] --blockDim=[8,8]

#include "common.h"

__global__ void NLM(
    TColor *dst,
    int imageW,
    int imageH,
    float Noise,
    float lerpC
)
{
    __requires(imageW == 320);
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;
    //Add half of a texel to always address exact texel centers
    const float x = (float)ix + 0.5f;
    const float y = (float)iy + 0.5f;

    if (ix < imageW && iy < imageH)
    {
        //Normalized counter for the NLM weight threshold
        float fCount = 0;
        //Total sum of pixel weights
        float sumWeights = 0;
        //Result accumulator
        float3 clr = {0, 0, 0};

        //Cycle through NLM window, surrounding (x, y) texel
        for (float i = -NLM_WINDOW_RADIUS; i <= NLM_WINDOW_RADIUS; i++)
            for (float j = -NLM_WINDOW_RADIUS; j <= NLM_WINDOW_RADIUS; j++)
            {
                //Find color distance from (x, y) to (x + j, y + i)
                float weightIJ = 0;

                for (float n = -NLM_BLOCK_RADIUS; n <= NLM_BLOCK_RADIUS; n++)
                    for (float m = -NLM_BLOCK_RADIUS; m <= NLM_BLOCK_RADIUS; m++)
                        weightIJ += vecLen(
                                        tex2D(texImage, x + j + m, y + i + n),
                                        tex2D(texImage,     x + m,     y + n)
                                    );

                //Derive final weight from color and geometric distance
                weightIJ     = __expf(-(weightIJ * Noise + (i * i + j * j) * INV_NLM_WINDOW_AREA));

                //Accumulate (x + j, y + i) texel color with computed weight
                float4 clrIJ = tex2D(texImage, x + j, y + i);
                clr.x       += clrIJ.x * weightIJ;
                clr.y       += clrIJ.y * weightIJ;
                clr.z       += clrIJ.z * weightIJ;

                //Sum of weights for color normalization to [0..1] range
                sumWeights  += weightIJ;

                //Update weight counter, if NLM weight for current window texel
                //exceeds the weight threshold
                fCount      += (weightIJ > NLM_WEIGHT_THRESHOLD) ? INV_NLM_WINDOW_AREA : 0;
            }

        //Normalize result color by sum of weights
        sumWeights = 1.0f / sumWeights;
        clr.x *= sumWeights;
        clr.y *= sumWeights;
        clr.z *= sumWeights;

        //Choose LERP quotent basing on how many texels
        //within the NLM window exceeded the weight threshold
        float lerpQ = (fCount > NLM_LERP_THRESHOLD) ? lerpC : 1.0f - lerpC;

        //Write final result to global memory
        float4 clr00 = tex2D(texImage, x, y);
        clr.x = lerpf(clr.x, clr00.x, lerpQ);
        clr.y = lerpf(clr.y, clr00.y, lerpQ);
        clr.z = lerpf(clr.z, clr00.z, lerpQ);
        dst[imageW * iy + ix] = make_color(clr.x, clr.y, clr.z, 0);
    }
}
