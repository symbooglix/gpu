#include "hip/hip_runtime.h"
//pass
//--gridDim=[40,51] --blockDim=[8,8]

#include "common.h"

__global__ void Copy(
    TColor *dst,
    int imageW,
    int imageH
)
{
    __requires(imageW == 320);
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;
    //Add half of a texel to always address exact texel centers
    const float x = (float)ix + 0.5f;
    const float y = (float)iy + 0.5f;

    if (ix < imageW && iy < imageH)
    {
        float4 fresult = tex2D(texImage, x, y);
        dst[imageW * iy + ix] = make_color(fresult.x, fresult.y, fresult.z, 0);
    }
}
