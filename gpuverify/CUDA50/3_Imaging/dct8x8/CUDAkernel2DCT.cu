#include "hip/hip_runtime.h"
//pass
//--gridDim=[16,32,1] --blockDim=[8,4,2] --warp-sync=32

#include "common2.h"

__global__ void CUDAkernel2DCT(float *dst, float *src, int ImgStride)
{
    __requires(ImgStride == 512);
    __shared__ float block[KER2_BLOCK_HEIGHT * KER2_SMEMBLOCK_STRIDE];

    int OffsThreadInRow = threadIdx.y * BLOCK_SIZE + threadIdx.x;
    int OffsThreadInCol = threadIdx.z * BLOCK_SIZE;
    src += FMUL(blockIdx.y * KER2_BLOCK_HEIGHT + OffsThreadInCol, ImgStride) + blockIdx.x * KER2_BLOCK_WIDTH + OffsThreadInRow;
    dst += FMUL(blockIdx.y * KER2_BLOCK_HEIGHT + OffsThreadInCol, ImgStride) + blockIdx.x * KER2_BLOCK_WIDTH + OffsThreadInRow;
    float *bl_ptr = block + OffsThreadInCol * KER2_SMEMBLOCK_STRIDE + OffsThreadInRow;

#pragma unroll

    for (unsigned int i = 0;
         #define tid (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y)
         __global_invariant(__implies(tid/32 == __other_int(tid)/32
                                      & blockIdx.x == __other_int(blockIdx.x)
                                      & blockIdx.y == __other_int(blockIdx.y)
                                      & blockIdx.z == __other_int(blockIdx.z), !__write(block))),
         i < BLOCK_SIZE; i++)
        bl_ptr[i * KER2_SMEMBLOCK_STRIDE] = src[i * ImgStride];

    //process rows
    CUDAsubroutineInplaceDCTvector(block + (OffsThreadInCol + threadIdx.x) * KER2_SMEMBLOCK_STRIDE + OffsThreadInRow - threadIdx.x, 1);

    //process columns
    CUDAsubroutineInplaceDCTvector(bl_ptr, KER2_SMEMBLOCK_STRIDE);

    for (unsigned int i = 0; i < BLOCK_SIZE; i++)
        dst[i * ImgStride] = bl_ptr[i * KER2_SMEMBLOCK_STRIDE];
}
