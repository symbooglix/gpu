#include "hip/hip_runtime.h"
//pass
//--gridDim=512 --blockDim=384

#include "common.h"

#define min(x,y) (x < y ? x : y)
#define max(x,y) (x < y ? y : x)

__global__ void
SobelCopyImage(Pixel *pSobelOriginal, unsigned int Pitch,
               int w, int h, float fscale)
{
    __requires(w == 512);
    __requires(Pitch == 512);

    unsigned char *pSobel =
        (unsigned char *)(((char *) pSobelOriginal)+blockIdx.x*Pitch);

    for (int i = threadIdx.x;
         __global_invariant(i % blockDim.x == threadIdx.x),
         __global_invariant(__write_implies(pSobelOriginal, __write_offset_bytes(pSobelOriginal)%Pitch%blockDim.x == threadIdx.x)),
         i < w; i += blockDim.x)
    {
        pSobel[i] = min(max((tex2D(tex, (float) i, (float) blockIdx.x) * fscale), 0.f), 255.f);
    }
}
