#include "hip/hip_runtime.h"
//pass
//--gridDim=16 --blockDim=64

#include "common.h"
  
__global__ void
d_boxfilter_rgba_x(unsigned int *od, int w, int h, int r)
{
    __requires(w == 1024);
    __requires(h == 1024);

    float scale = 1.0f / (float)((r << 1) + 1);
    unsigned int y = blockIdx.x*blockDim.x + threadIdx.x;

    // as long as address is always less than height, we do work
    if (y < h)
    {
        float4 t = make_float4(0.0f);

        for (int x = -r; x <= r; x++)
        {
            t += tex2D(rgbaTex, x, y);
        }

        od[y * w] = rgbaFloatToInt(t * scale);

        for (int x = 1;
             x < w; x++)
        {
            t += tex2D(rgbaTex, x + r, y);
            t -= tex2D(rgbaTex, x - r - 1, y);
            od[y * w + x] = rgbaFloatToInt(t * scale);
        }
    }
}
