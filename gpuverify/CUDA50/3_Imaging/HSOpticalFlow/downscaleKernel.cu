#include "hip/hip_runtime.h"
//pass
//--gridDim=[10,30]      --blockDim=[32,8]

texture<float, 2, hipReadModeElementType> texFine;

__global__ void DownscaleKernel(int width, int height, int stride, float *out)
{
    __requires(width == 320);
    __requires(height == 240);
    __requires(stride == 320);
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;

    if (ix >= width || iy >= height)
    {
        return;
    }

    float dx = 1.0f/(float)width;
    float dy = 1.0f/(float)height;

    float x = ((float)ix + 0.5f) * dx;
    float y = ((float)iy + 0.5f) * dy;

    out[ix + iy * stride] = 0.25f * (tex2D(texFine, x - dx * 0.25f, y) + tex2D(texFine, x + dx * 0.25f, y) +
                                     tex2D(texFine, x, y - dy * 0.25f) + tex2D(texFine, x, y + dy * 0.25f));
}
