#include "hip/hip_runtime.h"
//pass
//--gridDim=[10,40]      --blockDim=[32,6]

texture<float, 2, hipReadModeElementType> texSource;
texture<float, 2, hipReadModeElementType> texTarget;

__global__ void ComputeDerivativesKernel(int width, int height, int stride,
                                         float *Ix, float *Iy, float *Iz)
{
    __requires(width == 320);
    __requires(height == 240);
    __requires(stride == 320);
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;

    const int pos = ix + iy * stride;

    if (ix >= width || iy >= height) return;

    float dx = 1.0f / (float)width;
    float dy = 1.0f / (float)height;

    float x = ((float)ix + 0.5f) * dx;
    float y = ((float)iy + 0.5f) * dy;

    float t0, t1;
    // x derivative
    t0  = tex2D(texSource, x - 2.0f * dx, y);
    t0 -= tex2D(texSource, x - 1.0f * dx, y) * 8.0f;
    t0 += tex2D(texSource, x + 1.0f * dx, y) * 8.0f;
    t0 -= tex2D(texSource, x + 2.0f * dx, y);
    t0 /= 12.0f;

    t1  = tex2D(texTarget, x - 2.0f * dx, y);
    t1 -= tex2D(texTarget, x - 1.0f * dx, y) * 8.0f;
    t1 += tex2D(texTarget, x + 1.0f * dx, y) * 8.0f;
    t1 -= tex2D(texTarget, x + 2.0f * dx, y);
    t1 /= 12.0f;

    Ix[pos] = (t0 + t1) * 0.5f;

    // t derivative
    Iz[pos] = tex2D(texTarget, x, y) - tex2D(texSource, x, y);

    // y derivative
    t0  = tex2D(texSource, x, y - 2.0f * dy);
    t0 -= tex2D(texSource, x, y - 1.0f * dy) * 8.0f;
    t0 += tex2D(texSource, x, y + 1.0f * dy) * 8.0f;
    t0 -= tex2D(texSource, x, y + 2.0f * dy);
    t0 /= 12.0f;

    t1  = tex2D(texTarget, x, y - 2.0f * dy);
    t1 -= tex2D(texTarget, x, y - 1.0f * dy) * 8.0f;
    t1 += tex2D(texTarget, x, y + 1.0f * dy) * 8.0f;
    t1 -= tex2D(texTarget, x, y + 2.0f * dy);
    t1 /= 12.0f;

    Iy[pos] = (t0 + t1) * 0.5f;
}
