
#include <hip/hip_runtime.h>
//pass
//--gridDim=[1200,1,1]     --blockDim=[256,1,1]
  
__global__
void AddKernel(const float *op1, const float *op2, int count, float *sum)
{
    const int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos >= count) return;

    sum[pos] = op1[pos] + op2[pos];
}
