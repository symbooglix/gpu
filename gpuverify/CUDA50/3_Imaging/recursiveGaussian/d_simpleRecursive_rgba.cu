#include "hip/hip_runtime.h"
//pass
//--gridDim=8 --blockDim=64

#include "common.h"

__global__ void
d_simpleRecursive_rgba(uint *id, uint *od, int w, int h, float a)
{
    __requires(w == 512);
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

    if (x >= w) return;

    id += x;    // advance pointers to correct column
    od += x;

    // forward pass
    float4 yp = rgbaIntToFloat(*id);  // previous output

    for (int y = 0;
         __global_invariant(__implies(x < w, (__ptr_offset_bytes(od)/sizeof(uint))%w == x)),
         __global_invariant(__write_implies(od, (__write_offset_bytes(od)/sizeof(uint))%w == x)),
         y < h; y++)
    {
        float4 xc = rgbaIntToFloat(*id);
        float4 yc = xc + a*(yp - xc);   // simple lerp between current and previous value
        *od = rgbaFloatToInt(yc);
        id += w;
        od += w;    // move to next row
        yp = yc;
    }

    // reset pointers to point to last element in column
    id -= w;
    od -= w;

    // reverse pass
    // ensures response is symmetrical
    yp = rgbaIntToFloat(*id);

    for (int y = h-1;
         __global_invariant(__implies(x < w, (__ptr_offset_bytes(od)/sizeof(uint))%w == x)),
         __global_invariant(__write_implies(od, (__write_offset_bytes(od)/sizeof(uint))%w == x)),
         __global_invariant(__read_implies(od, (__read_offset_bytes(od)/sizeof(uint))%w == x)),
         y >= 0; y--)
    {
        float4 xc = rgbaIntToFloat(*id);
        float4 yc = xc + a*(yp - xc);
        *od = rgbaFloatToInt((rgbaIntToFloat(*od) + yc)*0.5f);
        id -= w;
        od -= w;  // move to previous row
        yp = yc;
    }
}
