#include "hip/hip_runtime.h"
//pass
//--gridDim=[64,256,1]     --blockDim=[32,8,1]

#include "common.h"

__global__ void padDataClampToBorder_kernel(
    float *d_Dst,
    float *d_Src,
    int fftH,
    int fftW,
    int dataH,
    int dataW,
    int kernelH,
    int kernelW,
    int kernelY,
    int kernelX
)
{
    __requires(fftH == 2048);
    __requires(fftW == 2048);

    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int borderH = dataH + kernelY;
    const int borderW = dataW + kernelX;

    if (y < fftH && x < fftW)
    {
        int dy, dx;

        if (y < dataH)
        {
            dy = y;
        }

        if (x < dataW)
        {
            dx = x;
        }

        if (y >= dataH && y < borderH)
        {
            dy = dataH - 1;
        }

        if (x >= dataW && x < borderW)
        {
            dx = dataW - 1;
        }

        if (y >= borderH)
        {
            dy = 0;
        }

        if (x >= borderW)
        {
            dx = 0;
        }

        d_Dst[y * fftW + x] = LOAD_FLOAT(dy * dataW + dx);
    }
}
