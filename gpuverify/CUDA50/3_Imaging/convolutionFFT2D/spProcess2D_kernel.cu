#include "hip/hip_runtime.h"
//pass
//--gridDim=4096             --blockDim=256

#include "common.h"

__global__ void spProcess2D_kernel(
    fComplex *d_Dst,
    fComplex *d_SrcA,
    fComplex *d_SrcB,
    uint DY,
    uint DX,
    uint threadCount,
    float phaseBase,
    float c
)
{
    __requires(DY == 2048);
    __requires(DX == 1024);
    __requires(threadCount == 1048576);

    const uint threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId >= threadCount)
    {
        return;
    }

    uint x, y, i = threadId;
    udivmod(i, DX, x);
    udivmod(i, DY / 2, y);

    const uint offset = i * DY * DX;

    //Avoid overwrites in rows 0 and DY / 2 by different threads (left and right halves)
    //Otherwise correctness for in-place transformations is affected
    if ((y == 0) && (x > DX / 2))
    {
        return;
    }

    fComplex twiddle;

    //Process y = [0 .. DY / 2 - 1] U [DY - (DY / 2) + 1 .. DY - 1]
    {
        const uint pos1 = offset +          y * DX +          x;
        const uint pos2 = offset + mod(y, DY) * DX + mod(x, DX);

        fComplex D1 = LOAD_FCOMPLEX_A(pos1);
        fComplex D2 = LOAD_FCOMPLEX_A(pos2);
        fComplex K1 = LOAD_FCOMPLEX_B(pos1);
        fComplex K2 = LOAD_FCOMPLEX_B(pos2);
        getTwiddle(twiddle, phaseBase * (float)x);

        spPostprocessC2C(D1, D2, twiddle);
        spPostprocessC2C(K1, K2, twiddle);
        mulAndScale(D1, K1, c);
        mulAndScale(D2, K2, c);
        spPreprocessC2C(D1, D2, twiddle);

        d_Dst[pos1] = D1;
        d_Dst[pos2] = D2;
    }

    if (y == 0)
    {
        const uint pos1 = offset + (DY / 2) * DX +          x;
        const uint pos2 = offset + (DY / 2) * DX + mod(x, DX);

        fComplex D1 = LOAD_FCOMPLEX_A(pos1);
        fComplex D2 = LOAD_FCOMPLEX_A(pos2);
        fComplex K1 = LOAD_FCOMPLEX_B(pos1);
        fComplex K2 = LOAD_FCOMPLEX_B(pos2);

        spPostprocessC2C(D1, D2, twiddle);
        spPostprocessC2C(K1, K2, twiddle);
        mulAndScale(D1, K1, c);
        mulAndScale(D2, K2, c);
        spPreprocessC2C(D1, D2, twiddle);

        d_Dst[pos1] = D1;
        d_Dst[pos2] = D2;
    }
}
