#include "hip/hip_runtime.h"
//pass
//--gridDim=8320             --blockDim=256

#include "common.h"

__global__ void modulateAndNormalize_kernel(
    fComplex *d_Dst,
    fComplex *d_Src,
    int dataSize,
    float c
)
{
    const int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= dataSize)
    {
        return;
    }

    fComplex a = d_Src[i];
    fComplex b = d_Dst[i];

    mulAndScale(a, b, c);

    d_Dst[i] = a;
}
