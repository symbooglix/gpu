#include "hip/hip_runtime.h"
//pass
//--gridDim=[192,128,1]    --blockDim=[16,12,1]

#include "common.h"

__global__ void convolutionColumnsKernel(
    float *d_Dst,
    int imageW,
    int imageH
)
{
    __requires(imageW == 3072);
    const   int ix = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
    const   int iy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);
    const float  x = (float)ix + 0.5f;
    const float  y = (float)iy + 0.5f;

    if (ix >= imageW || iy >= imageH)
    {
        return;
    }

    float sum = 0;

#if(UNROLL_INNER)
    sum = convolutionColumn<2 *KERNEL_RADIUS>(x, y);
#else

    for (int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++)
    {
        sum += tex2D(texSrc, x, y + (float)k) * c_Kernel[KERNEL_RADIUS - k];
    }

#endif

    d_Dst[IMAD(iy, imageW, ix)] = sum;
}
