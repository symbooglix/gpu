#include "hip/hip_runtime.h"
//pass
//--gridDim=[32,32,1] --blockDim=[16,16,1]

#include "common.h"

__global__ void
d_render(uchar4 *d_output, uint width, uint height, float tx, float ty, float scale, float cx, float cy)
{
    __requires(width == 512);
    __requires(height == 512);

    uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    uint i = __umul24(y, width) + x;

    float u = (x-cx)*scale+cx + tx;
    float v = (y-cy)*scale+cy + ty;

    if ((x < width) && (y < height))
    {
        // write output color
        float c = tex2D(tex, u, v);
        //float c = tex2DBilinear<uchar, float>(tex, u, v);
        //float c = tex2DBilinearGather<uchar, uchar4>(tex2, u, v, 0) / 255.0f;
        d_output[i] = make_uchar4(c * 0xff, c * 0xff, c * 0xff, 0);
    }
}
