#include "hip/hip_runtime.h"
//pass
//--gridDim=[32,32] --blockDim=[16,16]

typedef unsigned int  uint;
typedef unsigned char uchar;
texture<uint2, 2, hipReadModeElementType> atlasTexture;

__device__ static __attribute__((always_inline)) hipTextureObject_t decodeTextureObject(uint2 obj)
{
    return (((hipTextureObject_t)obj.x) | ((hipTextureObject_t)obj.y) << 32);
}

__device__ static __attribute__((always_inline)) uchar4 to_uchar4(float4 vec)
{
    return make_uchar4((uchar)vec.x, (uchar)vec.y, (uchar)vec.z, (uchar)vec.w);
}

__global__ void
d_render(uchar4 *d_output, uint imageW, uint imageH, float lod)
{
    __requires(imageW == 16*32 /*blockDim.x * gridDim.x*/);

    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    float u = x / (float) imageW;
    float v = y / (float) imageH;

    if ((x < imageW) && (y < imageH))
    {
        // read from 2D atlas texture and decode texture object
        uint2 texCoded = tex2D(atlasTexture, u, v);
        hipTextureObject_t tex = decodeTextureObject(texCoded);

        // read from cuda texture object, use template to specify what data will be
        // returned. tex2DLod allows us to pass the lod (mip map level) directly.
        // There is other functions with CUDA 5, e.g. tex2DGrad,    that allow you
        // to pass derivatives to perform automatic mipmap/anisotropic filtering.
        float4 color = tex2DLod<float4>(tex, u, 1-v, lod);
        // In our sample tex is always valid, but for something like your own
        // sparse texturing you would need to make sure to handle the zero case.

        // write output color
        uint i = y * imageW + x;
        d_output[i] = to_uchar4(color * 255.0);
    }
}
