#include "hip/hip_runtime.h"
//pass
//--gridDim=[32,1,1]       --blockDim=[32,1,1]

//REQUIRES: hipExtent
//REQUIRES: SURFACE

#include "common.h"

__global__ void
d_integrate_trapezoidal(hipExtent extent)
{
    uint x = blockIdx.x*blockDim.x + threadIdx.x;

    // for higher speed could use hierarchical approach for sum
    if (x >= extent.width)
    {
        return;
    }

    float stepsize = 1.0/float(extent.width-1);
    float to = float(x) * stepsize;

    float4 outclr = make_float4(0,0,0,0);
    float incr = stepsize;

    float4 lastval = tex1D(transferTex,0);

    float cur = incr;

    while (cur < to + incr * 0.5)
    {
        float4 val = tex1D(transferTex,cur);
        float4 trapezoid = (lastval+val)/2.0f;
        lastval = val;

        outclr += trapezoid;
        cur += incr;
    }

    // surface writes need byte offsets for x!
    surf1Dwrite(outclr,transferIntegrateSurf,x * sizeof(float4));
}
