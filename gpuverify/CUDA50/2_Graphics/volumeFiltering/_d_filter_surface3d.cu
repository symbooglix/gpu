#include "hip/hip_runtime.h"
//pass
//--gridDim=[1,1,32]     --blockDim=[32,32]

//REQUIRES: SURFACE

#include "common.h"

texture<VolumeType, 3, VolumeTypeInfo<VolumeType>::readMode>  volumeTexIn;
surface<void,  3>                                             volumeTexOut;

__constant__ float4 c_filterData[VOLUMEFILTER_MAXWEIGHTS];
  
__global__ void
d_filter_surface3d(int filterSize, float filter_offset,
                   hipExtent volumeSize)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    if (x >= volumeSize.width || y >= volumeSize.height || z >= volumeSize.depth)
    {
        return;
    }

    float filtered = 0;
    float4 basecoord = make_float4(x,y,z,0);

    for (int i = 0; i < filterSize; i++)
    {
        float4 coord = basecoord + c_filterData[i];
        filtered  += tex3D(volumeTexIn,coord.x,coord.y,coord.z) * c_filterData[i].w;
    }

    filtered    += filter_offset;

    VolumeType output = VolumeTypeInfo<VolumeType>::convert(filtered);

    // surface writes need byte offsets for x!
    surf3Dwrite(output,volumeTexOut,x * sizeof(VolumeType),y,z);

}
