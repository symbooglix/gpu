#include "hip/hip_runtime.h"
//pass
//--gridDim=512 --blockDim=384

typedef unsigned char Pixel;
texture<unsigned char, 2> tex;
#define min(x,y) (x < y ? x : y)
#define max(x,y) (x < y ? y : x)

__global__ void
SobelCopyImage(Pixel *pSobelOriginal, unsigned int Pitch,
               int w, int h, float fscale)
{
    __requires(Pitch == 512);
    __requires(w == 512);

    unsigned char *pSobel =
        (unsigned char *)(((char *) pSobelOriginal)+blockIdx.x*Pitch);

    for (int i = threadIdx.x;
         __global_invariant(i % blockDim.x == threadIdx.x),
         __global_invariant(__write_implies(pSobelOriginal, __write_offset_bytes(pSobelOriginal)/Pitch == blockIdx.x)),
         __global_invariant(__write_implies(pSobelOriginal, __write_offset_bytes(pSobelOriginal)%Pitch%blockDim.x == threadIdx.x)),
         i < w; i += blockDim.x)
    {
        pSobel[i] = min(max((tex2D(tex, (float) i, (float) blockIdx.x) * fscale), 0.f), 255.f);
    }
}
