#include "hip/hip_runtime.h"
//pass
//--gridDim=512 --blockDim=384 --no-inline

#include "common.h"

__global__ void
SobelTex(Pixel *pSobelOriginal, unsigned int Pitch,
         int w, int h, float fScale, int blockOperation, pointFunction_t pPointOperation)
{
    __requires(Pitch == 512);
    __requires(w == 512);
    __requires(h == 512);
    __requires(blockOperation == 0 | blockOperation == 1);
    __requires(blockFunction_table[0] == ComputeSobel);
    __requires(blockFunction_table[1] == ComputeBox);
    __requires(pPointOperation == Threshold | pPointOperation == NULL);
    unsigned char *pSobel =
        (unsigned char *)(((char *) pSobelOriginal)+blockIdx.x*Pitch);
    unsigned char tmp = 0;

    for (int i = threadIdx.x;
        __global_invariant(i % blockDim.x == threadIdx.x),
      //__global_invariant(__write_implies(pSobelOriginal, __write_offset_bytes(pSobelOriginal)/sizeof(Pixel)/Pitch == blockIdx.x)),
        __global_invariant(__write_implies(pSobelOriginal, __write_offset_bytes(pSobelOriginal)/sizeof(Pixel)%Pitch%blockDim.x == threadIdx.x)),
        i < w; i += blockDim.x)
    {
        unsigned char pix00 = tex2D(tex, (float) i-1, (float) blockIdx.x-1);
        unsigned char pix01 = tex2D(tex, (float) i+0, (float) blockIdx.x-1);
        unsigned char pix02 = tex2D(tex, (float) i+1, (float) blockIdx.x-1);
        unsigned char pix10 = tex2D(tex, (float) i-1, (float) blockIdx.x+0);
        unsigned char pix11 = tex2D(tex, (float) i+0, (float) blockIdx.x+0);
        unsigned char pix12 = tex2D(tex, (float) i+1, (float) blockIdx.x+0);
        unsigned char pix20 = tex2D(tex, (float) i-1, (float) blockIdx.x+1);
        unsigned char pix21 = tex2D(tex, (float) i+0, (float) blockIdx.x+1);
        unsigned char pix22 = tex2D(tex, (float) i+1, (float) blockIdx.x+1);
        tmp = (*(blockFunction_table[blockOperation]))(pix00, pix01, pix02,
                                                       pix10, pix11, pix12,
                                                       pix20, pix21, pix22, fScale);

        if (pPointOperation != NULL)
        {
            tmp = (*pPointOperation)(tmp, 150.0);
        }

        pSobel[i] = tmp;
    }
}
