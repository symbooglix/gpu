#include "hip/hip_runtime.h"
//pass
//--gridDim=40 --blockDim=256

typedef unsigned int uint;
__device__ float length(float2);

struct Ray
{
    float3 origin;
    float2 dir;
    int    length;
    float  oneOverLength;
};

__device__ static __attribute__((always_inline)) float2 getLocation(const Ray, int);
__device__ static __attribute__((always_inline)) float getAngle(const Ray, float2, float);

texture<float, 2, hipReadModeElementType> g_HeightFieldTex;

__device__ static __attribute__((always_inline)) float2 getLocation(const Ray ray, int i)
{
    float step = i * ray.oneOverLength;
    return make_float2(ray.origin.x, ray.origin.y) + step * ray.dir;
}

__device__ static __attribute__((always_inline)) float getAngle(const Ray ray, float2 location, float height)
{
    float2 dir = location - make_float2(ray.origin.x, ray.origin.y);
    return atanf((height - ray.origin.z) / length(dir));
}

__global__ void computeAngles_kernel(const Ray ray, float *angles)
{
    uint i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < ray.length)
    {
        float2 location = getLocation(ray, i + 1);
        float height = tex2D(g_HeightFieldTex, location.x, location.y);
        float angle = getAngle(ray, location, height);
        angles[i] = angle;
    }
}
