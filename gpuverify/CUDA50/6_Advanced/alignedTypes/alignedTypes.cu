
#include <hip/hip_runtime.h>
//pass
//--gridDim=64               --blockDim=256

template<class TData> __global__ void testKernel(TData *d_odata, TData *d_idata, int numElements);
template __global__ void testKernel<int>(int *d_odata, int *d_idata, int numElements);

template<class TData> __global__ void testKernel(
    TData *d_odata,
    TData *d_idata,
    int numElements
)
{
    const int        tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    for (int pos = tid; pos < numElements; pos += numThreads)
    {
        d_odata[pos] = d_idata[pos];
    }
}
