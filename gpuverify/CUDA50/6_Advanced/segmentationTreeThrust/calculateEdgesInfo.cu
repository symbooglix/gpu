#include "hip/hip_runtime.h"
//pass
//--gridDim=[11377,1,1]    --blockDim=[256,1,1]

#include "common.h"

__global__ void calculateEdgesInfo(const uint *startpoints,
                                   const uint *verticesMapping,
                                   const uint *edges,
                                   const float *weights,
                                   uint *newStartpoints,
                                   uint *survivedEdgesIDs,
                                   uint edgesCount,
                                   uint newVerticesCount)
{
    uint tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < edgesCount)
    {
        uint startpoint = startpoints[tid];
        uint endpoint = edges[tid];

        newStartpoints[tid] = endpoint < UINT_MAX ?
                              verticesMapping[startpoint] :
                              newVerticesCount + verticesMapping[startpoint];

        survivedEdgesIDs[tid] = endpoint < UINT_MAX ?
                                tid :
                                UINT_MAX;
    }
}
