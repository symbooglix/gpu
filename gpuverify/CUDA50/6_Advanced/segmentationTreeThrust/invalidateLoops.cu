#include "hip/hip_runtime.h"
//pass
//--gridDim=[11377,1,1]    --blockDim=[256,1,1]

#include "common.h"

__global__ void invalidateLoops(const uint *startpoints,
                                const uint *verticesMapping,
                                uint *edges,
                                uint edgesCount)
{
    uint tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < edgesCount)
    {
        uint startpoint = startpoints[tid];
        uint &endpoint = edges[tid];

        uint newStartpoint = verticesMapping[startpoint];
        uint newEndpoint = verticesMapping[endpoint];

        if (newStartpoint == newEndpoint)
        {
            endpoint = UINT_MAX;
        }
    }
}
